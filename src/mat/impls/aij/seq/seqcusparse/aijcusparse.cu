#include "hip/hip_runtime.h"
/*
  Defines the basic matrix operations for the AIJ (compressed row)
  matrix storage format using the CUSPARSE library,
*/
#define PETSC_SKIP_SPINLOCK
#define PETSC_SKIP_CXX_COMPLEX_FIX
#define PETSC_SKIP_IMMINTRIN_H_CUDAWORKAROUND 1

#include <petscconf.h>
#include <../src/mat/impls/aij/seq/aij.h>          /*I "petscmat.h" I*/
#include <../src/mat/impls/sbaij/seq/sbaij.h>
#include <../src/vec/vec/impls/dvecimpl.h>
#include <petsc/private/vecimpl.h>
#undef VecType
#include <../src/mat/impls/aij/seq/seqcusparse/cusparsematimpl.h>
#include <thrust/async/for_each.h>

const char *const MatCUSPARSEStorageFormats[]    = {"CSR","ELL","HYB","MatCUSPARSEStorageFormat","MAT_CUSPARSE_",0};
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  /* The following are copied from hipsparse.h in CUDA-11.0. In MatCUSPARSESpMVAlgorithms[] etc, we copy them in
    0-based integer value order, since we want to use PetscOptionsEnum() to parse user command line options for them.

  typedef enum {
      HIPSPARSE_MV_ALG_DEFAULT = 0,
      HIPSPARSE_COOMV_ALG      = 1,
      HIPSPARSE_CSRMV_ALG1     = 2,
      HIPSPARSE_CSRMV_ALG2     = 3
  } hipsparseSpMVAlg_t;

  typedef enum {
      HIPSPARSE_MM_ALG_DEFAULT     CUSPARSE_DEPRECATED_ENUM(HIPSPARSE_SPMM_ALG_DEFAULT) = 0,
      HIPSPARSE_COOMM_ALG1         CUSPARSE_DEPRECATED_ENUM(HIPSPARSE_SPMM_COO_ALG1)    = 1,
      HIPSPARSE_COOMM_ALG2         CUSPARSE_DEPRECATED_ENUM(HIPSPARSE_SPMM_COO_ALG2)    = 2,
      HIPSPARSE_COOMM_ALG3         CUSPARSE_DEPRECATED_ENUM(HIPSPARSE_SPMM_COO_ALG3)    = 3,
      HIPSPARSE_CSRMM_ALG1         CUSPARSE_DEPRECATED_ENUM(HIPSPARSE_SPMM_CSR_ALG1)    = 4,
      HIPSPARSE_SPMM_ALG_DEFAULT = 0,
      HIPSPARSE_SPMM_COO_ALG1    = 1,
      HIPSPARSE_SPMM_COO_ALG2    = 2,
      HIPSPARSE_SPMM_COO_ALG3    = 3,
      HIPSPARSE_SPMM_COO_ALG4    = 5,
      HIPSPARSE_SPMM_CSR_ALG1    = 4,
      HIPSPARSE_SPMM_CSR_ALG2    = 6,
  } hipsparseSpMMAlg_t;

  typedef enum {
      HIPSPARSE_CSR2CSC_ALG1 = 1, // faster than V2 (in general), deterministc
      HIPSPARSE_CSR2CSC_ALG2 = 2  // low memory requirement, non-deterministc
  } hipsparseCsr2CscAlg_t;
  */
  const char *const MatCUSPARSESpMVAlgorithms[]    = {"MV_ALG_DEFAULT","COOMV_ALG", "CSRMV_ALG1","CSRMV_ALG2", "hipsparseSpMVAlg_t","CUSPARSE_",0};
  const char *const MatCUSPARSESpMMAlgorithms[]    = {"ALG_DEFAULT","COO_ALG1","COO_ALG2","COO_ALG3","CSR_ALG1","COO_ALG4","CSR_ALG2","hipsparseSpMMAlg_t","CUSPARSE_SPMM_",0};
  const char *const MatCUSPARSECsr2CscAlgorithms[] = {"INVALID"/*cusparse does not have enum 0! We created one*/,"ALG1","ALG2","hipsparseCsr2CscAlg_t","CUSPARSE_CSR2CSC_",0};
#endif

static PetscErrorCode MatICCFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,const MatFactorInfo*);
static PetscErrorCode MatCholeskyFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,const MatFactorInfo*);
static PetscErrorCode MatCholeskyFactorNumeric_SeqAIJCUSPARSE(Mat,Mat,const MatFactorInfo*);

static PetscErrorCode MatILUFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,IS,const MatFactorInfo*);
static PetscErrorCode MatLUFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,IS,const MatFactorInfo*);
static PetscErrorCode MatLUFactorNumeric_SeqAIJCUSPARSE(Mat,Mat,const MatFactorInfo*);

static PetscErrorCode MatSolve_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatSolve_SeqAIJCUSPARSE_NaturalOrdering(Mat,Vec,Vec);
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering(Mat,Vec,Vec);
static PetscErrorCode MatSetFromOptions_SeqAIJCUSPARSE(PetscOptionItems *PetscOptionsObject,Mat);
static PetscErrorCode MatAXPY_SeqAIJCUSPARSE(Mat,PetscScalar,Mat,MatStructure);
static PetscErrorCode MatScale_SeqAIJCUSPARSE(Mat,PetscScalar);
static PetscErrorCode MatMult_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatMultAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);
static PetscErrorCode MatMultTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatMultTransposeAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);
static PetscErrorCode MatMultHermitianTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatMultHermitianTransposeAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);
static PetscErrorCode MatMultAddKernel_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec,PetscBool,PetscBool);

static PetscErrorCode CsrMatrix_Destroy(CsrMatrix**);
static PetscErrorCode MatSeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSETriFactorStruct**);
static PetscErrorCode MatSeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSEMultStruct**,MatCUSPARSEStorageFormat);
static PetscErrorCode MatSeqAIJCUSPARSETriFactors_Destroy(Mat_SeqAIJCUSPARSETriFactors**);
static PetscErrorCode MatSeqAIJCUSPARSE_Destroy(Mat_SeqAIJCUSPARSE**);

static PetscErrorCode MatSeqAIJCUSPARSECopyToGPU(Mat);
static PetscErrorCode MatSeqAIJCUSPARSECopyFromGPU(Mat);
static PetscErrorCode MatSeqAIJCUSPARSEInvalidateTranspose(Mat,PetscBool);

PETSC_INTERN PetscErrorCode MatSetPreallocationCOO_SeqAIJCUSPARSE(Mat,PetscInt,const PetscInt[],const PetscInt[]);
PETSC_INTERN PetscErrorCode MatSetValuesCOO_SeqAIJCUSPARSE(Mat,const PetscScalar[],InsertMode);

static PetscErrorCode MatSeqAIJCopySubArray_SeqAIJCUSPARSE(Mat,PetscInt,const PetscInt[],PetscScalar[]);

PetscErrorCode MatCUSPARSESetStream(Mat A,const hipStream_t stream)
{
  hipsparseStatus_t   stat;
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  if (!cusparsestruct) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing spptr");
  cusparsestruct->stream = stream;
  stat = hipsparseSetStream(cusparsestruct->handle,cusparsestruct->stream);CHKERRCUSPARSE(stat);
  PetscFunctionReturn(0);
}

PetscErrorCode MatCUSPARSESetHandle(Mat A,const hipsparseHandle_t handle)
{
  hipsparseStatus_t   stat;
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  if (!cusparsestruct) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing spptr");
  if (cusparsestruct->handle != handle) {
    if (cusparsestruct->handle) {
      stat = hipsparseDestroy(cusparsestruct->handle);CHKERRCUSPARSE(stat);
    }
    cusparsestruct->handle = handle;
  }
  stat = hipsparseSetPointerMode(cusparsestruct->handle, HIPSPARSE_POINTER_MODE_DEVICE);CHKERRCUSPARSE(stat);
  PetscFunctionReturn(0);
}

PetscErrorCode MatCUSPARSEClearHandle(Mat A)
{
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  PetscBool          flg;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr = PetscObjectTypeCompare((PetscObject)A,MATSEQAIJCUSPARSE,&flg);CHKERRQ(ierr);
  if (!flg || !cusparsestruct) PetscFunctionReturn(0);
  if (cusparsestruct->handle) cusparsestruct->handle = 0;
  PetscFunctionReturn(0);
}

PetscErrorCode MatFactorGetSolverType_seqaij_cusparse(Mat A,MatSolverType *type)
{
  PetscFunctionBegin;
  *type = MATSOLVERCUSPARSE;
  PetscFunctionReturn(0);
}

/*MC
  MATSOLVERCUSPARSE = "cusparse" - A matrix type providing triangular solvers for seq matrices
  on a single GPU of type, seqaijcusparse, aijcusparse, or seqaijcusp, aijcusp. Currently supported
  algorithms are ILU(k) and ICC(k). Typically, deeper factorizations (larger k) results in poorer
  performance in the triangular solves. Full LU, and Cholesky decompositions can be solved through the
  CUSPARSE triangular solve algorithm. However, the performance can be quite poor and thus these
  algorithms are not recommended. This class does NOT support direct solver operations.

  Level: beginner

.seealso: PCFactorSetMatSolverType(), MatSolverType, MatCreateSeqAIJCUSPARSE(), MATAIJCUSPARSE, MatCreateAIJCUSPARSE(), MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M*/

PETSC_EXTERN PetscErrorCode MatGetFactor_seqaijcusparse_cusparse(Mat A,MatFactorType ftype,Mat *B)
{
  PetscErrorCode ierr;
  PetscInt       n = A->rmap->n;

  PetscFunctionBegin;
  ierr = MatCreate(PetscObjectComm((PetscObject)A),B);CHKERRQ(ierr);
  ierr = MatSetSizes(*B,n,n,n,n);CHKERRQ(ierr);
  (*B)->factortype = ftype;
  ierr = MatSetType(*B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);

  if (ftype == MAT_FACTOR_LU || ftype == MAT_FACTOR_ILU || ftype == MAT_FACTOR_ILUDT) {
    ierr = MatSetBlockSizesFromMats(*B,A,A);CHKERRQ(ierr);
    (*B)->ops->ilufactorsymbolic = MatILUFactorSymbolic_SeqAIJCUSPARSE;
    (*B)->ops->lufactorsymbolic  = MatLUFactorSymbolic_SeqAIJCUSPARSE;
    ierr = PetscStrallocpy(MATORDERINGND,(char**)&(*B)->preferredordering[MAT_FACTOR_LU]);CHKERRQ(ierr);
    ierr = PetscStrallocpy(MATORDERINGNATURAL,(char**)&(*B)->preferredordering[MAT_FACTOR_ILU]);CHKERRQ(ierr);
    ierr = PetscStrallocpy(MATORDERINGNATURAL,(char**)&(*B)->preferredordering[MAT_FACTOR_ILUDT]);CHKERRQ(ierr);
  } else if (ftype == MAT_FACTOR_CHOLESKY || ftype == MAT_FACTOR_ICC) {
    (*B)->ops->iccfactorsymbolic      = MatICCFactorSymbolic_SeqAIJCUSPARSE;
    (*B)->ops->choleskyfactorsymbolic = MatCholeskyFactorSymbolic_SeqAIJCUSPARSE;
    ierr = PetscStrallocpy(MATORDERINGND,(char**)&(*B)->preferredordering[MAT_FACTOR_CHOLESKY]);CHKERRQ(ierr);
    ierr = PetscStrallocpy(MATORDERINGNATURAL,(char**)&(*B)->preferredordering[MAT_FACTOR_ICC]);CHKERRQ(ierr);
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Factor type not supported for CUSPARSE Matrix Types");

  ierr = MatSeqAIJSetPreallocation(*B,MAT_SKIP_ALLOCATION,NULL);CHKERRQ(ierr);
  (*B)->canuseordering = PETSC_TRUE;
  ierr = PetscObjectComposeFunction((PetscObject)(*B),"MatFactorGetSolverType_C",MatFactorGetSolverType_seqaij_cusparse);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatCUSPARSESetFormat_SeqAIJCUSPARSE(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  switch (op) {
  case MAT_CUSPARSE_MULT:
    cusparsestruct->format = format;
    break;
  case MAT_CUSPARSE_ALL:
    cusparsestruct->format = format;
    break;
  default:
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"unsupported operation %d for MatCUSPARSEFormatOperation. MAT_CUSPARSE_MULT and MAT_CUSPARSE_ALL are currently supported.",op);
  }
  PetscFunctionReturn(0);
}

/*@
   MatCUSPARSESetFormat - Sets the storage format of CUSPARSE matrices for a particular
   operation. Only the MatMult operation can use different GPU storage formats
   for MPIAIJCUSPARSE matrices.
   Not Collective

   Input Parameters:
+  A - Matrix of type SEQAIJCUSPARSE
.  op - MatCUSPARSEFormatOperation. SEQAIJCUSPARSE matrices support MAT_CUSPARSE_MULT and MAT_CUSPARSE_ALL. MPIAIJCUSPARSE matrices support MAT_CUSPARSE_MULT_DIAG, MAT_CUSPARSE_MULT_OFFDIAG, and MAT_CUSPARSE_ALL.
-  format - MatCUSPARSEStorageFormat (one of MAT_CUSPARSE_CSR, MAT_CUSPARSE_ELL, MAT_CUSPARSE_HYB. The latter two require CUDA 4.2)

   Output Parameter:

   Level: intermediate

.seealso: MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
@*/
PetscErrorCode MatCUSPARSESetFormat(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A, MAT_CLASSID,1);
  ierr = PetscTryMethod(A,"MatCUSPARSESetFormat_C",(Mat,MatCUSPARSEFormatOperation,MatCUSPARSEStorageFormat),(A,op,format));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatSetOption_SeqAIJCUSPARSE(Mat A,MatOption op,PetscBool flg)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  switch (op) {
    case MAT_FORM_EXPLICIT_TRANSPOSE:
      /* need to destroy the transpose matrix if present to prevent from logic errors if flg is set to true later */
      if (A->form_explicit_transpose && !flg) {ierr = MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_TRUE);CHKERRQ(ierr);}
      A->form_explicit_transpose = flg;
      break;
    default:
      ierr = MatSetOption_SeqAIJ(A,op,flg);CHKERRQ(ierr);
      break;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU(Mat A);

static PetscErrorCode MatLUFactorNumeric_SeqAIJCUSPARSE(Mat B,Mat A,const MatFactorInfo *info)
{
  Mat_SeqAIJ     *b = (Mat_SeqAIJ*)B->data;
  IS             isrow = b->row,iscol = b->col;
  PetscBool      row_identity,col_identity;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSECopyFromGPU(A);CHKERRQ(ierr);
  ierr = MatLUFactorNumeric_SeqAIJ(B,A,info);CHKERRQ(ierr);
  B->offloadmask = PETSC_OFFLOAD_CPU;
  /* determine which version of MatSolve needs to be used. */
  ierr = ISIdentity(isrow,&row_identity);CHKERRQ(ierr);
  ierr = ISIdentity(iscol,&col_identity);CHKERRQ(ierr);
  if (row_identity && col_identity) {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE_NaturalOrdering;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering;
    B->ops->matsolve = NULL;
    B->ops->matsolvetranspose = NULL;
  } else {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE;
    B->ops->matsolve = NULL;
    B->ops->matsolvetranspose = NULL;
  }

  /* get the triangular factors */
  ierr = MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU(B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSetFromOptions_SeqAIJCUSPARSE(PetscOptionItems *PetscOptionsObject,Mat A)
{
  PetscErrorCode           ierr;
  MatCUSPARSEStorageFormat format;
  PetscBool                flg;
  Mat_SeqAIJCUSPARSE       *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  ierr = PetscOptionsHead(PetscOptionsObject,"SeqAIJCUSPARSE options");CHKERRQ(ierr);
  if (A->factortype == MAT_FACTOR_NONE) {
    ierr = PetscOptionsEnum("-mat_cusparse_mult_storage_format","sets storage format of (seq)aijcusparse gpu matrices for SpMV",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparsestruct->format,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_MULT,format);CHKERRQ(ierr);}

    ierr = PetscOptionsEnum("-mat_cusparse_storage_format","sets storage format of (seq)aijcusparse gpu matrices for SpMV and TriSolve",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparsestruct->format,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_ALL,format);CHKERRQ(ierr);}
   #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    ierr = PetscOptionsEnum("-mat_cusparse_spmv_alg","sets cuSPARSE algorithm used in sparse-mat dense-vector multiplication (SpMV)",
                            "hipsparseSpMVAlg_t",MatCUSPARSESpMVAlgorithms,(PetscEnum)cusparsestruct->spmvAlg,(PetscEnum*)&cusparsestruct->spmvAlg,&flg);CHKERRQ(ierr);
    /* If user did use this option, check its consistency with cuSPARSE, since PetscOptionsEnum() sets enum values based on their position in MatCUSPARSESpMVAlgorithms[] */
    if (flg && HIPSPARSE_CSRMV_ALG1 != 2) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"cuSPARSE enum hipsparseSpMVAlg_t has been changed but PETSc has not been updated accordingly");

    ierr = PetscOptionsEnum("-mat_cusparse_spmm_alg","sets cuSPARSE algorithm used in sparse-mat dense-mat multiplication (SpMM)",
                            "hipsparseSpMMAlg_t",MatCUSPARSESpMMAlgorithms,(PetscEnum)cusparsestruct->spmmAlg,(PetscEnum*)&cusparsestruct->spmmAlg,&flg);CHKERRQ(ierr);
    if (flg && HIPSPARSE_SPMM_CSR_ALG1 != 4) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"cuSPARSE enum hipsparseSpMMAlg_t has been changed but PETSc has not been updated accordingly");

    ierr = PetscOptionsEnum("-mat_cusparse_csr2csc_alg","sets cuSPARSE algorithm used in converting CSR matrices to CSC matrices",
                            "hipsparseCsr2CscAlg_t",MatCUSPARSECsr2CscAlgorithms,(PetscEnum)cusparsestruct->csr2cscAlg,(PetscEnum*)&cusparsestruct->csr2cscAlg,&flg);CHKERRQ(ierr);
    if (flg && HIPSPARSE_CSR2CSC_ALG1 != 1) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"cuSPARSE enum hipsparseCsr2CscAlg_t has been changed but PETSc has not been updated accordingly");
   #endif
  }
  ierr = PetscOptionsTail();CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatILUFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)B->spptr;
  PetscErrorCode               ierr;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSETriFactors_Reset(&cusparseTriFactors);CHKERRQ(ierr);
  ierr = MatILUFactorSymbolic_SeqAIJ(B,A,isrow,iscol,info);CHKERRQ(ierr);
  B->ops->lufactornumeric = MatLUFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatLUFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)B->spptr;
  PetscErrorCode               ierr;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSETriFactors_Reset(&cusparseTriFactors);CHKERRQ(ierr);
  ierr = MatLUFactorSymbolic_SeqAIJ(B,A,isrow,iscol,info);CHKERRQ(ierr);
  B->ops->lufactornumeric = MatLUFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatICCFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS perm,const MatFactorInfo *info)
{
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)B->spptr;
  PetscErrorCode               ierr;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSETriFactors_Reset(&cusparseTriFactors);CHKERRQ(ierr);
  ierr = MatICCFactorSymbolic_SeqAIJ(B,A,perm,info);CHKERRQ(ierr);
  B->ops->choleskyfactornumeric = MatCholeskyFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCholeskyFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS perm,const MatFactorInfo *info)
{
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)B->spptr;
  PetscErrorCode               ierr;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSETriFactors_Reset(&cusparseTriFactors);CHKERRQ(ierr);
  ierr = MatCholeskyFactorSymbolic_SeqAIJ(B,A,perm,info);CHKERRQ(ierr);
  B->ops->choleskyfactornumeric = MatCholeskyFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEBuildILULowerTriMatrix(Mat A)
{
  Mat_SeqAIJ                        *a = (Mat_SeqAIJ*)A->data;
  PetscInt                          n = A->rmap->n;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  hipsparseStatus_t                  stat;
  const PetscInt                    *ai = a->i,*aj = a->j,*vi;
  const MatScalar                   *aa = a->a,*v;
  PetscInt                          *AiLo, *AjLo;
  PetscInt                          i,nz, nzLower, offset, rowOffset;
  PetscErrorCode                    ierr;
  hipError_t                       cerr;

  PetscFunctionBegin;
  if (!n) PetscFunctionReturn(0);
  if (A->offloadmask == PETSC_OFFLOAD_UNALLOCATED || A->offloadmask == PETSC_OFFLOAD_CPU) {
    try {
      /* first figure out the number of nonzeros in the lower triangular matrix including 1's on the diagonal. */
      nzLower=n+ai[n]-ai[1];
      if (!loTriFactor) {
        PetscScalar                       *AALo;

        cerr = hipHostMalloc((void**) &AALo, nzLower*sizeof(PetscScalar));CHKERRCUDA(cerr);

        /* Allocate Space for the lower triangular matrix */
        cerr = hipHostMalloc((void**) &AiLo, (n+1)*sizeof(PetscInt));CHKERRCUDA(cerr);
        cerr = hipHostMalloc((void**) &AjLo, nzLower*sizeof(PetscInt));CHKERRCUDA(cerr);

        /* Fill the lower triangular matrix */
        AiLo[0]  = (PetscInt) 0;
        AiLo[n]  = nzLower;
        AjLo[0]  = (PetscInt) 0;
        AALo[0]  = (MatScalar) 1.0;
        v        = aa;
        vi       = aj;
        offset   = 1;
        rowOffset= 1;
        for (i=1; i<n; i++) {
          nz = ai[i+1] - ai[i];
          /* additional 1 for the term on the diagonal */
          AiLo[i]    = rowOffset;
          rowOffset += nz+1;

          ierr = PetscArraycpy(&(AjLo[offset]), vi, nz);CHKERRQ(ierr);
          ierr = PetscArraycpy(&(AALo[offset]), v, nz);CHKERRQ(ierr);

          offset      += nz;
          AjLo[offset] = (PetscInt) i;
          AALo[offset] = (MatScalar) 1.0;
          offset      += 1;

          v  += nz;
          vi += nz;
        }

        /* allocate space for the triangular factor information */
        ierr = PetscNew(&loTriFactor);CHKERRQ(ierr);
        loTriFactor->solvePolicy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        /* Create the matrix description */
        stat = hipsparseCreateMatDescr(&loTriFactor->descr);CHKERRCUSPARSE(stat);
        stat = hipsparseSetMatIndexBase(loTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
       #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        stat = hipsparseSetMatType(loTriFactor->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUSPARSE(stat);
       #else
        stat = hipsparseSetMatType(loTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);CHKERRCUSPARSE(stat);
       #endif
        stat = hipsparseSetMatFillMode(loTriFactor->descr, HIPSPARSE_FILL_MODE_LOWER);CHKERRCUSPARSE(stat);
        stat = hipsparseSetMatDiagType(loTriFactor->descr, HIPSPARSE_DIAG_TYPE_UNIT);CHKERRCUSPARSE(stat);

        /* set the operation */
        loTriFactor->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

        /* set the matrix */
        loTriFactor->csrMat = new CsrMatrix;
        loTriFactor->csrMat->num_rows = n;
        loTriFactor->csrMat->num_cols = n;
        loTriFactor->csrMat->num_entries = nzLower;

        loTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(n+1);
        loTriFactor->csrMat->row_offsets->assign(AiLo, AiLo+n+1);

        loTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(nzLower);
        loTriFactor->csrMat->column_indices->assign(AjLo, AjLo+nzLower);

        loTriFactor->csrMat->values = new THRUSTARRAY(nzLower);
        loTriFactor->csrMat->values->assign(AALo, AALo+nzLower);

        /* Create the solve analysis information */
        ierr = PetscLogEventBegin(MAT_CUSPARSESolveAnalysis,A,0,0,0);CHKERRQ(ierr);
        stat = cusparse_create_analysis_info(&loTriFactor->solveInfo);CHKERRCUSPARSE(stat);
      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        stat = cusparse_get_svbuffsize(cusparseTriFactors->handle, loTriFactor->solveOp,
                                       loTriFactor->csrMat->num_rows, loTriFactor->csrMat->num_entries, loTriFactor->descr,
                                       loTriFactor->csrMat->values->data().get(), loTriFactor->csrMat->row_offsets->data().get(),
                                       loTriFactor->csrMat->column_indices->data().get(), loTriFactor->solveInfo,
                                       &loTriFactor->solveBufferSize);CHKERRCUSPARSE(stat);
        cerr = hipMalloc(&loTriFactor->solveBuffer,loTriFactor->solveBufferSize);CHKERRCUDA(cerr);
      #endif

        /* perform the solve analysis */
        stat = cusparse_analysis(cusparseTriFactors->handle, loTriFactor->solveOp,
                                 loTriFactor->csrMat->num_rows, loTriFactor->csrMat->num_entries, loTriFactor->descr,
                                 loTriFactor->csrMat->values->data().get(), loTriFactor->csrMat->row_offsets->data().get(),
                                 loTriFactor->csrMat->column_indices->data().get(), loTriFactor->solveInfo
                               #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                                 ,loTriFactor->solvePolicy, loTriFactor->solveBuffer
                               #endif
);CHKERRCUSPARSE(stat);
        cerr = WaitForCUDA();CHKERRCUDA(cerr);
        ierr = PetscLogEventEnd(MAT_CUSPARSESolveAnalysis,A,0,0,0);CHKERRQ(ierr);

        /* assign the pointer */
        ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtr = loTriFactor;
        loTriFactor->AA_h = AALo;
        cerr = hipHostFree(AiLo);CHKERRCUDA(cerr);
        cerr = hipHostFree(AjLo);CHKERRCUDA(cerr);
        ierr = PetscLogCpuToGpu((n+1+nzLower)*sizeof(int)+nzLower*sizeof(PetscScalar));CHKERRQ(ierr);
      } else { /* update values only */
        if (!loTriFactor->AA_h) {
          cerr = hipHostMalloc((void**) &loTriFactor->AA_h, nzLower*sizeof(PetscScalar));CHKERRCUDA(cerr);
        }
        /* Fill the lower triangular matrix */
        loTriFactor->AA_h[0]  = 1.0;
        v        = aa;
        vi       = aj;
        offset   = 1;
        for (i=1; i<n; i++) {
          nz = ai[i+1] - ai[i];
          ierr = PetscArraycpy(&(loTriFactor->AA_h[offset]), v, nz);CHKERRQ(ierr);
          offset      += nz;
          loTriFactor->AA_h[offset] = 1.0;
          offset      += 1;
          v  += nz;
        }
        loTriFactor->csrMat->values->assign(loTriFactor->AA_h, loTriFactor->AA_h+nzLower);
        ierr = PetscLogCpuToGpu(nzLower*sizeof(PetscScalar));CHKERRQ(ierr);
      }
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEBuildILUUpperTriMatrix(Mat A)
{
  Mat_SeqAIJ                        *a = (Mat_SeqAIJ*)A->data;
  PetscInt                          n = A->rmap->n;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  hipsparseStatus_t                  stat;
  const PetscInt                    *aj = a->j,*adiag = a->diag,*vi;
  const MatScalar                   *aa = a->a,*v;
  PetscInt                          *AiUp, *AjUp;
  PetscInt                          i,nz, nzUpper, offset;
  PetscErrorCode                    ierr;
  hipError_t                       cerr;

  PetscFunctionBegin;
  if (!n) PetscFunctionReturn(0);
  if (A->offloadmask == PETSC_OFFLOAD_UNALLOCATED || A->offloadmask == PETSC_OFFLOAD_CPU) {
    try {
      /* next, figure out the number of nonzeros in the upper triangular matrix. */
      nzUpper = adiag[0]-adiag[n];
      if (!upTriFactor) {
        PetscScalar *AAUp;

        cerr = hipHostMalloc((void**) &AAUp, nzUpper*sizeof(PetscScalar));CHKERRCUDA(cerr);

        /* Allocate Space for the upper triangular matrix */
        cerr = hipHostMalloc((void**) &AiUp, (n+1)*sizeof(PetscInt));CHKERRCUDA(cerr);
        cerr = hipHostMalloc((void**) &AjUp, nzUpper*sizeof(PetscInt));CHKERRCUDA(cerr);

        /* Fill the upper triangular matrix */
        AiUp[0]=(PetscInt) 0;
        AiUp[n]=nzUpper;
        offset = nzUpper;
        for (i=n-1; i>=0; i--) {
          v  = aa + adiag[i+1] + 1;
          vi = aj + adiag[i+1] + 1;

          /* number of elements NOT on the diagonal */
          nz = adiag[i] - adiag[i+1]-1;

          /* decrement the offset */
          offset -= (nz+1);

          /* first, set the diagonal elements */
          AjUp[offset] = (PetscInt) i;
          AAUp[offset] = (MatScalar)1./v[nz];
          AiUp[i]      = AiUp[i+1] - (nz+1);

          ierr = PetscArraycpy(&(AjUp[offset+1]), vi, nz);CHKERRQ(ierr);
          ierr = PetscArraycpy(&(AAUp[offset+1]), v, nz);CHKERRQ(ierr);
        }

        /* allocate space for the triangular factor information */
        ierr = PetscNew(&upTriFactor);CHKERRQ(ierr);
        upTriFactor->solvePolicy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

        /* Create the matrix description */
        stat = hipsparseCreateMatDescr(&upTriFactor->descr);CHKERRCUSPARSE(stat);
        stat = hipsparseSetMatIndexBase(upTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
       #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        stat = hipsparseSetMatType(upTriFactor->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUSPARSE(stat);
       #else
        stat = hipsparseSetMatType(upTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);CHKERRCUSPARSE(stat);
       #endif
        stat = hipsparseSetMatFillMode(upTriFactor->descr, HIPSPARSE_FILL_MODE_UPPER);CHKERRCUSPARSE(stat);
        stat = hipsparseSetMatDiagType(upTriFactor->descr, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSPARSE(stat);

        /* set the operation */
        upTriFactor->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

        /* set the matrix */
        upTriFactor->csrMat = new CsrMatrix;
        upTriFactor->csrMat->num_rows = n;
        upTriFactor->csrMat->num_cols = n;
        upTriFactor->csrMat->num_entries = nzUpper;

        upTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(n+1);
        upTriFactor->csrMat->row_offsets->assign(AiUp, AiUp+n+1);

        upTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(nzUpper);
        upTriFactor->csrMat->column_indices->assign(AjUp, AjUp+nzUpper);

        upTriFactor->csrMat->values = new THRUSTARRAY(nzUpper);
        upTriFactor->csrMat->values->assign(AAUp, AAUp+nzUpper);

        /* Create the solve analysis information */
        ierr = PetscLogEventBegin(MAT_CUSPARSESolveAnalysis,A,0,0,0);CHKERRQ(ierr);
        stat = cusparse_create_analysis_info(&upTriFactor->solveInfo);CHKERRCUSPARSE(stat);
      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        stat = cusparse_get_svbuffsize(cusparseTriFactors->handle, upTriFactor->solveOp,
                                     upTriFactor->csrMat->num_rows, upTriFactor->csrMat->num_entries, upTriFactor->descr,
                                     upTriFactor->csrMat->values->data().get(), upTriFactor->csrMat->row_offsets->data().get(),
                                     upTriFactor->csrMat->column_indices->data().get(), upTriFactor->solveInfo,
                                     &upTriFactor->solveBufferSize);CHKERRCUSPARSE(stat);
        cerr = hipMalloc(&upTriFactor->solveBuffer,upTriFactor->solveBufferSize);CHKERRCUDA(cerr);
      #endif

        /* perform the solve analysis */
        stat = cusparse_analysis(cusparseTriFactors->handle, upTriFactor->solveOp,
                                 upTriFactor->csrMat->num_rows, upTriFactor->csrMat->num_entries, upTriFactor->descr,
                                 upTriFactor->csrMat->values->data().get(), upTriFactor->csrMat->row_offsets->data().get(),
                                 upTriFactor->csrMat->column_indices->data().get(), upTriFactor->solveInfo
                               #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                                 ,upTriFactor->solvePolicy, upTriFactor->solveBuffer
                               #endif
);CHKERRCUSPARSE(stat);
        cerr = WaitForCUDA();CHKERRCUDA(cerr);
        ierr = PetscLogEventEnd(MAT_CUSPARSESolveAnalysis,A,0,0,0);CHKERRQ(ierr);

        /* assign the pointer */
        ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtr = upTriFactor;
        upTriFactor->AA_h = AAUp;
        cerr = hipHostFree(AiUp);CHKERRCUDA(cerr);
        cerr = hipHostFree(AjUp);CHKERRCUDA(cerr);
        ierr = PetscLogCpuToGpu((n+1+nzUpper)*sizeof(int)+nzUpper*sizeof(PetscScalar));CHKERRQ(ierr);
      } else {
        if (!upTriFactor->AA_h) {
          cerr = hipHostMalloc((void**) &upTriFactor->AA_h, nzUpper*sizeof(PetscScalar));CHKERRCUDA(cerr);
        }
        /* Fill the upper triangular matrix */
        offset = nzUpper;
        for (i=n-1; i>=0; i--) {
          v  = aa + adiag[i+1] + 1;

          /* number of elements NOT on the diagonal */
          nz = adiag[i] - adiag[i+1]-1;

          /* decrement the offset */
          offset -= (nz+1);

          /* first, set the diagonal elements */
          upTriFactor->AA_h[offset] = 1./v[nz];
          ierr = PetscArraycpy(&(upTriFactor->AA_h[offset+1]), v, nz);CHKERRQ(ierr);
        }
        upTriFactor->csrMat->values->assign(upTriFactor->AA_h, upTriFactor->AA_h+nzUpper);
        ierr = PetscLogCpuToGpu(nzUpper*sizeof(PetscScalar));CHKERRQ(ierr);
      }
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU(Mat A)
{
  PetscErrorCode               ierr;
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  IS                           isrow = a->row,iscol = a->icol;
  PetscBool                    row_identity,col_identity;
  PetscInt                     n = A->rmap->n;

  PetscFunctionBegin;
  if (!cusparseTriFactors) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing cusparseTriFactors");
  ierr = MatSeqAIJCUSPARSEBuildILULowerTriMatrix(A);CHKERRQ(ierr);
  ierr = MatSeqAIJCUSPARSEBuildILUUpperTriMatrix(A);CHKERRQ(ierr);

  if (!cusparseTriFactors->workVector) { cusparseTriFactors->workVector = new THRUSTARRAY(n); }
  cusparseTriFactors->nnz=a->nz;

  A->offloadmask = PETSC_OFFLOAD_BOTH;
  /* lower triangular indices */
  ierr = ISIdentity(isrow,&row_identity);CHKERRQ(ierr);
  if (!row_identity && !cusparseTriFactors->rpermIndices) {
    const PetscInt *r;

    ierr = ISGetIndices(isrow,&r);CHKERRQ(ierr);
    cusparseTriFactors->rpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->rpermIndices->assign(r, r+n);
    ierr = ISRestoreIndices(isrow,&r);CHKERRQ(ierr);
    ierr = PetscLogCpuToGpu(n*sizeof(PetscInt));CHKERRQ(ierr);
  }

  /* upper triangular indices */
  ierr = ISIdentity(iscol,&col_identity);CHKERRQ(ierr);
  if (!col_identity && !cusparseTriFactors->cpermIndices) {
    const PetscInt *c;

    ierr = ISGetIndices(iscol,&c);CHKERRQ(ierr);
    cusparseTriFactors->cpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->cpermIndices->assign(c, c+n);
    ierr = ISRestoreIndices(iscol,&c);CHKERRQ(ierr);
    ierr = PetscLogCpuToGpu(n*sizeof(PetscInt));CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEBuildICCTriMatrices(Mat A)
{
  Mat_SeqAIJ                        *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  hipsparseStatus_t                  stat;
  PetscErrorCode                    ierr;
  hipError_t                       cerr;
  PetscInt                          *AiUp, *AjUp;
  PetscScalar                       *AAUp;
  PetscScalar                       *AALo;
  PetscInt                          nzUpper = a->nz,n = A->rmap->n,i,offset,nz,j;
  Mat_SeqSBAIJ                      *b = (Mat_SeqSBAIJ*)A->data;
  const PetscInt                    *ai = b->i,*aj = b->j,*vj;
  const MatScalar                   *aa = b->a,*v;

  PetscFunctionBegin;
  if (!n) PetscFunctionReturn(0);
  if (A->offloadmask == PETSC_OFFLOAD_UNALLOCATED || A->offloadmask == PETSC_OFFLOAD_CPU) {
    try {
      cerr = hipHostMalloc((void**) &AAUp, nzUpper*sizeof(PetscScalar));CHKERRCUDA(cerr);
      cerr = hipHostMalloc((void**) &AALo, nzUpper*sizeof(PetscScalar));CHKERRCUDA(cerr);
      if (!upTriFactor && !loTriFactor) {
        /* Allocate Space for the upper triangular matrix */
        cerr = hipHostMalloc((void**) &AiUp, (n+1)*sizeof(PetscInt));CHKERRCUDA(cerr);
        cerr = hipHostMalloc((void**) &AjUp, nzUpper*sizeof(PetscInt));CHKERRCUDA(cerr);

        /* Fill the upper triangular matrix */
        AiUp[0]=(PetscInt) 0;
        AiUp[n]=nzUpper;
        offset = 0;
        for (i=0; i<n; i++) {
          /* set the pointers */
          v  = aa + ai[i];
          vj = aj + ai[i];
          nz = ai[i+1] - ai[i] - 1; /* exclude diag[i] */

          /* first, set the diagonal elements */
          AjUp[offset] = (PetscInt) i;
          AAUp[offset] = (MatScalar)1.0/v[nz];
          AiUp[i]      = offset;
          AALo[offset] = (MatScalar)1.0/v[nz];

          offset+=1;
          if (nz>0) {
            ierr = PetscArraycpy(&(AjUp[offset]), vj, nz);CHKERRQ(ierr);
            ierr = PetscArraycpy(&(AAUp[offset]), v, nz);CHKERRQ(ierr);
            for (j=offset; j<offset+nz; j++) {
              AAUp[j] = -AAUp[j];
              AALo[j] = AAUp[j]/v[nz];
            }
            offset+=nz;
          }
        }

        /* allocate space for the triangular factor information */
        ierr = PetscNew(&upTriFactor);CHKERRQ(ierr);
        upTriFactor->solvePolicy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

        /* Create the matrix description */
        stat = hipsparseCreateMatDescr(&upTriFactor->descr);CHKERRCUSPARSE(stat);
        stat = hipsparseSetMatIndexBase(upTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
       #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        stat = hipsparseSetMatType(upTriFactor->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUSPARSE(stat);
       #else
        stat = hipsparseSetMatType(upTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);CHKERRCUSPARSE(stat);
       #endif
        stat = hipsparseSetMatFillMode(upTriFactor->descr, HIPSPARSE_FILL_MODE_UPPER);CHKERRCUSPARSE(stat);
        stat = hipsparseSetMatDiagType(upTriFactor->descr, HIPSPARSE_DIAG_TYPE_UNIT);CHKERRCUSPARSE(stat);

        /* set the matrix */
        upTriFactor->csrMat = new CsrMatrix;
        upTriFactor->csrMat->num_rows = A->rmap->n;
        upTriFactor->csrMat->num_cols = A->cmap->n;
        upTriFactor->csrMat->num_entries = a->nz;

        upTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
        upTriFactor->csrMat->row_offsets->assign(AiUp, AiUp+A->rmap->n+1);

        upTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(a->nz);
        upTriFactor->csrMat->column_indices->assign(AjUp, AjUp+a->nz);

        upTriFactor->csrMat->values = new THRUSTARRAY(a->nz);
        upTriFactor->csrMat->values->assign(AAUp, AAUp+a->nz);

        /* set the operation */
        upTriFactor->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

        /* Create the solve analysis information */
        ierr = PetscLogEventBegin(MAT_CUSPARSESolveAnalysis,A,0,0,0);CHKERRQ(ierr);
        stat = cusparse_create_analysis_info(&upTriFactor->solveInfo);CHKERRCUSPARSE(stat);
      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        stat = cusparse_get_svbuffsize(cusparseTriFactors->handle, upTriFactor->solveOp,
                                       upTriFactor->csrMat->num_rows, upTriFactor->csrMat->num_entries, upTriFactor->descr,
                                       upTriFactor->csrMat->values->data().get(), upTriFactor->csrMat->row_offsets->data().get(),
                                       upTriFactor->csrMat->column_indices->data().get(), upTriFactor->solveInfo,
                                       &upTriFactor->solveBufferSize);CHKERRCUSPARSE(stat);
        cerr = hipMalloc(&upTriFactor->solveBuffer,upTriFactor->solveBufferSize);CHKERRCUDA(cerr);
      #endif

        /* perform the solve analysis */
        stat = cusparse_analysis(cusparseTriFactors->handle, upTriFactor->solveOp,
                                 upTriFactor->csrMat->num_rows, upTriFactor->csrMat->num_entries, upTriFactor->descr,
                                 upTriFactor->csrMat->values->data().get(), upTriFactor->csrMat->row_offsets->data().get(),
                                 upTriFactor->csrMat->column_indices->data().get(), upTriFactor->solveInfo
                                #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                                 ,upTriFactor->solvePolicy, upTriFactor->solveBuffer
                                #endif
);CHKERRCUSPARSE(stat);
        cerr = WaitForCUDA();CHKERRCUDA(cerr);
        ierr = PetscLogEventEnd(MAT_CUSPARSESolveAnalysis,A,0,0,0);CHKERRQ(ierr);

        /* assign the pointer */
        ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtr = upTriFactor;

        /* allocate space for the triangular factor information */
        ierr = PetscNew(&loTriFactor);CHKERRQ(ierr);
        loTriFactor->solvePolicy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

        /* Create the matrix description */
        stat = hipsparseCreateMatDescr(&loTriFactor->descr);CHKERRCUSPARSE(stat);
        stat = hipsparseSetMatIndexBase(loTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
       #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        stat = hipsparseSetMatType(loTriFactor->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUSPARSE(stat);
       #else
        stat = hipsparseSetMatType(loTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);CHKERRCUSPARSE(stat);
       #endif
        stat = hipsparseSetMatFillMode(loTriFactor->descr, HIPSPARSE_FILL_MODE_UPPER);CHKERRCUSPARSE(stat);
        stat = hipsparseSetMatDiagType(loTriFactor->descr, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSPARSE(stat);

        /* set the operation */
        loTriFactor->solveOp = HIPSPARSE_OPERATION_TRANSPOSE;

        /* set the matrix */
        loTriFactor->csrMat = new CsrMatrix;
        loTriFactor->csrMat->num_rows = A->rmap->n;
        loTriFactor->csrMat->num_cols = A->cmap->n;
        loTriFactor->csrMat->num_entries = a->nz;

        loTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
        loTriFactor->csrMat->row_offsets->assign(AiUp, AiUp+A->rmap->n+1);

        loTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(a->nz);
        loTriFactor->csrMat->column_indices->assign(AjUp, AjUp+a->nz);

        loTriFactor->csrMat->values = new THRUSTARRAY(a->nz);
        loTriFactor->csrMat->values->assign(AALo, AALo+a->nz);

        /* Create the solve analysis information */
        ierr = PetscLogEventBegin(MAT_CUSPARSESolveAnalysis,A,0,0,0);CHKERRQ(ierr);
        stat = cusparse_create_analysis_info(&loTriFactor->solveInfo);CHKERRCUSPARSE(stat);
      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        stat = cusparse_get_svbuffsize(cusparseTriFactors->handle, loTriFactor->solveOp,
                                       loTriFactor->csrMat->num_rows, loTriFactor->csrMat->num_entries, loTriFactor->descr,
                                       loTriFactor->csrMat->values->data().get(), loTriFactor->csrMat->row_offsets->data().get(),
                                       loTriFactor->csrMat->column_indices->data().get(), loTriFactor->solveInfo,
                                       &loTriFactor->solveBufferSize);CHKERRCUSPARSE(stat);
        cerr = hipMalloc(&loTriFactor->solveBuffer,loTriFactor->solveBufferSize);CHKERRCUDA(cerr);
      #endif

        /* perform the solve analysis */
        stat = cusparse_analysis(cusparseTriFactors->handle, loTriFactor->solveOp,
                                 loTriFactor->csrMat->num_rows, loTriFactor->csrMat->num_entries, loTriFactor->descr,
                                 loTriFactor->csrMat->values->data().get(), loTriFactor->csrMat->row_offsets->data().get(),
                                 loTriFactor->csrMat->column_indices->data().get(), loTriFactor->solveInfo
                                #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                                 ,loTriFactor->solvePolicy, loTriFactor->solveBuffer
                                #endif
);CHKERRCUSPARSE(stat);
        cerr = WaitForCUDA();CHKERRCUDA(cerr);
        ierr = PetscLogEventEnd(MAT_CUSPARSESolveAnalysis,A,0,0,0);CHKERRQ(ierr);

        /* assign the pointer */
        ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtr = loTriFactor;

        ierr = PetscLogCpuToGpu(2*(((A->rmap->n+1)+(a->nz))*sizeof(int)+(a->nz)*sizeof(PetscScalar)));CHKERRQ(ierr);
        cerr = hipHostFree(AiUp);CHKERRCUDA(cerr);
        cerr = hipHostFree(AjUp);CHKERRCUDA(cerr);
      } else {
        /* Fill the upper triangular matrix */
        offset = 0;
        for (i=0; i<n; i++) {
          /* set the pointers */
          v  = aa + ai[i];
          nz = ai[i+1] - ai[i] - 1; /* exclude diag[i] */

          /* first, set the diagonal elements */
          AAUp[offset] = 1.0/v[nz];
          AALo[offset] = 1.0/v[nz];

          offset+=1;
          if (nz>0) {
            ierr = PetscArraycpy(&(AAUp[offset]), v, nz);CHKERRQ(ierr);
            for (j=offset; j<offset+nz; j++) {
              AAUp[j] = -AAUp[j];
              AALo[j] = AAUp[j]/v[nz];
            }
            offset+=nz;
          }
        }
        if (!upTriFactor) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing cusparseTriFactors");
        if (!loTriFactor) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing cusparseTriFactors");
        upTriFactor->csrMat->values->assign(AAUp, AAUp+a->nz);
        loTriFactor->csrMat->values->assign(AALo, AALo+a->nz);
        ierr = PetscLogCpuToGpu(2*(a->nz)*sizeof(PetscScalar));CHKERRQ(ierr);
      }
      cerr = hipHostFree(AAUp);CHKERRCUDA(cerr);
      cerr = hipHostFree(AALo);CHKERRCUDA(cerr);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEICCAnalysisAndCopyToGPU(Mat A)
{
  PetscErrorCode               ierr;
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  IS                           ip = a->row;
  PetscBool                    perm_identity;
  PetscInt                     n = A->rmap->n;

  PetscFunctionBegin;
  if (!cusparseTriFactors) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing cusparseTriFactors");
  ierr = MatSeqAIJCUSPARSEBuildICCTriMatrices(A);CHKERRQ(ierr);
  if (!cusparseTriFactors->workVector) { cusparseTriFactors->workVector = new THRUSTARRAY(n); }
  cusparseTriFactors->nnz=(a->nz-n)*2 + n;

  A->offloadmask = PETSC_OFFLOAD_BOTH;

  /* lower triangular indices */
  ierr = ISIdentity(ip,&perm_identity);CHKERRQ(ierr);
  if (!perm_identity) {
    IS             iip;
    const PetscInt *irip,*rip;

    ierr = ISInvertPermutation(ip,PETSC_DECIDE,&iip);CHKERRQ(ierr);
    ierr = ISGetIndices(iip,&irip);CHKERRQ(ierr);
    ierr = ISGetIndices(ip,&rip);CHKERRQ(ierr);
    cusparseTriFactors->rpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->rpermIndices->assign(rip, rip+n);
    cusparseTriFactors->cpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->cpermIndices->assign(irip, irip+n);
    ierr = ISRestoreIndices(iip,&irip);CHKERRQ(ierr);
    ierr = ISDestroy(&iip);CHKERRQ(ierr);
    ierr = ISRestoreIndices(ip,&rip);CHKERRQ(ierr);
    ierr = PetscLogCpuToGpu(2.*n*sizeof(PetscInt));CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCholeskyFactorNumeric_SeqAIJCUSPARSE(Mat B,Mat A,const MatFactorInfo *info)
{
  Mat_SeqAIJ     *b = (Mat_SeqAIJ*)B->data;
  IS             ip = b->row;
  PetscBool      perm_identity;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSECopyFromGPU(A);CHKERRQ(ierr);
  ierr = MatCholeskyFactorNumeric_SeqAIJ(B,A,info);CHKERRQ(ierr);
  B->offloadmask = PETSC_OFFLOAD_CPU;
  /* determine which version of MatSolve needs to be used. */
  ierr = ISIdentity(ip,&perm_identity);CHKERRQ(ierr);
  if (perm_identity) {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE_NaturalOrdering;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering;
    B->ops->matsolve = NULL;
    B->ops->matsolvetranspose = NULL;
  } else {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE;
    B->ops->matsolve = NULL;
    B->ops->matsolvetranspose = NULL;
  }

  /* get the triangular factors */
  ierr = MatSeqAIJCUSPARSEICCAnalysisAndCopyToGPU(B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(Mat A)
{
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactorT;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactorT;
  hipsparseStatus_t                  stat;
  hipsparseIndexBase_t               indexBase;
  hipsparseMatrixType_t              matrixType;
  hipsparseFillMode_t                fillMode;
  hipsparseDiagType_t                diagType;
  hipError_t                       cerr;
  PetscErrorCode                    ierr;

  PetscFunctionBegin;
  /* allocate space for the transpose of the lower triangular factor */
  ierr = PetscNew(&loTriFactorT);CHKERRQ(ierr);
  loTriFactorT->solvePolicy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

  /* set the matrix descriptors of the lower triangular factor */
  matrixType = hipsparseGetMatType(loTriFactor->descr);
  indexBase = hipsparseGetMatIndexBase(loTriFactor->descr);
  fillMode = hipsparseGetMatFillMode(loTriFactor->descr)==HIPSPARSE_FILL_MODE_UPPER ?
    HIPSPARSE_FILL_MODE_LOWER : HIPSPARSE_FILL_MODE_UPPER;
  diagType = hipsparseGetMatDiagType(loTriFactor->descr);

  /* Create the matrix description */
  stat = hipsparseCreateMatDescr(&loTriFactorT->descr);CHKERRCUSPARSE(stat);
  stat = hipsparseSetMatIndexBase(loTriFactorT->descr, indexBase);CHKERRCUSPARSE(stat);
  stat = hipsparseSetMatType(loTriFactorT->descr, matrixType);CHKERRCUSPARSE(stat);
  stat = hipsparseSetMatFillMode(loTriFactorT->descr, fillMode);CHKERRCUSPARSE(stat);
  stat = hipsparseSetMatDiagType(loTriFactorT->descr, diagType);CHKERRCUSPARSE(stat);

  /* set the operation */
  loTriFactorT->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  /* allocate GPU space for the CSC of the lower triangular factor*/
  loTriFactorT->csrMat = new CsrMatrix;
  loTriFactorT->csrMat->num_rows       = loTriFactor->csrMat->num_cols;
  loTriFactorT->csrMat->num_cols       = loTriFactor->csrMat->num_rows;
  loTriFactorT->csrMat->num_entries    = loTriFactor->csrMat->num_entries;
  loTriFactorT->csrMat->row_offsets    = new THRUSTINTARRAY32(loTriFactorT->csrMat->num_rows+1);
  loTriFactorT->csrMat->column_indices = new THRUSTINTARRAY32(loTriFactorT->csrMat->num_entries);
  loTriFactorT->csrMat->values         = new THRUSTARRAY(loTriFactorT->csrMat->num_entries);

  /* compute the transpose of the lower triangular factor, i.e. the CSC */
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  stat = hipsparseCsr2cscEx2_bufferSize(cusparseTriFactors->handle, loTriFactor->csrMat->num_rows,
                                       loTriFactor->csrMat->num_cols, loTriFactor->csrMat->num_entries,
                                       loTriFactor->csrMat->values->data().get(),
                                       loTriFactor->csrMat->row_offsets->data().get(),
                                       loTriFactor->csrMat->column_indices->data().get(),
                                       loTriFactorT->csrMat->values->data().get(),
                                       loTriFactorT->csrMat->row_offsets->data().get(), loTriFactorT->csrMat->column_indices->data().get(), cusparse_scalartype,
                                       HIPSPARSE_ACTION_NUMERIC,indexBase,
                                       HIPSPARSE_CSR2CSC_ALG1, &loTriFactor->csr2cscBufferSize);CHKERRCUSPARSE(stat);
  cerr = hipMalloc(&loTriFactor->csr2cscBuffer,loTriFactor->csr2cscBufferSize);CHKERRCUDA(cerr);
#endif

  ierr = PetscLogEventBegin(MAT_CUSPARSEGenerateTranspose,A,0,0,0);CHKERRQ(ierr);
  stat = cusparse_csr2csc(cusparseTriFactors->handle, loTriFactor->csrMat->num_rows,
                          loTriFactor->csrMat->num_cols, loTriFactor->csrMat->num_entries,
                          loTriFactor->csrMat->values->data().get(),
                          loTriFactor->csrMat->row_offsets->data().get(),
                          loTriFactor->csrMat->column_indices->data().get(),
                          loTriFactorT->csrMat->values->data().get(),
                        #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
                          loTriFactorT->csrMat->row_offsets->data().get(), loTriFactorT->csrMat->column_indices->data().get(), cusparse_scalartype,
                          HIPSPARSE_ACTION_NUMERIC, indexBase,
                          HIPSPARSE_CSR2CSC_ALG1, loTriFactor->csr2cscBuffer
                        #else
                          loTriFactorT->csrMat->column_indices->data().get(), loTriFactorT->csrMat->row_offsets->data().get(),
                          HIPSPARSE_ACTION_NUMERIC, indexBase
                        #endif
);CHKERRCUSPARSE(stat);
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogEventBegin(MAT_CUSPARSEGenerateTranspose,A,0,0,0);CHKERRQ(ierr);

  /* Create the solve analysis information */
  ierr = PetscLogEventBegin(MAT_CUSPARSESolveAnalysis,A,0,0,0);CHKERRQ(ierr);
  stat = cusparse_create_analysis_info(&loTriFactorT->solveInfo);CHKERRCUSPARSE(stat);
#if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
  stat = cusparse_get_svbuffsize(cusparseTriFactors->handle, loTriFactorT->solveOp,
                                loTriFactorT->csrMat->num_rows, loTriFactorT->csrMat->num_entries, loTriFactorT->descr,
                                loTriFactorT->csrMat->values->data().get(), loTriFactorT->csrMat->row_offsets->data().get(),
                                loTriFactorT->csrMat->column_indices->data().get(), loTriFactorT->solveInfo,
                                &loTriFactorT->solveBufferSize);CHKERRCUSPARSE(stat);
  cerr = hipMalloc(&loTriFactorT->solveBuffer,loTriFactorT->solveBufferSize);CHKERRCUDA(cerr);
#endif

  /* perform the solve analysis */
  stat = cusparse_analysis(cusparseTriFactors->handle, loTriFactorT->solveOp,
                           loTriFactorT->csrMat->num_rows, loTriFactorT->csrMat->num_entries, loTriFactorT->descr,
                           loTriFactorT->csrMat->values->data().get(), loTriFactorT->csrMat->row_offsets->data().get(),
                           loTriFactorT->csrMat->column_indices->data().get(), loTriFactorT->solveInfo
                          #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                           ,loTriFactorT->solvePolicy, loTriFactorT->solveBuffer
                          #endif
);CHKERRCUSPARSE(stat);
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogEventEnd(MAT_CUSPARSESolveAnalysis,A,0,0,0);CHKERRQ(ierr);

  /* assign the pointer */
  ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtrTranspose = loTriFactorT;

  /*********************************************/
  /* Now the Transpose of the Upper Tri Factor */
  /*********************************************/

  /* allocate space for the transpose of the upper triangular factor */
  ierr = PetscNew(&upTriFactorT);CHKERRQ(ierr);
  upTriFactorT->solvePolicy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

  /* set the matrix descriptors of the upper triangular factor */
  matrixType = hipsparseGetMatType(upTriFactor->descr);
  indexBase = hipsparseGetMatIndexBase(upTriFactor->descr);
  fillMode = hipsparseGetMatFillMode(upTriFactor->descr)==HIPSPARSE_FILL_MODE_UPPER ?
    HIPSPARSE_FILL_MODE_LOWER : HIPSPARSE_FILL_MODE_UPPER;
  diagType = hipsparseGetMatDiagType(upTriFactor->descr);

  /* Create the matrix description */
  stat = hipsparseCreateMatDescr(&upTriFactorT->descr);CHKERRCUSPARSE(stat);
  stat = hipsparseSetMatIndexBase(upTriFactorT->descr, indexBase);CHKERRCUSPARSE(stat);
  stat = hipsparseSetMatType(upTriFactorT->descr, matrixType);CHKERRCUSPARSE(stat);
  stat = hipsparseSetMatFillMode(upTriFactorT->descr, fillMode);CHKERRCUSPARSE(stat);
  stat = hipsparseSetMatDiagType(upTriFactorT->descr, diagType);CHKERRCUSPARSE(stat);

  /* set the operation */
  upTriFactorT->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  /* allocate GPU space for the CSC of the upper triangular factor*/
  upTriFactorT->csrMat = new CsrMatrix;
  upTriFactorT->csrMat->num_rows       = upTriFactor->csrMat->num_cols;
  upTriFactorT->csrMat->num_cols       = upTriFactor->csrMat->num_rows;
  upTriFactorT->csrMat->num_entries    = upTriFactor->csrMat->num_entries;
  upTriFactorT->csrMat->row_offsets    = new THRUSTINTARRAY32(upTriFactorT->csrMat->num_rows+1);
  upTriFactorT->csrMat->column_indices = new THRUSTINTARRAY32(upTriFactorT->csrMat->num_entries);
  upTriFactorT->csrMat->values         = new THRUSTARRAY(upTriFactorT->csrMat->num_entries);

  /* compute the transpose of the upper triangular factor, i.e. the CSC */
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  stat = hipsparseCsr2cscEx2_bufferSize(cusparseTriFactors->handle,upTriFactor->csrMat->num_rows,
                                upTriFactor->csrMat->num_cols, upTriFactor->csrMat->num_entries,
                                upTriFactor->csrMat->values->data().get(),
                                upTriFactor->csrMat->row_offsets->data().get(),
                                upTriFactor->csrMat->column_indices->data().get(),
                                upTriFactorT->csrMat->values->data().get(),
                                upTriFactorT->csrMat->row_offsets->data().get(), upTriFactorT->csrMat->column_indices->data().get(), cusparse_scalartype,
                                HIPSPARSE_ACTION_NUMERIC,indexBase,
                                HIPSPARSE_CSR2CSC_ALG1, &upTriFactor->csr2cscBufferSize);CHKERRCUSPARSE(stat);
  cerr = hipMalloc(&upTriFactor->csr2cscBuffer,upTriFactor->csr2cscBufferSize);CHKERRCUDA(cerr);
#endif

  ierr = PetscLogEventBegin(MAT_CUSPARSEGenerateTranspose,A,0,0,0);CHKERRQ(ierr);
  stat = cusparse_csr2csc(cusparseTriFactors->handle, upTriFactor->csrMat->num_rows,
                          upTriFactor->csrMat->num_cols, upTriFactor->csrMat->num_entries,
                          upTriFactor->csrMat->values->data().get(),
                          upTriFactor->csrMat->row_offsets->data().get(),
                          upTriFactor->csrMat->column_indices->data().get(),
                          upTriFactorT->csrMat->values->data().get(),
                        #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
                          upTriFactorT->csrMat->row_offsets->data().get(), upTriFactorT->csrMat->column_indices->data().get(), cusparse_scalartype,
                          HIPSPARSE_ACTION_NUMERIC, indexBase,
                          HIPSPARSE_CSR2CSC_ALG1, upTriFactor->csr2cscBuffer
                        #else
                          upTriFactorT->csrMat->column_indices->data().get(), upTriFactorT->csrMat->row_offsets->data().get(),
                          HIPSPARSE_ACTION_NUMERIC, indexBase
                        #endif
);CHKERRCUSPARSE(stat);
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogEventBegin(MAT_CUSPARSEGenerateTranspose,A,0,0,0);CHKERRQ(ierr);

  /* Create the solve analysis information */
  ierr = PetscLogEventBegin(MAT_CUSPARSESolveAnalysis,A,0,0,0);CHKERRQ(ierr);
  stat = cusparse_create_analysis_info(&upTriFactorT->solveInfo);CHKERRCUSPARSE(stat);
  #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
  stat = cusparse_get_svbuffsize(cusparseTriFactors->handle, upTriFactorT->solveOp,
                                 upTriFactorT->csrMat->num_rows, upTriFactorT->csrMat->num_entries, upTriFactorT->descr,
                                 upTriFactorT->csrMat->values->data().get(), upTriFactorT->csrMat->row_offsets->data().get(),
                                 upTriFactorT->csrMat->column_indices->data().get(), upTriFactorT->solveInfo,
                                 &upTriFactorT->solveBufferSize);CHKERRCUSPARSE(stat);
  cerr = hipMalloc(&upTriFactorT->solveBuffer,upTriFactorT->solveBufferSize);CHKERRCUDA(cerr);
  #endif

  /* perform the solve analysis */
  stat = cusparse_analysis(cusparseTriFactors->handle, upTriFactorT->solveOp,
                           upTriFactorT->csrMat->num_rows, upTriFactorT->csrMat->num_entries, upTriFactorT->descr,
                           upTriFactorT->csrMat->values->data().get(), upTriFactorT->csrMat->row_offsets->data().get(),
                           upTriFactorT->csrMat->column_indices->data().get(), upTriFactorT->solveInfo
                          #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                           ,upTriFactorT->solvePolicy, upTriFactorT->solveBuffer
                          #endif
);CHKERRCUSPARSE(stat);
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogEventEnd(MAT_CUSPARSESolveAnalysis,A,0,0,0);CHKERRQ(ierr);

  /* assign the pointer */
  ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtrTranspose = upTriFactorT;
  PetscFunctionReturn(0);
}

struct PetscScalarToPetscInt
{
  __host__ __device__
  PetscInt operator()(PetscScalar s)
  {
    return (PetscInt)PetscRealPart(s);
  }
};

static PetscErrorCode MatSeqAIJCUSPARSEFormExplicitTransposeForMult(Mat A)
{
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct, *matstructT;
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  hipsparseStatus_t             stat;
  hipsparseIndexBase_t          indexBase;
  hipError_t                  err;
  PetscErrorCode               ierr;

  PetscFunctionBegin;
  if (!A->form_explicit_transpose || !A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
  if (!matstruct) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing mat struct");
  matstructT = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
  if (A->transupdated && !matstructT) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing matTranspose struct");
  if (A->transupdated) PetscFunctionReturn(0);
  ierr = PetscLogEventBegin(MAT_CUSPARSEGenerateTranspose,A,0,0,0);CHKERRQ(ierr);
  if (cusparsestruct->format != MAT_CUSPARSE_CSR) {
    ierr = MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_TRUE);CHKERRQ(ierr);
  }
  if (!cusparsestruct->matTranspose) { /* create cusparse matrix */
    matstructT = new Mat_SeqAIJCUSPARSEMultStruct;
    stat = hipsparseCreateMatDescr(&matstructT->descr);CHKERRCUSPARSE(stat);
    indexBase = hipsparseGetMatIndexBase(matstruct->descr);
    stat = hipsparseSetMatIndexBase(matstructT->descr, indexBase);CHKERRCUSPARSE(stat);
    stat = hipsparseSetMatType(matstructT->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUSPARSE(stat);

    /* set alpha and beta */
    err = hipMalloc((void **)&(matstructT->alpha_one),sizeof(PetscScalar));CHKERRCUDA(err);
    err = hipMalloc((void **)&(matstructT->beta_zero),sizeof(PetscScalar));CHKERRCUDA(err);
    err = hipMalloc((void **)&(matstructT->beta_one), sizeof(PetscScalar));CHKERRCUDA(err);
    err = hipMemcpy(matstructT->alpha_one,&PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
    err = hipMemcpy(matstructT->beta_zero,&PETSC_CUSPARSE_ZERO,sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
    err = hipMemcpy(matstructT->beta_one, &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);

    if (cusparsestruct->format == MAT_CUSPARSE_CSR) {
      CsrMatrix *matrixT = new CsrMatrix;
      matstructT->mat = matrixT;
      matrixT->num_rows = A->cmap->n;
      matrixT->num_cols = A->rmap->n;
      matrixT->num_entries = a->nz;
      matrixT->row_offsets = new THRUSTINTARRAY32(matrixT->num_rows+1);
      matrixT->column_indices = new THRUSTINTARRAY32(a->nz);
      matrixT->values = new THRUSTARRAY(a->nz);

      if (!cusparsestruct->rowoffsets_gpu) { cusparsestruct->rowoffsets_gpu = new THRUSTINTARRAY32(A->rmap->n+1); }
      cusparsestruct->rowoffsets_gpu->assign(a->i,a->i+A->rmap->n+1);

     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      stat = hipsparseCreateCsr(&matstructT->matDescr,
                               matrixT->num_rows, matrixT->num_cols, matrixT->num_entries,
                               matrixT->row_offsets->data().get(), matrixT->column_indices->data().get(),
                               matrixT->values->data().get(),
                               HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I, /* row offset, col idx type due to THRUSTINTARRAY32 */
                               indexBase,cusparse_scalartype);CHKERRCUSPARSE(stat);
     #endif
    } else if (cusparsestruct->format == MAT_CUSPARSE_ELL || cusparsestruct->format == MAT_CUSPARSE_HYB) {
   #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"MAT_CUSPARSE_ELL and MAT_CUSPARSE_HYB are not supported since CUDA-11.0");
   #else
      CsrMatrix *temp  = new CsrMatrix;
      CsrMatrix *tempT = new CsrMatrix;
      /* First convert HYB to CSR */
      temp->num_rows = A->rmap->n;
      temp->num_cols = A->cmap->n;
      temp->num_entries = a->nz;
      temp->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
      temp->column_indices = new THRUSTINTARRAY32(a->nz);
      temp->values = new THRUSTARRAY(a->nz);

      stat = cusparse_hyb2csr(cusparsestruct->handle,
                              matstruct->descr, (hipsparseHybMat_t)matstruct->mat,
                              temp->values->data().get(),
                              temp->row_offsets->data().get(),
                              temp->column_indices->data().get());CHKERRCUSPARSE(stat);

      /* Next, convert CSR to CSC (i.e. the matrix transpose) */
      tempT->num_rows = A->rmap->n;
      tempT->num_cols = A->cmap->n;
      tempT->num_entries = a->nz;
      tempT->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
      tempT->column_indices = new THRUSTINTARRAY32(a->nz);
      tempT->values = new THRUSTARRAY(a->nz);

      stat = cusparse_csr2csc(cusparsestruct->handle, temp->num_rows,
                              temp->num_cols, temp->num_entries,
                              temp->values->data().get(),
                              temp->row_offsets->data().get(),
                              temp->column_indices->data().get(),
                              tempT->values->data().get(),
                              tempT->column_indices->data().get(),
                              tempT->row_offsets->data().get(),
                              HIPSPARSE_ACTION_NUMERIC, indexBase);CHKERRCUSPARSE(stat);

      /* Last, convert CSC to HYB */
      hipsparseHybMat_t hybMat;
      stat = hipsparseCreateHybMat(&hybMat);CHKERRCUSPARSE(stat);
      hipsparseHybPartition_t partition = cusparsestruct->format==MAT_CUSPARSE_ELL ?
        HIPSPARSE_HYB_PARTITION_MAX : HIPSPARSE_HYB_PARTITION_AUTO;
      stat = cusparse_csr2hyb(cusparsestruct->handle, A->rmap->n, A->cmap->n,
                              matstructT->descr, tempT->values->data().get(),
                              tempT->row_offsets->data().get(),
                              tempT->column_indices->data().get(),
                              hybMat, 0, partition);CHKERRCUSPARSE(stat);

      /* assign the pointer */
      matstructT->mat = hybMat;
      A->transupdated = PETSC_TRUE;
      /* delete temporaries */
      if (tempT) {
        if (tempT->values) delete (THRUSTARRAY*) tempT->values;
        if (tempT->column_indices) delete (THRUSTINTARRAY32*) tempT->column_indices;
        if (tempT->row_offsets) delete (THRUSTINTARRAY32*) tempT->row_offsets;
        delete (CsrMatrix*) tempT;
      }
      if (temp) {
        if (temp->values) delete (THRUSTARRAY*) temp->values;
        if (temp->column_indices) delete (THRUSTINTARRAY32*) temp->column_indices;
        if (temp->row_offsets) delete (THRUSTINTARRAY32*) temp->row_offsets;
        delete (CsrMatrix*) temp;
      }
     #endif
    }
  }
  if (cusparsestruct->format == MAT_CUSPARSE_CSR) { /* transpose mat struct may be already present, update data */
    CsrMatrix *matrix  = (CsrMatrix*)matstruct->mat;
    CsrMatrix *matrixT = (CsrMatrix*)matstructT->mat;
    if (!matrix) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrix");
    if (!matrix->row_offsets) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrix rows");
    if (!matrix->column_indices) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrix cols");
    if (!matrix->values) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrix values");
    if (!matrixT) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrixT");
    if (!matrixT->row_offsets) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrixT rows");
    if (!matrixT->column_indices) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrixT cols");
    if (!matrixT->values) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrixT values");
    if (!cusparsestruct->rowoffsets_gpu) { /* this may be absent when we did not construct the transpose with csr2csc */
      cusparsestruct->rowoffsets_gpu  = new THRUSTINTARRAY32(A->rmap->n + 1);
      cusparsestruct->rowoffsets_gpu->assign(a->i,a->i + A->rmap->n + 1);
      ierr = PetscLogCpuToGpu((A->rmap->n + 1)*sizeof(PetscInt));CHKERRQ(ierr);
    }
    if (!cusparsestruct->csr2csc_i) {
      THRUSTARRAY csr2csc_a(matrix->num_entries);
      PetscStackCallThrust(thrust::sequence(thrust::device, csr2csc_a.begin(), csr2csc_a.end(), 0.0));

      indexBase = hipsparseGetMatIndexBase(matstruct->descr);
     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      void   *csr2cscBuffer;
      size_t csr2cscBufferSize;
      stat = hipsparseCsr2cscEx2_bufferSize(cusparsestruct->handle, A->rmap->n,
                                           A->cmap->n, matrix->num_entries,
                                           matrix->values->data().get(),
                                           cusparsestruct->rowoffsets_gpu->data().get(),
                                           matrix->column_indices->data().get(),
                                           matrixT->values->data().get(),
                                           matrixT->row_offsets->data().get(), matrixT->column_indices->data().get(), cusparse_scalartype,
                                           HIPSPARSE_ACTION_NUMERIC,indexBase,
                                           cusparsestruct->csr2cscAlg, &csr2cscBufferSize);CHKERRCUSPARSE(stat);
      err = hipMalloc(&csr2cscBuffer,csr2cscBufferSize);CHKERRCUDA(err);
     #endif

      if (matrix->num_entries) {
        /* When there are no nonzeros, this routine mistakenly returns HIPSPARSE_STATUS_INVALID_VALUE in
           mat_tests-ex62_15_mpiaijcusparse on ranks 0 and 2 with CUDA-11. But CUDA-10 is OK.
           I checked every parameters and they were just fine. I have no clue why cusparse complains.

           Per https://docs.nvidia.com/cuda/cusparse/index.html#csr2cscEx2, when nnz = 0, matrixT->row_offsets[]
           should be filled with indexBase. So I just take a shortcut here.
        */
        stat = cusparse_csr2csc(cusparsestruct->handle, A->rmap->n,
                              A->cmap->n,matrix->num_entries,
                              csr2csc_a.data().get(),
                              cusparsestruct->rowoffsets_gpu->data().get(),
                              matrix->column_indices->data().get(),
                              matrixT->values->data().get(),
                             #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
                              matrixT->row_offsets->data().get(), matrixT->column_indices->data().get(), cusparse_scalartype,
                              HIPSPARSE_ACTION_NUMERIC,indexBase,
                              cusparsestruct->csr2cscAlg, csr2cscBuffer);CHKERRCUSPARSE(stat);
                             #else
                              matrixT->column_indices->data().get(), matrixT->row_offsets->data().get(),
                              HIPSPARSE_ACTION_NUMERIC, indexBase);CHKERRCUSPARSE(stat);
                             #endif
      } else {
        matrixT->row_offsets->assign(matrixT->row_offsets->size(),indexBase);
      }

      cusparsestruct->csr2csc_i = new THRUSTINTARRAY(matrix->num_entries);
      PetscStackCallThrust(thrust::transform(thrust::device,matrixT->values->begin(),matrixT->values->end(),cusparsestruct->csr2csc_i->begin(),PetscScalarToPetscInt()));
     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      err = hipFree(csr2cscBuffer);CHKERRCUDA(err);
     #endif
    }
    PetscStackCallThrust(thrust::copy(thrust::device,thrust::make_permutation_iterator(matrix->values->begin(), cusparsestruct->csr2csc_i->begin()),
                                                     thrust::make_permutation_iterator(matrix->values->begin(), cusparsestruct->csr2csc_i->end()),
                                                     matrixT->values->begin()));
  }
  ierr = PetscLogEventEnd(MAT_CUSPARSEGenerateTranspose,A,0,0,0);CHKERRQ(ierr);
  /* the compressed row indices is not used for matTranspose */
  matstructT->cprowIndices = NULL;
  /* assign the pointer */
  ((Mat_SeqAIJCUSPARSE*)A->spptr)->matTranspose = matstructT;
  A->transupdated = PETSC_TRUE;
  PetscFunctionReturn(0);
}

/* Why do we need to analyze the transposed matrix again? Can't we just use op(A) = HIPSPARSE_OPERATION_TRANSPOSE in MatSolve_SeqAIJCUSPARSE? */
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE(Mat A,Vec bb,Vec xx)
{
  PetscInt                              n = xx->map->n;
  const PetscScalar                     *barray;
  PetscScalar                           *xarray;
  thrust::device_ptr<const PetscScalar> bGPU;
  thrust::device_ptr<PetscScalar>       xGPU;
  hipsparseStatus_t                      stat;
  Mat_SeqAIJCUSPARSETriFactors          *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct     *loTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
  Mat_SeqAIJCUSPARSETriFactorStruct     *upTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  THRUSTARRAY                           *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;
  PetscErrorCode                        ierr;
  hipError_t                           cerr;

  PetscFunctionBegin;
  /* Analyze the matrix and create the transpose ... on the fly */
  if (!loTriFactorT && !upTriFactorT) {
    ierr = MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(A);CHKERRQ(ierr);
    loTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
    upTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  }

  /* Get the GPU pointers */
  ierr = VecCUDAGetArrayWrite(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(bb,&barray);CHKERRQ(ierr);
  xGPU = thrust::device_pointer_cast(xarray);
  bGPU = thrust::device_pointer_cast(barray);

  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  /* First, reorder with the row permutation */
  thrust::copy(thrust::cuda::par.on(PetscDefaultCudaStream),thrust::make_permutation_iterator(bGPU, cusparseTriFactors->rpermIndices->begin()),
               thrust::make_permutation_iterator(bGPU+n, cusparseTriFactors->rpermIndices->end()),
               xGPU);

  /* First, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactorT->solveOp,
                        upTriFactorT->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        upTriFactorT->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, upTriFactorT->descr,
                        upTriFactorT->csrMat->values->data().get(),
                        upTriFactorT->csrMat->row_offsets->data().get(),
                        upTriFactorT->csrMat->column_indices->data().get(),
                        upTriFactorT->solveInfo,
                        xarray, tempGPU->data().get()
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        ,upTriFactorT->solvePolicy, upTriFactorT->solveBuffer
                      #endif
);CHKERRCUSPARSE(stat);

  /* Then, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactorT->solveOp,
                        loTriFactorT->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        loTriFactorT->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, loTriFactorT->descr,
                        loTriFactorT->csrMat->values->data().get(),
                        loTriFactorT->csrMat->row_offsets->data().get(),
                        loTriFactorT->csrMat->column_indices->data().get(),
                        loTriFactorT->solveInfo,
                        tempGPU->data().get(), xarray
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        ,loTriFactorT->solvePolicy, loTriFactorT->solveBuffer
                      #endif
);CHKERRCUSPARSE(stat);

  /* Last, copy the solution, xGPU, into a temporary with the column permutation ... can't be done in place. */
  thrust::copy(thrust::cuda::par.on(PetscDefaultCudaStream),thrust::make_permutation_iterator(xGPU, cusparseTriFactors->cpermIndices->begin()),
               thrust::make_permutation_iterator(xGPU+n, cusparseTriFactors->cpermIndices->end()),
               tempGPU->begin());

  /* Copy the temporary to the full solution. */
  thrust::copy(thrust::cuda::par.on(PetscDefaultCudaStream),tempGPU->begin(), tempGPU->end(), xGPU);

  /* restore */
  ierr = VecCUDARestoreArrayRead(bb,&barray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(xx,&xarray);CHKERRQ(ierr);
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*cusparseTriFactors->nnz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering(Mat A,Vec bb,Vec xx)
{
  const PetscScalar                 *barray;
  PetscScalar                       *xarray;
  hipsparseStatus_t                  stat;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  THRUSTARRAY                       *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;
  PetscErrorCode                    ierr;
  hipError_t                       cerr;

  PetscFunctionBegin;
  /* Analyze the matrix and create the transpose ... on the fly */
  if (!loTriFactorT && !upTriFactorT) {
    ierr = MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(A);CHKERRQ(ierr);
    loTriFactorT       = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
    upTriFactorT       = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  }

  /* Get the GPU pointers */
  ierr = VecCUDAGetArrayWrite(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(bb,&barray);CHKERRQ(ierr);

  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  /* First, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactorT->solveOp,
                        upTriFactorT->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        upTriFactorT->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, upTriFactorT->descr,
                        upTriFactorT->csrMat->values->data().get(),
                        upTriFactorT->csrMat->row_offsets->data().get(),
                        upTriFactorT->csrMat->column_indices->data().get(),
                        upTriFactorT->solveInfo,
                        barray, tempGPU->data().get()
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        ,upTriFactorT->solvePolicy, upTriFactorT->solveBuffer
                      #endif
);CHKERRCUSPARSE(stat);

  /* Then, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactorT->solveOp,
                        loTriFactorT->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        loTriFactorT->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, loTriFactorT->descr,
                        loTriFactorT->csrMat->values->data().get(),
                        loTriFactorT->csrMat->row_offsets->data().get(),
                        loTriFactorT->csrMat->column_indices->data().get(),
                        loTriFactorT->solveInfo,
                        tempGPU->data().get(), xarray
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        ,loTriFactorT->solvePolicy, loTriFactorT->solveBuffer
                      #endif
);CHKERRCUSPARSE(stat);

  /* restore */
  ierr = VecCUDARestoreArrayRead(bb,&barray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(xx,&xarray);CHKERRQ(ierr);
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*cusparseTriFactors->nnz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqAIJCUSPARSE(Mat A,Vec bb,Vec xx)
{
  const PetscScalar                     *barray;
  PetscScalar                           *xarray;
  thrust::device_ptr<const PetscScalar> bGPU;
  thrust::device_ptr<PetscScalar>       xGPU;
  hipsparseStatus_t                      stat;
  Mat_SeqAIJCUSPARSETriFactors          *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct     *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct     *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  THRUSTARRAY                           *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;
  PetscErrorCode                        ierr;
  hipError_t                           cerr;

  PetscFunctionBegin;

  /* Get the GPU pointers */
  ierr = VecCUDAGetArrayWrite(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(bb,&barray);CHKERRQ(ierr);
  xGPU = thrust::device_pointer_cast(xarray);
  bGPU = thrust::device_pointer_cast(barray);

  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  /* First, reorder with the row permutation */
  thrust::copy(thrust::cuda::par.on(PetscDefaultCudaStream),thrust::make_permutation_iterator(bGPU, cusparseTriFactors->rpermIndices->begin()),
               thrust::make_permutation_iterator(bGPU, cusparseTriFactors->rpermIndices->end()),
               tempGPU->begin());

  /* Next, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactor->solveOp,
                        loTriFactor->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        loTriFactor->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, loTriFactor->descr,
                        loTriFactor->csrMat->values->data().get(),
                        loTriFactor->csrMat->row_offsets->data().get(),
                        loTriFactor->csrMat->column_indices->data().get(),
                        loTriFactor->solveInfo,
                        tempGPU->data().get(), xarray
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        ,loTriFactor->solvePolicy, loTriFactor->solveBuffer
                      #endif
);CHKERRCUSPARSE(stat);

  /* Then, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactor->solveOp,
                        upTriFactor->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        upTriFactor->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, upTriFactor->descr,
                        upTriFactor->csrMat->values->data().get(),
                        upTriFactor->csrMat->row_offsets->data().get(),
                        upTriFactor->csrMat->column_indices->data().get(),
                        upTriFactor->solveInfo,
                        xarray, tempGPU->data().get()
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        ,upTriFactor->solvePolicy, upTriFactor->solveBuffer
                      #endif
);CHKERRCUSPARSE(stat);

  /* Last, reorder with the column permutation */
  thrust::copy(thrust::cuda::par.on(PetscDefaultCudaStream),thrust::make_permutation_iterator(tempGPU->begin(), cusparseTriFactors->cpermIndices->begin()),
               thrust::make_permutation_iterator(tempGPU->begin(), cusparseTriFactors->cpermIndices->end()),
               xGPU);

  ierr = VecCUDARestoreArrayRead(bb,&barray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(xx,&xarray);CHKERRQ(ierr);
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*cusparseTriFactors->nnz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqAIJCUSPARSE_NaturalOrdering(Mat A,Vec bb,Vec xx)
{
  const PetscScalar                 *barray;
  PetscScalar                       *xarray;
  hipsparseStatus_t                  stat;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  THRUSTARRAY                       *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;
  PetscErrorCode                    ierr;
  hipError_t                       cerr;

  PetscFunctionBegin;
  /* Get the GPU pointers */
  ierr = VecCUDAGetArrayWrite(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(bb,&barray);CHKERRQ(ierr);

  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  /* First, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactor->solveOp,
                        loTriFactor->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        loTriFactor->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, loTriFactor->descr,
                        loTriFactor->csrMat->values->data().get(),
                        loTriFactor->csrMat->row_offsets->data().get(),
                        loTriFactor->csrMat->column_indices->data().get(),
                        loTriFactor->solveInfo,
                        barray, tempGPU->data().get()
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        ,loTriFactor->solvePolicy, loTriFactor->solveBuffer
                      #endif
);CHKERRCUSPARSE(stat);

  /* Next, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactor->solveOp,
                        upTriFactor->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        upTriFactor->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, upTriFactor->descr,
                        upTriFactor->csrMat->values->data().get(),
                        upTriFactor->csrMat->row_offsets->data().get(),
                        upTriFactor->csrMat->column_indices->data().get(),
                        upTriFactor->solveInfo,
                        tempGPU->data().get(), xarray
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        ,upTriFactor->solvePolicy, upTriFactor->solveBuffer
                      #endif
);CHKERRCUSPARSE(stat);

  ierr = VecCUDARestoreArrayRead(bb,&barray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(xx,&xarray);CHKERRQ(ierr);
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*cusparseTriFactors->nnz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSECopyFromGPU(Mat A)
{
  Mat_SeqAIJ         *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  hipError_t        cerr;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (A->offloadmask == PETSC_OFFLOAD_GPU) {
    CsrMatrix *matrix = (CsrMatrix*)cusp->mat->mat;

    ierr = PetscLogEventBegin(MAT_CUSPARSECopyFromGPU,A,0,0,0);CHKERRQ(ierr);
    cerr = hipMemcpy(a->a, matrix->values->data().get(), a->nz*sizeof(PetscScalar), hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    cerr = WaitForCUDA();CHKERRCUDA(cerr);
    ierr = PetscLogGpuToCpu(a->nz*sizeof(PetscScalar));CHKERRQ(ierr);
    ierr = PetscLogEventEnd(MAT_CUSPARSECopyFromGPU,A,0,0,0);CHKERRQ(ierr);
    A->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJGetArray_SeqAIJCUSPARSE(Mat A,PetscScalar *array[])
{
  Mat_SeqAIJ     *a = (Mat_SeqAIJ*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSECopyFromGPU(A);CHKERRQ(ierr);
  *array = a->a;
  A->offloadmask = PETSC_OFFLOAD_CPU;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSECopyToGPU(Mat A)
{
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct = cusparsestruct->mat;
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  PetscInt                     m = A->rmap->n,*ii,*ridx,tmp;
  PetscErrorCode               ierr;
  hipsparseStatus_t             stat;
  PetscBool                    both = PETSC_TRUE;
  hipError_t                  err;

  PetscFunctionBegin;
  if (A->boundtocpu) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Cannot copy to GPU");
  if (A->offloadmask == PETSC_OFFLOAD_UNALLOCATED || A->offloadmask == PETSC_OFFLOAD_CPU) {
    if (A->nonzerostate == cusparsestruct->nonzerostate && cusparsestruct->format == MAT_CUSPARSE_CSR) { /* Copy values only */
      CsrMatrix *matrix;
      matrix = (CsrMatrix*)cusparsestruct->mat->mat;

      if (a->nz && !a->a) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CSR values");
      ierr = PetscLogEventBegin(MAT_CUSPARSECopyToGPU,A,0,0,0);CHKERRQ(ierr);
      matrix->values->assign(a->a, a->a+a->nz);
      err  = WaitForCUDA();CHKERRCUDA(err);
      ierr = PetscLogCpuToGpu((a->nz)*sizeof(PetscScalar));CHKERRQ(ierr);
      ierr = PetscLogEventEnd(MAT_CUSPARSECopyToGPU,A,0,0,0);CHKERRQ(ierr);
      ierr = MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_FALSE);CHKERRQ(ierr);
    } else {
      PetscInt nnz;
      ierr = PetscLogEventBegin(MAT_CUSPARSECopyToGPU,A,0,0,0);CHKERRQ(ierr);
      ierr = MatSeqAIJCUSPARSEMultStruct_Destroy(&cusparsestruct->mat,cusparsestruct->format);CHKERRQ(ierr);
      ierr = MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_TRUE);CHKERRQ(ierr);
      delete cusparsestruct->workVector;
      delete cusparsestruct->rowoffsets_gpu;
      cusparsestruct->workVector = NULL;
      cusparsestruct->rowoffsets_gpu = NULL;
      try {
        if (a->compressedrow.use) {
          m    = a->compressedrow.nrows;
          ii   = a->compressedrow.i;
          ridx = a->compressedrow.rindex;
        } else {
          m    = A->rmap->n;
          ii   = a->i;
          ridx = NULL;
        }
        if (!ii) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CSR row data");
        if (m && !a->j) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CSR column data");
        if (!a->a) { nnz = ii[m]; both = PETSC_FALSE; }
        else nnz = a->nz;

        /* create cusparse matrix */
        cusparsestruct->nrows = m;
        matstruct = new Mat_SeqAIJCUSPARSEMultStruct;
        stat = hipsparseCreateMatDescr(&matstruct->descr);CHKERRCUSPARSE(stat);
        stat = hipsparseSetMatIndexBase(matstruct->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
        stat = hipsparseSetMatType(matstruct->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUSPARSE(stat);

        err = hipMalloc((void **)&(matstruct->alpha_one),sizeof(PetscScalar));CHKERRCUDA(err);
        err = hipMalloc((void **)&(matstruct->beta_zero),sizeof(PetscScalar));CHKERRCUDA(err);
        err = hipMalloc((void **)&(matstruct->beta_one), sizeof(PetscScalar));CHKERRCUDA(err);
        err = hipMemcpy(matstruct->alpha_one,&PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
        err = hipMemcpy(matstruct->beta_zero,&PETSC_CUSPARSE_ZERO,sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
        err = hipMemcpy(matstruct->beta_one, &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
        stat = hipsparseSetPointerMode(cusparsestruct->handle, HIPSPARSE_POINTER_MODE_DEVICE);CHKERRCUSPARSE(stat);

        /* Build a hybrid/ellpack matrix if this option is chosen for the storage */
        if (cusparsestruct->format==MAT_CUSPARSE_CSR) {
          /* set the matrix */
          CsrMatrix *mat= new CsrMatrix;
          mat->num_rows = m;
          mat->num_cols = A->cmap->n;
          mat->num_entries = nnz;
          mat->row_offsets = new THRUSTINTARRAY32(m+1);
          mat->row_offsets->assign(ii, ii + m+1);

          mat->column_indices = new THRUSTINTARRAY32(nnz);
          mat->column_indices->assign(a->j, a->j+nnz);

          mat->values = new THRUSTARRAY(nnz);
          if (a->a) mat->values->assign(a->a, a->a+nnz);

          /* assign the pointer */
          matstruct->mat = mat;
         #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
          if (mat->num_rows) { /* cusparse errors on empty matrices! */
            stat = hipsparseCreateCsr(&matstruct->matDescr,
                                    mat->num_rows, mat->num_cols, mat->num_entries,
                                    mat->row_offsets->data().get(), mat->column_indices->data().get(),
                                    mat->values->data().get(),
                                    HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I, /* row offset, col idx types due to THRUSTINTARRAY32 */
                                    HIPSPARSE_INDEX_BASE_ZERO,cusparse_scalartype);CHKERRCUSPARSE(stat);
          }
         #endif
        } else if (cusparsestruct->format==MAT_CUSPARSE_ELL || cusparsestruct->format==MAT_CUSPARSE_HYB) {
         #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
          SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"MAT_CUSPARSE_ELL and MAT_CUSPARSE_HYB are not supported since CUDA-11.0");
         #else
          CsrMatrix *mat= new CsrMatrix;
          mat->num_rows = m;
          mat->num_cols = A->cmap->n;
          mat->num_entries = nnz;
          mat->row_offsets = new THRUSTINTARRAY32(m+1);
          mat->row_offsets->assign(ii, ii + m+1);

          mat->column_indices = new THRUSTINTARRAY32(nnz);
          mat->column_indices->assign(a->j, a->j+nnz);

          mat->values = new THRUSTARRAY(nnz);
          if (a->a) mat->values->assign(a->a, a->a+nnz);

          hipsparseHybMat_t hybMat;
          stat = hipsparseCreateHybMat(&hybMat);CHKERRCUSPARSE(stat);
          hipsparseHybPartition_t partition = cusparsestruct->format==MAT_CUSPARSE_ELL ?
            HIPSPARSE_HYB_PARTITION_MAX : HIPSPARSE_HYB_PARTITION_AUTO;
          stat = cusparse_csr2hyb(cusparsestruct->handle, mat->num_rows, mat->num_cols,
              matstruct->descr, mat->values->data().get(),
              mat->row_offsets->data().get(),
              mat->column_indices->data().get(),
              hybMat, 0, partition);CHKERRCUSPARSE(stat);
          /* assign the pointer */
          matstruct->mat = hybMat;

          if (mat) {
            if (mat->values) delete (THRUSTARRAY*)mat->values;
            if (mat->column_indices) delete (THRUSTINTARRAY32*)mat->column_indices;
            if (mat->row_offsets) delete (THRUSTINTARRAY32*)mat->row_offsets;
            delete (CsrMatrix*)mat;
          }
         #endif
        }

        /* assign the compressed row indices */
        if (a->compressedrow.use) {
          cusparsestruct->workVector = new THRUSTARRAY(m);
          matstruct->cprowIndices    = new THRUSTINTARRAY(m);
          matstruct->cprowIndices->assign(ridx,ridx+m);
          tmp = m;
        } else {
          cusparsestruct->workVector = NULL;
          matstruct->cprowIndices    = NULL;
          tmp = 0;
        }
        ierr = PetscLogCpuToGpu(((m+1)+(a->nz))*sizeof(int)+tmp*sizeof(PetscInt)+(3+(a->nz))*sizeof(PetscScalar));CHKERRQ(ierr);

        /* assign the pointer */
        cusparsestruct->mat = matstruct;
      } catch(char *ex) {
        SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
      }
      err  = WaitForCUDA();CHKERRCUDA(err);
      ierr = PetscLogEventEnd(MAT_CUSPARSECopyToGPU,A,0,0,0);CHKERRQ(ierr);
      cusparsestruct->nonzerostate = A->nonzerostate;
    }
    if (both) A->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

struct VecCUDAPlusEquals
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<1>(t) = thrust::get<1>(t) + thrust::get<0>(t);
  }
};

struct VecCUDAEquals
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<1>(t) = thrust::get<0>(t);
  }
};

struct VecCUDAEqualsReverse
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t);
  }
};

struct MatMatCusparse {
  PetscBool             cisdense;
  PetscScalar           *Bt;
  Mat                   X;
  PetscBool             reusesym; /* Cusparse does not have split symbolic and numeric phases for sparse matmat operations */
  PetscLogDouble        flops;
  CsrMatrix             *Bcsr;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  hipsparseSpMatDescr_t  matSpBDescr;
  PetscBool             initialized;   /* C = alpha op(A) op(B) + beta C */
  hipsparseDnMatDescr_t  matBDescr;
  hipsparseDnMatDescr_t  matCDescr;
  PetscInt              Blda,Clda; /* Record leading dimensions of B and C here to detect changes*/
  size_t                mmBufferSize;
  void                  *mmBuffer;
  void                  *mmBuffer2; /* SpGEMM WorkEstimation buffer */
  hipsparseSpGEMMDescr_t spgemmDesc;
#endif
};

static PetscErrorCode MatDestroy_MatMatCusparse(void *data)
{
  PetscErrorCode   ierr;
  MatMatCusparse   *mmdata = (MatMatCusparse *)data;
  hipError_t      cerr;
 #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  hipsparseStatus_t stat;
 #endif

  PetscFunctionBegin;
  cerr = hipFree(mmdata->Bt);CHKERRCUDA(cerr);
  delete mmdata->Bcsr;
 #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  if (mmdata->matSpBDescr) { stat = hipsparseDestroySpMat(mmdata->matSpBDescr);CHKERRCUSPARSE(stat); }
  if (mmdata->mmBuffer)    { cerr = hipFree(mmdata->mmBuffer);CHKERRCUDA(cerr); }
  if (mmdata->mmBuffer2)   { cerr = hipFree(mmdata->mmBuffer2);CHKERRCUDA(cerr); }
  if (mmdata->matBDescr)   { stat = hipsparseDestroyDnMat(mmdata->matBDescr);CHKERRCUSPARSE(stat); }
  if (mmdata->matCDescr)   { stat = hipsparseDestroyDnMat(mmdata->matCDescr);CHKERRCUSPARSE(stat); }
  if (mmdata->spgemmDesc)  { stat = hipsparseSpGEMM_destroyDescr(mmdata->spgemmDesc);CHKERRCUSPARSE(stat); }
 #endif
  ierr = MatDestroy(&mmdata->X);CHKERRQ(ierr);
  ierr = PetscFree(data);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(Mat,Mat,Mat,PetscBool,PetscBool);

static PetscErrorCode MatProductNumeric_SeqAIJCUSPARSE_SeqDENSECUDA(Mat C)
{
  Mat_Product                  *product = C->product;
  Mat                          A,B;
  PetscInt                     m,n,blda,clda;
  PetscBool                    flg,biscuda;
  Mat_SeqAIJCUSPARSE           *cusp;
  hipsparseStatus_t             stat;
  hipsparseOperation_t          opA;
  const PetscScalar            *barray;
  PetscScalar                  *carray;
  PetscErrorCode               ierr;
  MatMatCusparse               *mmdata;
  Mat_SeqAIJCUSPARSEMultStruct *mat;
  CsrMatrix                    *csrmat;
  hipError_t                  cerr;

  PetscFunctionBegin;
  MatCheckProduct(C,1);
  if (!C->product->data) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Product data empty");
  mmdata = (MatMatCusparse*)product->data;
  A    = product->A;
  B    = product->B;
  ierr = PetscObjectTypeCompare((PetscObject)A,MATSEQAIJCUSPARSE,&flg);CHKERRQ(ierr);
  if (!flg) SETERRQ1(PetscObjectComm((PetscObject)A),PETSC_ERR_GPU,"Not for type %s",((PetscObject)A)->type_name);
  /* currently CopyToGpu does not copy if the matrix is bound to CPU
     Instead of silently accepting the wrong answer, I prefer to raise the error */
  if (A->boundtocpu) SETERRQ(PetscObjectComm((PetscObject)A),PETSC_ERR_ARG_WRONG,"Cannot bind to CPU a CUSPARSE matrix between MatProductSymbolic and MatProductNumeric phases");
  ierr   = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  cusp   = (Mat_SeqAIJCUSPARSE*)A->spptr;
  switch (product->type) {
  case MATPRODUCT_AB:
  case MATPRODUCT_PtAP:
    mat = cusp->mat;
    opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    m   = A->rmap->n;
    n   = B->cmap->n;
    break;
  case MATPRODUCT_AtB:
    if (!A->form_explicit_transpose) {
      mat = cusp->mat;
      opA = HIPSPARSE_OPERATION_TRANSPOSE;
    } else {
      ierr = MatSeqAIJCUSPARSEFormExplicitTransposeForMult(A);CHKERRQ(ierr);
      mat  = cusp->matTranspose;
      opA  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    }
    m = A->cmap->n;
    n = B->cmap->n;
    break;
  case MATPRODUCT_ABt:
  case MATPRODUCT_RARt:
    mat = cusp->mat;
    opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    m   = A->rmap->n;
    n   = B->rmap->n;
    break;
  default:
    SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Unsupported product type %s",MatProductTypes[product->type]);
  }
  if (!mat) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing Mat_SeqAIJCUSPARSEMultStruct");
  csrmat = (CsrMatrix*)mat->mat;
  /* if the user passed a CPU matrix, copy the data to the GPU */
  ierr = PetscObjectTypeCompare((PetscObject)B,MATSEQDENSECUDA,&biscuda);CHKERRQ(ierr);
  if (!biscuda) {ierr = MatConvert(B,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&B);CHKERRQ(ierr);}
  ierr = MatDenseCUDAGetArrayRead(B,&barray);CHKERRQ(ierr);

  ierr = MatDenseGetLDA(B,&blda);CHKERRQ(ierr);
  if (product->type == MATPRODUCT_RARt || product->type == MATPRODUCT_PtAP) {
    ierr = MatDenseCUDAGetArrayWrite(mmdata->X,&carray);CHKERRQ(ierr);
    ierr = MatDenseGetLDA(mmdata->X,&clda);CHKERRQ(ierr);
  } else {
    ierr = MatDenseCUDAGetArrayWrite(C,&carray);CHKERRQ(ierr);
    ierr = MatDenseGetLDA(C,&clda);CHKERRQ(ierr);
  }

  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
 #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  hipsparseOperation_t opB = (product->type == MATPRODUCT_ABt || product->type == MATPRODUCT_RARt) ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;
  /* (re)allcoate mmBuffer if not initialized or LDAs are different */
  if (!mmdata->initialized || mmdata->Blda != blda || mmdata->Clda != clda) {
    size_t mmBufferSize;
    if (mmdata->initialized && mmdata->Blda != blda) {stat = hipsparseDestroyDnMat(mmdata->matBDescr);CHKERRCUSPARSE(stat); mmdata->matBDescr = NULL;}
    if (!mmdata->matBDescr) {
      stat         = hipsparseCreateDnMat(&mmdata->matBDescr,B->rmap->n,B->cmap->n,blda,(void*)barray,cusparse_scalartype,HIPSPARSE_ORDER_COL);CHKERRCUSPARSE(stat);
      mmdata->Blda = blda;
    }

    if (mmdata->initialized && mmdata->Clda != clda) {stat = hipsparseDestroyDnMat(mmdata->matCDescr);CHKERRCUSPARSE(stat); mmdata->matCDescr = NULL;}
    if (!mmdata->matCDescr) { /* matCDescr is for C or mmdata->X */
      stat         = hipsparseCreateDnMat(&mmdata->matCDescr,m,n,clda,(void*)carray,cusparse_scalartype,HIPSPARSE_ORDER_COL);CHKERRCUSPARSE(stat);
      mmdata->Clda = clda;
    }

    if (!mat->matDescr) {
      stat = hipsparseCreateCsr(&mat->matDescr,
                               csrmat->num_rows, csrmat->num_cols, csrmat->num_entries,
                               csrmat->row_offsets->data().get(), csrmat->column_indices->data().get(),
                               csrmat->values->data().get(),
                               HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I, /* row offset, col idx types due to THRUSTINTARRAY32 */
                               HIPSPARSE_INDEX_BASE_ZERO,cusparse_scalartype);CHKERRCUSPARSE(stat);
    }
    stat = hipsparseSpMM_bufferSize(cusp->handle,opA,opB,mat->alpha_one,
                                   mat->matDescr,mmdata->matBDescr,mat->beta_zero,
                                   mmdata->matCDescr,cusparse_scalartype,
                                   cusp->spmmAlg,&mmBufferSize);CHKERRCUSPARSE(stat);
    if ((mmdata->mmBuffer && mmdata->mmBufferSize < mmBufferSize) || !mmdata->mmBuffer) {
      cerr = hipFree(mmdata->mmBuffer);CHKERRCUDA(cerr);
      cerr = hipMalloc(&mmdata->mmBuffer,mmBufferSize);CHKERRCUDA(cerr);
      mmdata->mmBufferSize = mmBufferSize;
    }
    mmdata->initialized = PETSC_TRUE;
  } else {
    /* to be safe, always update pointers of the mats */
    stat = hipsparseSpMatSetValues(mat->matDescr,csrmat->values->data().get());CHKERRCUSPARSE(stat);
    stat = hipsparseDnMatSetValues(mmdata->matBDescr,(void*)barray);CHKERRCUSPARSE(stat);
    stat = hipsparseDnMatSetValues(mmdata->matCDescr,(void*)carray);CHKERRCUSPARSE(stat);
  }

  /* do hipsparseSpMM, which supports transpose on B */
  stat = hipsparseSpMM(cusp->handle,opA,opB,mat->alpha_one,
                      mat->matDescr,mmdata->matBDescr,mat->beta_zero,
                      mmdata->matCDescr,cusparse_scalartype,
                      cusp->spmmAlg,mmdata->mmBuffer);CHKERRCUSPARSE(stat);
 #else
  PetscInt k;
  /* cusparseXcsrmm does not support transpose on B */
  if (product->type == MATPRODUCT_ABt || product->type == MATPRODUCT_RARt) {
    hipblasHandle_t cublasv2handle;
    hipblasStatus_t cerr;

    ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
    cerr = cublasXgeam(cublasv2handle,HIPBLAS_OP_T,HIPBLAS_OP_T,
                       B->cmap->n,B->rmap->n,
                       &PETSC_CUSPARSE_ONE ,barray,blda,
                       &PETSC_CUSPARSE_ZERO,barray,blda,
                       mmdata->Bt,B->cmap->n);CHKERRCUBLAS(cerr);
    blda = B->cmap->n;
    k    = B->cmap->n;
  } else {
    k    = B->rmap->n;
  }

  /* perform the MatMat operation, op(A) is m x k, op(B) is k x n */
  stat = cusparse_csr_spmm(cusp->handle,opA,m,n,k,
                           csrmat->num_entries,mat->alpha_one,mat->descr,
                           csrmat->values->data().get(),
                           csrmat->row_offsets->data().get(),
                           csrmat->column_indices->data().get(),
                           mmdata->Bt ? mmdata->Bt : barray,blda,mat->beta_zero,
                           carray,clda);CHKERRCUSPARSE(stat);
 #endif
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(n*2.0*csrmat->num_entries);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(B,&barray);CHKERRQ(ierr);
  if (product->type == MATPRODUCT_RARt) {
    ierr = MatDenseCUDARestoreArrayWrite(mmdata->X,&carray);CHKERRQ(ierr);
    ierr = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(B,mmdata->X,C,PETSC_FALSE,PETSC_FALSE);CHKERRQ(ierr);
  } else if (product->type == MATPRODUCT_PtAP) {
    ierr = MatDenseCUDARestoreArrayWrite(mmdata->X,&carray);CHKERRQ(ierr);
    ierr = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(B,mmdata->X,C,PETSC_TRUE,PETSC_FALSE);CHKERRQ(ierr);
  } else {
    ierr = MatDenseCUDARestoreArrayWrite(C,&carray);CHKERRQ(ierr);
  }
  if (mmdata->cisdense) {
    ierr = MatConvert(C,MATSEQDENSE,MAT_INPLACE_MATRIX,&C);CHKERRQ(ierr);
  }
  if (!biscuda) {
    ierr = MatConvert(B,MATSEQDENSE,MAT_INPLACE_MATRIX,&B);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatProductSymbolic_SeqAIJCUSPARSE_SeqDENSECUDA(Mat C)
{
  Mat_Product        *product = C->product;
  Mat                A,B;
  PetscInt           m,n;
  PetscBool          cisdense,flg;
  PetscErrorCode     ierr;
  MatMatCusparse     *mmdata;
  Mat_SeqAIJCUSPARSE *cusp;

  PetscFunctionBegin;
  MatCheckProduct(C,1);
  if (C->product->data) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Product data not empty");
  A    = product->A;
  B    = product->B;
  ierr = PetscObjectTypeCompare((PetscObject)A,MATSEQAIJCUSPARSE,&flg);CHKERRQ(ierr);
  if (!flg) SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Not for type %s",((PetscObject)A)->type_name);
  cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  if (cusp->format != MAT_CUSPARSE_CSR) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");
  switch (product->type) {
  case MATPRODUCT_AB:
    m = A->rmap->n;
    n = B->cmap->n;
    break;
  case MATPRODUCT_AtB:
    m = A->cmap->n;
    n = B->cmap->n;
    break;
  case MATPRODUCT_ABt:
    m = A->rmap->n;
    n = B->rmap->n;
    break;
  case MATPRODUCT_PtAP:
    m = B->cmap->n;
    n = B->cmap->n;
    break;
  case MATPRODUCT_RARt:
    m = B->rmap->n;
    n = B->rmap->n;
    break;
  default:
    SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Unsupported product type %s",MatProductTypes[product->type]);
  }
  ierr = MatSetSizes(C,m,n,m,n);CHKERRQ(ierr);
  /* if C is of type MATSEQDENSE (CPU), perform the operation on the GPU and then copy on the CPU */
  ierr = PetscObjectTypeCompare((PetscObject)C,MATSEQDENSE,&cisdense);CHKERRQ(ierr);
  ierr = MatSetType(C,MATSEQDENSECUDA);CHKERRQ(ierr);

  /* product data */
  ierr = PetscNew(&mmdata);CHKERRQ(ierr);
  mmdata->cisdense = cisdense;
 #if PETSC_PKG_CUDA_VERSION_LT(11,0,0)
  /* cusparseXcsrmm does not support transpose on B, so we allocate buffer to store B^T */
  if (product->type == MATPRODUCT_ABt || product->type == MATPRODUCT_RARt) {
    hipError_t cerr = hipMalloc((void**)&mmdata->Bt,(size_t)B->rmap->n*(size_t)B->cmap->n*sizeof(PetscScalar));CHKERRCUDA(cerr);
  }
 #endif
  /* for these products we need intermediate storage */
  if (product->type == MATPRODUCT_RARt || product->type == MATPRODUCT_PtAP) {
    ierr = MatCreate(PetscObjectComm((PetscObject)C),&mmdata->X);CHKERRQ(ierr);
    ierr = MatSetType(mmdata->X,MATSEQDENSECUDA);CHKERRQ(ierr);
    if (product->type == MATPRODUCT_RARt) { /* do not preallocate, since the first call to MatDenseCUDAGetArray will preallocate on the GPU for us */
      ierr = MatSetSizes(mmdata->X,A->rmap->n,B->rmap->n,A->rmap->n,B->rmap->n);CHKERRQ(ierr);
    } else {
      ierr = MatSetSizes(mmdata->X,A->rmap->n,B->cmap->n,A->rmap->n,B->cmap->n);CHKERRQ(ierr);
    }
  }
  C->product->data    = mmdata;
  C->product->destroy = MatDestroy_MatMatCusparse;

  C->ops->productnumeric = MatProductNumeric_SeqAIJCUSPARSE_SeqDENSECUDA;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatProductNumeric_SeqAIJCUSPARSE_SeqAIJCUSPARSE(Mat C)
{
  Mat_Product                  *product = C->product;
  Mat                          A,B;
  Mat_SeqAIJCUSPARSE           *Acusp,*Bcusp,*Ccusp;
  Mat_SeqAIJ                   *c = (Mat_SeqAIJ*)C->data;
  Mat_SeqAIJCUSPARSEMultStruct *Amat,*Bmat,*Cmat;
  CsrMatrix                    *Acsr,*Bcsr,*Ccsr;
  PetscBool                    flg;
  PetscErrorCode               ierr;
  hipsparseStatus_t             stat;
  hipError_t                  cerr;
  MatProductType               ptype;
  MatMatCusparse               *mmdata;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  hipsparseSpMatDescr_t         BmatSpDescr;
#endif

  PetscFunctionBegin;
  MatCheckProduct(C,1);
  if (!C->product->data) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Product data empty");
  ierr = PetscObjectTypeCompare((PetscObject)C,MATSEQAIJCUSPARSE,&flg);CHKERRQ(ierr);
  if (!flg) SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Not for C of type %s",((PetscObject)C)->type_name);
  mmdata = (MatMatCusparse*)C->product->data;
  A = product->A;
  B = product->B;
  if (mmdata->reusesym) { /* this happens when api_user is true, meaning that the matrix values have been already computed in the MatProductSymbolic phase */
    mmdata->reusesym = PETSC_FALSE;
    Ccusp = (Mat_SeqAIJCUSPARSE*)C->spptr;
    if (Ccusp->format != MAT_CUSPARSE_CSR) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");
    Cmat = Ccusp->mat;
    if (!Cmat) SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing C mult struct for product type %s",MatProductTypes[C->product->type]);
    Ccsr = (CsrMatrix*)Cmat->mat;
    if (!Ccsr) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing C CSR struct");
    goto finalize;
  }
  if (!c->nz) goto finalize;
  ierr = PetscObjectTypeCompare((PetscObject)A,MATSEQAIJCUSPARSE,&flg);CHKERRQ(ierr);
  if (!flg) SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Not for type %s",((PetscObject)A)->type_name);
  ierr = PetscObjectTypeCompare((PetscObject)B,MATSEQAIJCUSPARSE,&flg);CHKERRQ(ierr);
  if (!flg) SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Not for B of type %s",((PetscObject)B)->type_name);
  if (A->boundtocpu) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_ARG_WRONG,"Cannot bind to CPU a CUSPARSE matrix between MatProductSymbolic and MatProductNumeric phases");
  if (B->boundtocpu) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_ARG_WRONG,"Cannot bind to CPU a CUSPARSE matrix between MatProductSymbolic and MatProductNumeric phases");
  Acusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Bcusp = (Mat_SeqAIJCUSPARSE*)B->spptr;
  Ccusp = (Mat_SeqAIJCUSPARSE*)C->spptr;
  if (Acusp->format != MAT_CUSPARSE_CSR) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");
  if (Bcusp->format != MAT_CUSPARSE_CSR) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");
  if (Ccusp->format != MAT_CUSPARSE_CSR) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");
  ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  ierr = MatSeqAIJCUSPARSECopyToGPU(B);CHKERRQ(ierr);

  ptype = product->type;
  if (A->symmetric && ptype == MATPRODUCT_AtB) ptype = MATPRODUCT_AB;
  if (B->symmetric && ptype == MATPRODUCT_ABt) ptype = MATPRODUCT_AB;
  switch (ptype) {
  case MATPRODUCT_AB:
    Amat = Acusp->mat;
    Bmat = Bcusp->mat;
    break;
  case MATPRODUCT_AtB:
    Amat = Acusp->matTranspose;
    Bmat = Bcusp->mat;
    break;
  case MATPRODUCT_ABt:
    Amat = Acusp->mat;
    Bmat = Bcusp->matTranspose;
    break;
  default:
    SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Unsupported product type %s",MatProductTypes[product->type]);
  }
  Cmat = Ccusp->mat;
  if (!Amat) SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing A mult struct for product type %s",MatProductTypes[ptype]);
  if (!Bmat) SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing B mult struct for product type %s",MatProductTypes[ptype]);
  if (!Cmat) SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing C mult struct for product type %s",MatProductTypes[ptype]);
  Acsr = (CsrMatrix*)Amat->mat;
  Bcsr = mmdata->Bcsr ? mmdata->Bcsr : (CsrMatrix*)Bmat->mat; /* B may be in compressed row storage */
  Ccsr = (CsrMatrix*)Cmat->mat;
  if (!Acsr) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing A CSR struct");
  if (!Bcsr) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing B CSR struct");
  if (!Ccsr) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing C CSR struct");
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  BmatSpDescr = mmdata->Bcsr ? mmdata->matSpBDescr : Bmat->matDescr; /* B may be in compressed row storage */
  stat = hipsparseSpGEMM_compute(Ccusp->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                                cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT,
                                mmdata->spgemmDesc, &mmdata->mmBufferSize, mmdata->mmBuffer);CHKERRCUSPARSE(stat);
  stat = hipsparseSpGEMM_copy(Ccusp->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                             cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT, mmdata->spgemmDesc);CHKERRCUSPARSE(stat);
#else
  stat = cusparse_csr_spgemm(Ccusp->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             Acsr->num_rows, Bcsr->num_cols, Acsr->num_cols,
                             Amat->descr, Acsr->num_entries, Acsr->values->data().get(), Acsr->row_offsets->data().get(), Acsr->column_indices->data().get(),
                             Bmat->descr, Bcsr->num_entries, Bcsr->values->data().get(), Bcsr->row_offsets->data().get(), Bcsr->column_indices->data().get(),
                             Cmat->descr, Ccsr->values->data().get(), Ccsr->row_offsets->data().get(), Ccsr->column_indices->data().get());CHKERRCUSPARSE(stat);
#endif
  ierr = PetscLogGpuFlops(mmdata->flops);CHKERRQ(ierr);
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  C->offloadmask = PETSC_OFFLOAD_GPU;
finalize:
  /* shorter version of MatAssemblyEnd_SeqAIJ */
  ierr = PetscInfo3(C,"Matrix size: %D X %D; storage space: 0 unneeded,%D used\n",C->rmap->n,C->cmap->n,c->nz);CHKERRQ(ierr);
  ierr = PetscInfo(C,"Number of mallocs during MatSetValues() is 0\n");CHKERRQ(ierr);
  ierr = PetscInfo1(C,"Maximum nonzeros in any row is %D\n",c->rmax);CHKERRQ(ierr);
  c->reallocs         = 0;
  C->info.mallocs    += 0;
  C->info.nz_unneeded = 0;
  C->assembled = C->was_assembled = PETSC_TRUE;
  C->num_ass++;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatProductSymbolic_SeqAIJCUSPARSE_SeqAIJCUSPARSE(Mat C)
{
  Mat_Product                  *product = C->product;
  Mat                          A,B;
  Mat_SeqAIJCUSPARSE           *Acusp,*Bcusp,*Ccusp;
  Mat_SeqAIJ                   *a,*b,*c;
  Mat_SeqAIJCUSPARSEMultStruct *Amat,*Bmat,*Cmat;
  CsrMatrix                    *Acsr,*Bcsr,*Ccsr;
  PetscInt                     i,j,m,n,k;
  PetscBool                    flg;
  PetscErrorCode               ierr;
  hipsparseStatus_t             stat;
  hipError_t                  cerr;
  MatProductType               ptype;
  MatMatCusparse               *mmdata;
  PetscLogDouble               flops;
  PetscBool                    biscompressed,ciscompressed;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  int64_t                      C_num_rows1, C_num_cols1, C_nnz1;
  size_t                       bufSize2;
  hipsparseSpMatDescr_t         BmatSpDescr;
#else
  int                          cnz;
#endif

  PetscFunctionBegin;
  MatCheckProduct(C,1);
  if (C->product->data) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Product data not empty");
  A    = product->A;
  B    = product->B;
  ierr = PetscObjectTypeCompare((PetscObject)A,MATSEQAIJCUSPARSE,&flg);CHKERRQ(ierr);
  if (!flg) SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Not for type %s",((PetscObject)A)->type_name);
  ierr = PetscObjectTypeCompare((PetscObject)B,MATSEQAIJCUSPARSE,&flg);CHKERRQ(ierr);
  if (!flg) SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Not for B of type %s",((PetscObject)B)->type_name);
  a = (Mat_SeqAIJ*)A->data;
  b = (Mat_SeqAIJ*)B->data;
  Acusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Bcusp = (Mat_SeqAIJCUSPARSE*)B->spptr;
  if (Acusp->format != MAT_CUSPARSE_CSR) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");
  if (Bcusp->format != MAT_CUSPARSE_CSR) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");

  /* product data */
  ierr = PetscNew(&mmdata);CHKERRQ(ierr);
  C->product->data    = mmdata;
  C->product->destroy = MatDestroy_MatMatCusparse;

  ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  ierr = MatSeqAIJCUSPARSECopyToGPU(B);CHKERRQ(ierr);
  ptype = product->type;
  if (A->symmetric && ptype == MATPRODUCT_AtB) ptype = MATPRODUCT_AB;
  if (B->symmetric && ptype == MATPRODUCT_ABt) ptype = MATPRODUCT_AB;
  biscompressed = PETSC_FALSE;
  ciscompressed = PETSC_FALSE;
  switch (ptype) {
  case MATPRODUCT_AB:
    m = A->rmap->n;
    n = B->cmap->n;
    k = A->cmap->n;
    Amat = Acusp->mat;
    Bmat = Bcusp->mat;
    if (a->compressedrow.use) ciscompressed = PETSC_TRUE;
    if (b->compressedrow.use) biscompressed = PETSC_TRUE;
    break;
  case MATPRODUCT_AtB:
    m = A->cmap->n;
    n = B->cmap->n;
    k = A->rmap->n;
    ierr = MatSeqAIJCUSPARSEFormExplicitTransposeForMult(A);CHKERRQ(ierr);
    Amat = Acusp->matTranspose;
    Bmat = Bcusp->mat;
    if (b->compressedrow.use) biscompressed = PETSC_TRUE;
    break;
  case MATPRODUCT_ABt:
    m = A->rmap->n;
    n = B->rmap->n;
    k = A->cmap->n;
    ierr = MatSeqAIJCUSPARSEFormExplicitTransposeForMult(B);CHKERRQ(ierr);
    Amat = Acusp->mat;
    Bmat = Bcusp->matTranspose;
    if (a->compressedrow.use) ciscompressed = PETSC_TRUE;
    break;
  default:
    SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Unsupported product type %s",MatProductTypes[product->type]);
  }

  /* create cusparse matrix */
  ierr  = MatSetSizes(C,m,n,m,n);CHKERRQ(ierr);
  ierr  = MatSetType(C,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
  c     = (Mat_SeqAIJ*)C->data;
  Ccusp = (Mat_SeqAIJCUSPARSE*)C->spptr;
  Cmat  = new Mat_SeqAIJCUSPARSEMultStruct;
  Ccsr  = new CsrMatrix;

  c->compressedrow.use = ciscompressed;
  if (c->compressedrow.use) { /* if a is in compressed row, than c will be in compressed row format */
    c->compressedrow.nrows = a->compressedrow.nrows;
    ierr = PetscMalloc2(c->compressedrow.nrows+1,&c->compressedrow.i,c->compressedrow.nrows,&c->compressedrow.rindex);CHKERRQ(ierr);
    ierr = PetscArraycpy(c->compressedrow.rindex,a->compressedrow.rindex,c->compressedrow.nrows);CHKERRQ(ierr);
    Ccusp->workVector  = new THRUSTARRAY(c->compressedrow.nrows);
    Cmat->cprowIndices = new THRUSTINTARRAY(c->compressedrow.nrows);
    Cmat->cprowIndices->assign(c->compressedrow.rindex,c->compressedrow.rindex + c->compressedrow.nrows);
  } else {
    c->compressedrow.nrows  = 0;
    c->compressedrow.i      = NULL;
    c->compressedrow.rindex = NULL;
    Ccusp->workVector       = NULL;
    Cmat->cprowIndices      = NULL;
  }
  Ccusp->nrows    = ciscompressed ? c->compressedrow.nrows : m;
  Ccusp->mat      = Cmat;
  Ccusp->mat->mat = Ccsr;
  Ccsr->num_rows    = Ccusp->nrows;
  Ccsr->num_cols    = n;
  Ccsr->row_offsets = new THRUSTINTARRAY32(Ccusp->nrows+1);
  stat = hipsparseCreateMatDescr(&Cmat->descr);CHKERRCUSPARSE(stat);
  stat = hipsparseSetMatIndexBase(Cmat->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
  stat = hipsparseSetMatType(Cmat->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUSPARSE(stat);
  cerr = hipMalloc((void **)&(Cmat->alpha_one),sizeof(PetscScalar));CHKERRCUDA(cerr);
  cerr = hipMalloc((void **)&(Cmat->beta_zero),sizeof(PetscScalar));CHKERRCUDA(cerr);
  cerr = hipMalloc((void **)&(Cmat->beta_one), sizeof(PetscScalar));CHKERRCUDA(cerr);
  cerr = hipMemcpy(Cmat->alpha_one,&PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
  cerr = hipMemcpy(Cmat->beta_zero,&PETSC_CUSPARSE_ZERO,sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
  cerr = hipMemcpy(Cmat->beta_one, &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
  if (!Ccsr->num_rows || !Ccsr->num_cols || !a->nz || !b->nz) { /* cusparse raise errors in different calls when matrices have zero rows/columns! */
    thrust::fill(thrust::device,Ccsr->row_offsets->begin(),Ccsr->row_offsets->end(),0);
    c->nz = 0;
    Ccsr->column_indices = new THRUSTINTARRAY32(c->nz);
    Ccsr->values = new THRUSTARRAY(c->nz);
    goto finalizesym;
  }

  if (!Amat) SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing A mult struct for product type %s",MatProductTypes[ptype]);
  if (!Bmat) SETERRQ1(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing B mult struct for product type %s",MatProductTypes[ptype]);
  Acsr = (CsrMatrix*)Amat->mat;
  if (!biscompressed) {
    Bcsr = (CsrMatrix*)Bmat->mat;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    BmatSpDescr = Bmat->matDescr;
#endif
  } else { /* we need to use row offsets for the full matrix */
    CsrMatrix *cBcsr = (CsrMatrix*)Bmat->mat;
    Bcsr = new CsrMatrix;
    Bcsr->num_rows       = B->rmap->n;
    Bcsr->num_cols       = cBcsr->num_cols;
    Bcsr->num_entries    = cBcsr->num_entries;
    Bcsr->column_indices = cBcsr->column_indices;
    Bcsr->values         = cBcsr->values;
    if (!Bcusp->rowoffsets_gpu) {
      Bcusp->rowoffsets_gpu  = new THRUSTINTARRAY32(B->rmap->n + 1);
      Bcusp->rowoffsets_gpu->assign(b->i,b->i + B->rmap->n + 1);
      ierr = PetscLogCpuToGpu((B->rmap->n + 1)*sizeof(PetscInt));CHKERRQ(ierr);
    }
    Bcsr->row_offsets = Bcusp->rowoffsets_gpu;
    mmdata->Bcsr = Bcsr;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    if (Bcsr->num_rows && Bcsr->num_cols) {
      stat = hipsparseCreateCsr(&mmdata->matSpBDescr, Bcsr->num_rows, Bcsr->num_cols, Bcsr->num_entries,
                               Bcsr->row_offsets->data().get(), Bcsr->column_indices->data().get(),
                               Bcsr->values->data().get(),
                               HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                               HIPSPARSE_INDEX_BASE_ZERO, cusparse_scalartype);CHKERRCUSPARSE(stat);
    }
    BmatSpDescr = mmdata->matSpBDescr;
#endif
  }
  if (!Acsr) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing A CSR struct");
  if (!Bcsr) SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing B CSR struct");
  /* precompute flops count */
  if (ptype == MATPRODUCT_AB) {
    for (i=0, flops = 0; i<A->rmap->n; i++) {
      const PetscInt st = a->i[i];
      const PetscInt en = a->i[i+1];
      for (j=st; j<en; j++) {
        const PetscInt brow = a->j[j];
        flops += 2.*(b->i[brow+1] - b->i[brow]);
      }
    }
  } else if (ptype == MATPRODUCT_AtB) {
    for (i=0, flops = 0; i<A->rmap->n; i++) {
      const PetscInt anzi = a->i[i+1] - a->i[i];
      const PetscInt bnzi = b->i[i+1] - b->i[i];
      flops += (2.*anzi)*bnzi;
    }
  } else { /* TODO */
    flops = 0.;
  }

  mmdata->flops = flops;
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  stat = hipsparseSetPointerMode(Ccusp->handle, HIPSPARSE_POINTER_MODE_DEVICE);CHKERRCUSPARSE(stat);
  stat = hipsparseCreateCsr(&Cmat->matDescr, Ccsr->num_rows, Ccsr->num_cols, 0,
                           NULL, NULL, NULL,
                           HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                           HIPSPARSE_INDEX_BASE_ZERO, cusparse_scalartype);CHKERRCUSPARSE(stat);
  stat = hipsparseSpGEMM_createDescr(&mmdata->spgemmDesc);CHKERRCUSPARSE(stat);
  /* ask bufferSize bytes for external memory */
  stat = hipsparseSpGEMM_workEstimation(Ccusp->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                                       cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT,
                                       mmdata->spgemmDesc, &bufSize2, NULL);CHKERRCUSPARSE(stat);
  cerr = hipMalloc((void**) &mmdata->mmBuffer2, bufSize2);CHKERRCUDA(cerr);
  /* inspect the matrices A and B to understand the memory requirement for the next step */
  stat = hipsparseSpGEMM_workEstimation(Ccusp->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                                       cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT,
                                       mmdata->spgemmDesc, &bufSize2, mmdata->mmBuffer2);CHKERRCUSPARSE(stat);
  /* ask bufferSize again bytes for external memory */
  stat = hipsparseSpGEMM_compute(Ccusp->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                                cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT,
                                mmdata->spgemmDesc, &mmdata->mmBufferSize, NULL);CHKERRCUSPARSE(stat);
  /* The CUSPARSE documentation is not clear, nor the API
     We need both buffers to perform the operations properly!
     mmdata->mmBuffer2 does not appear anywhere in the compute/copy API
     it only appears for the workEstimation stuff, but it seems it is needed in compute, so probably the address
     is stored in the descriptor! What a messy API... */
  cerr = hipMalloc((void**) &mmdata->mmBuffer, mmdata->mmBufferSize);CHKERRCUDA(cerr);
  /* compute the intermediate product of A * B */
  stat = hipsparseSpGEMM_compute(Ccusp->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                                cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT,
                                mmdata->spgemmDesc, &mmdata->mmBufferSize, mmdata->mmBuffer);CHKERRCUSPARSE(stat);
  /* get matrix C non-zero entries C_nnz1 */
  stat = hipsparseSpMatGetSize(Cmat->matDescr, &C_num_rows1, &C_num_cols1, &C_nnz1);CHKERRCUSPARSE(stat);
  c->nz = (PetscInt) C_nnz1;
  ierr = PetscInfo9(C,"Buffer sizes for type %s, result %D x %D (k %D, nzA %D, nzB %D, nzC %D) are: %ldKB %ldKB\n",MatProductTypes[ptype],m,n,k,a->nz,b->nz,c->nz,bufSize2/1024,mmdata->mmBufferSize/1024);CHKERRQ(ierr);
  Ccsr->column_indices = new THRUSTINTARRAY32(c->nz);
  CHKERRCUDA(hipPeekAtLastError()); /* catch out of memory errors */
  Ccsr->values = new THRUSTARRAY(c->nz);
  CHKERRCUDA(hipPeekAtLastError()); /* catch out of memory errors */
  stat = hipsparseCsrSetPointers(Cmat->matDescr, Ccsr->row_offsets->data().get(), Ccsr->column_indices->data().get(),
                                Ccsr->values->data().get());CHKERRCUSPARSE(stat);
  stat = hipsparseSpGEMM_copy(Ccusp->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                             cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT, mmdata->spgemmDesc);CHKERRCUSPARSE(stat);
#else
  stat = hipsparseSetPointerMode(Ccusp->handle, HIPSPARSE_POINTER_MODE_HOST);CHKERRCUSPARSE(stat);
  stat = hipsparseXcsrgemmNnz(Ccusp->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             Acsr->num_rows, Bcsr->num_cols, Acsr->num_cols,
                             Amat->descr, Acsr->num_entries, Acsr->row_offsets->data().get(), Acsr->column_indices->data().get(),
                             Bmat->descr, Bcsr->num_entries, Bcsr->row_offsets->data().get(), Bcsr->column_indices->data().get(),
                             Cmat->descr, Ccsr->row_offsets->data().get(), &cnz);CHKERRCUSPARSE(stat);
  c->nz = cnz;
  Ccsr->column_indices = new THRUSTINTARRAY32(c->nz);
  CHKERRCUDA(hipPeekAtLastError()); /* catch out of memory errors */
  Ccsr->values = new THRUSTARRAY(c->nz);
  CHKERRCUDA(hipPeekAtLastError()); /* catch out of memory errors */

  stat = hipsparseSetPointerMode(Ccusp->handle, HIPSPARSE_POINTER_MODE_DEVICE);CHKERRCUSPARSE(stat);
  /* with the old gemm interface (removed from 11.0 on) we cannot compute the symbolic factorization only.
     I have tried using the gemm2 interface (alpha * A * B + beta * D), which allows to do symbolic by passing NULL for values, but it seems quite buggy when
     D is NULL, despite the fact that CUSPARSE documentation claims it is supported! */
  stat = cusparse_csr_spgemm(Ccusp->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             Acsr->num_rows, Bcsr->num_cols, Acsr->num_cols,
                             Amat->descr, Acsr->num_entries, Acsr->values->data().get(), Acsr->row_offsets->data().get(), Acsr->column_indices->data().get(),
                             Bmat->descr, Bcsr->num_entries, Bcsr->values->data().get(), Bcsr->row_offsets->data().get(), Bcsr->column_indices->data().get(),
                             Cmat->descr, Ccsr->values->data().get(), Ccsr->row_offsets->data().get(), Ccsr->column_indices->data().get());CHKERRCUSPARSE(stat);
#endif
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuFlops(mmdata->flops);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
finalizesym:
  c->singlemalloc = PETSC_FALSE;
  c->free_a       = PETSC_TRUE;
  c->free_ij      = PETSC_TRUE;
  ierr = PetscMalloc1(m+1,&c->i);CHKERRQ(ierr);
  ierr = PetscMalloc1(c->nz,&c->j);CHKERRQ(ierr);
  if (PetscDefined(USE_64BIT_INDICES)) { /* 32 to 64 bit conversion on the GPU and then copy to host (lazy) */
    PetscInt *d_i = c->i;
    THRUSTINTARRAY ii(Ccsr->row_offsets->size());
    THRUSTINTARRAY jj(Ccsr->column_indices->size());
    ii   = *Ccsr->row_offsets;
    jj   = *Ccsr->column_indices;
    if (ciscompressed) d_i = c->compressedrow.i;
    cerr = hipMemcpy(d_i,ii.data().get(),Ccsr->row_offsets->size()*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    cerr = hipMemcpy(c->j,jj.data().get(),Ccsr->column_indices->size()*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
  } else {
    PetscInt *d_i = c->i;
    if (ciscompressed) d_i = c->compressedrow.i;
    cerr = hipMemcpy(d_i,Ccsr->row_offsets->data().get(),Ccsr->row_offsets->size()*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    cerr = hipMemcpy(c->j,Ccsr->column_indices->data().get(),Ccsr->column_indices->size()*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
  }
  if (ciscompressed) { /* need to expand host row offsets */
    PetscInt r = 0;
    c->i[0] = 0;
    for (k = 0; k < c->compressedrow.nrows; k++) {
      const PetscInt next = c->compressedrow.rindex[k];
      const PetscInt old = c->compressedrow.i[k];
      for (; r < next; r++) c->i[r+1] = old;
    }
    for (; r < m; r++) c->i[r+1] = c->compressedrow.i[c->compressedrow.nrows];
  }
  ierr = PetscLogGpuToCpu((Ccsr->column_indices->size() + Ccsr->row_offsets->size())*sizeof(PetscInt));CHKERRQ(ierr);
  ierr = PetscMalloc1(m,&c->ilen);CHKERRQ(ierr);
  ierr = PetscMalloc1(m,&c->imax);CHKERRQ(ierr);
  c->maxnz = c->nz;
  c->nonzerorowcnt = 0;
  c->rmax = 0;
  for (k = 0; k < m; k++) {
    const PetscInt nn = c->i[k+1] - c->i[k];
    c->ilen[k] = c->imax[k] = nn;
    c->nonzerorowcnt += (PetscInt)!!nn;
    c->rmax = PetscMax(c->rmax,nn);
  }
  ierr = MatMarkDiagonal_SeqAIJ(C);CHKERRQ(ierr);
  ierr = PetscMalloc1(c->nz,&c->a);CHKERRQ(ierr);
  Ccsr->num_entries = c->nz;

  C->nonzerostate++;
  ierr = PetscLayoutSetUp(C->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(C->cmap);CHKERRQ(ierr);
  Ccusp->nonzerostate = C->nonzerostate;
  C->offloadmask   = PETSC_OFFLOAD_UNALLOCATED;
  C->preallocated  = PETSC_TRUE;
  C->assembled     = PETSC_FALSE;
  C->was_assembled = PETSC_FALSE;
  if (product->api_user && A->offloadmask == PETSC_OFFLOAD_BOTH && B->offloadmask == PETSC_OFFLOAD_BOTH) { /* flag the matrix C values as computed, so that the numeric phase will only call MatAssembly */
    mmdata->reusesym = PETSC_TRUE;
    C->offloadmask   = PETSC_OFFLOAD_GPU;
  }
  C->ops->productnumeric = MatProductNumeric_SeqAIJCUSPARSE_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatProductSetFromOptions_SeqAIJ_SeqDense(Mat);

/* handles sparse or dense B */
static PetscErrorCode MatProductSetFromOptions_SeqAIJCUSPARSE(Mat mat)
{
  Mat_Product    *product = mat->product;
  PetscErrorCode ierr;
  PetscBool      isdense = PETSC_FALSE,Biscusp = PETSC_FALSE,Ciscusp = PETSC_TRUE;

  PetscFunctionBegin;
  MatCheckProduct(mat,1);
  ierr = PetscObjectBaseTypeCompare((PetscObject)product->B,MATSEQDENSE,&isdense);CHKERRQ(ierr);
  if (!product->A->boundtocpu && !product->B->boundtocpu) {
    ierr = PetscObjectTypeCompare((PetscObject)product->B,MATSEQAIJCUSPARSE,&Biscusp);CHKERRQ(ierr);
  }
  if (product->type == MATPRODUCT_ABC) {
    Ciscusp = PETSC_FALSE;
    if (!product->C->boundtocpu) {
      ierr = PetscObjectTypeCompare((PetscObject)product->C,MATSEQAIJCUSPARSE,&Ciscusp);CHKERRQ(ierr);
    }
  }
  if (isdense) {
    switch (product->type) {
    case MATPRODUCT_AB:
    case MATPRODUCT_AtB:
    case MATPRODUCT_ABt:
    case MATPRODUCT_PtAP:
    case MATPRODUCT_RARt:
     if (product->A->boundtocpu) {
        ierr = MatProductSetFromOptions_SeqAIJ_SeqDense(mat);CHKERRQ(ierr);
      } else {
        mat->ops->productsymbolic = MatProductSymbolic_SeqAIJCUSPARSE_SeqDENSECUDA;
      }
      break;
    case MATPRODUCT_ABC:
      mat->ops->productsymbolic = MatProductSymbolic_ABC_Basic;
      break;
    default:
      break;
    }
  } else if (Biscusp && Ciscusp) {
    switch (product->type) {
    case MATPRODUCT_AB:
    case MATPRODUCT_AtB:
    case MATPRODUCT_ABt:
      mat->ops->productsymbolic = MatProductSymbolic_SeqAIJCUSPARSE_SeqAIJCUSPARSE;
      break;
    case MATPRODUCT_PtAP:
    case MATPRODUCT_RARt:
    case MATPRODUCT_ABC:
      mat->ops->productsymbolic = MatProductSymbolic_ABC_Basic;
      break;
    default:
      break;
    }
  } else { /* fallback for AIJ */
    ierr = MatProductSetFromOptions_SeqAIJ(mat);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMult_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAddKernel_SeqAIJCUSPARSE(A,xx,NULL,yy,PETSC_FALSE,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMultAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy, Vec zz)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAddKernel_SeqAIJCUSPARSE(A,xx,yy,zz,PETSC_FALSE,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMultHermitianTranspose_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAddKernel_SeqAIJCUSPARSE(A,xx,NULL,yy,PETSC_TRUE,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMultHermitianTransposeAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAddKernel_SeqAIJCUSPARSE(A,xx,yy,zz,PETSC_TRUE,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMultTranspose_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAddKernel_SeqAIJCUSPARSE(A,xx,NULL,yy,PETSC_TRUE,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

__global__ static void ScatterAdd(PetscInt n, PetscInt *idx,const PetscScalar *x,PetscScalar *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[idx[i]] += x[i];
}

/* z = op(A) x + y. If trans & !herm, op = ^T; if trans & herm, op = ^H; if !trans, op = no-op */
static PetscErrorCode MatMultAddKernel_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz,PetscBool trans,PetscBool herm)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct;
  PetscScalar                  *xarray,*zarray,*dptr,*beta,*xptr;
  PetscErrorCode               ierr;
  hipError_t                  cerr;
  hipsparseStatus_t             stat;
  hipsparseOperation_t          opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  PetscBool                    compressed;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  PetscInt                     nx,ny;
#endif

  PetscFunctionBegin;
  if (herm && !trans) SETERRQ(PetscObjectComm((PetscObject)A),PETSC_ERR_GPU,"Hermitian and not transpose not supported");
  if (!a->nonzerorowcnt) {
    if (!yy) {ierr = VecSet_SeqCUDA(zz,0);CHKERRQ(ierr);}
    else {ierr = VecCopy_SeqCUDA(yy,zz);CHKERRQ(ierr);}
    PetscFunctionReturn(0);
  }
  /* The line below is necessary due to the operations that modify the matrix on the CPU (axpy, scale, etc) */
  ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  if (!trans) {
    matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
    if (!matstruct) SETERRQ(PetscObjectComm((PetscObject)A),PETSC_ERR_GPU,"SeqAIJCUSPARSE does not have a 'mat' (need to fix)");
  } else {
    if (herm || !A->form_explicit_transpose) {
      opA = herm ? HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE : HIPSPARSE_OPERATION_TRANSPOSE;
      matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
    } else {
      if (!cusparsestruct->matTranspose) {ierr = MatSeqAIJCUSPARSEFormExplicitTransposeForMult(A);CHKERRQ(ierr);}
      matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
    }
  }
  /* Does the matrix use compressed rows (i.e., drop zero rows)? */
  compressed = matstruct->cprowIndices ? PETSC_TRUE : PETSC_FALSE;

  try {
    ierr = VecCUDAGetArrayRead(xx,(const PetscScalar**)&xarray);CHKERRQ(ierr);
    if (yy == zz) {ierr = VecCUDAGetArray(zz,&zarray);CHKERRQ(ierr);} /* read & write zz, so need to get uptodate zarray on GPU */
    else {ierr = VecCUDAGetArrayWrite(zz,&zarray);CHKERRQ(ierr);} /* write zz, so no need to init zarray on GPU */

    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    if (opA == HIPSPARSE_OPERATION_NON_TRANSPOSE) {
      /* z = A x + beta y.
         If A is compressed (with less rows), then Ax is shorter than the full z, so we need a work vector to store Ax.
         When A is non-compressed, and z = y, we can set beta=1 to compute y = Ax + y in one call.
      */
      xptr = xarray;
      dptr = compressed ? cusparsestruct->workVector->data().get() : zarray;
      beta = (yy == zz && !compressed) ? matstruct->beta_one : matstruct->beta_zero;
     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      /* Get length of x, y for y=Ax. ny might be shorter than the work vector's allocated length, since the work vector is
          allocated to accommodate different uses. So we get the length info directly from mat.
       */
      if (cusparsestruct->format == MAT_CUSPARSE_CSR) {
        CsrMatrix *mat = (CsrMatrix*)matstruct->mat;
        nx = mat->num_cols;
        ny = mat->num_rows;
      }
     #endif
    } else {
      /* z = A^T x + beta y
         If A is compressed, then we need a work vector as the shorter version of x to compute A^T x.
         Note A^Tx is of full length, so we set beta to 1.0 if y exists.
       */
      xptr = compressed ? cusparsestruct->workVector->data().get() : xarray;
      dptr = zarray;
      beta = yy ? matstruct->beta_one : matstruct->beta_zero;
      if (compressed) { /* Scatter x to work vector */
        thrust::device_ptr<PetscScalar> xarr = thrust::device_pointer_cast(xarray);
        thrust::for_each(thrust::cuda::par.on(PetscDefaultCudaStream),thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(xarr, matstruct->cprowIndices->begin()))),
                         thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(xarr, matstruct->cprowIndices->begin()))) + matstruct->cprowIndices->size(),
                         VecCUDAEqualsReverse());
      }
     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      if (cusparsestruct->format == MAT_CUSPARSE_CSR) {
        CsrMatrix *mat = (CsrMatrix*)matstruct->mat;
        nx = mat->num_rows;
        ny = mat->num_cols;
      }
     #endif
    }

    /* csr_spmv does y = alpha op(A) x + beta y */
    if (cusparsestruct->format == MAT_CUSPARSE_CSR) {
     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      if (opA < 0 || opA > 2) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"cuSPARSE ABI on hipsparseOperation_t has changed and PETSc has not been updated accordingly");
      if (!matstruct->cuSpMV[opA].initialized) { /* built on demand */
        stat = hipsparseCreateDnVec(&matstruct->cuSpMV[opA].vecXDescr,nx,xptr,cusparse_scalartype);CHKERRCUSPARSE(stat);
        stat = hipsparseCreateDnVec(&matstruct->cuSpMV[opA].vecYDescr,ny,dptr,cusparse_scalartype);CHKERRCUSPARSE(stat);
        stat = hipsparseSpMV_bufferSize(cusparsestruct->handle, opA, matstruct->alpha_one,
                                matstruct->matDescr,
                                matstruct->cuSpMV[opA].vecXDescr, beta,
                                matstruct->cuSpMV[opA].vecYDescr,
                                cusparse_scalartype,
                                cusparsestruct->spmvAlg,
                                &matstruct->cuSpMV[opA].spmvBufferSize);CHKERRCUSPARSE(stat);
        cerr = hipMalloc(&matstruct->cuSpMV[opA].spmvBuffer,matstruct->cuSpMV[opA].spmvBufferSize);CHKERRCUDA(cerr);

        matstruct->cuSpMV[opA].initialized = PETSC_TRUE;
      } else {
        /* x, y's value pointers might change between calls, but their shape is kept, so we just update pointers */
        stat = hipsparseDnVecSetValues(matstruct->cuSpMV[opA].vecXDescr,xptr);CHKERRCUSPARSE(stat);
        stat = hipsparseDnVecSetValues(matstruct->cuSpMV[opA].vecYDescr,dptr);CHKERRCUSPARSE(stat);
      }

      stat = hipsparseSpMV(cusparsestruct->handle, opA,
                               matstruct->alpha_one,
                               matstruct->matDescr, /* built in MatSeqAIJCUSPARSECopyToGPU() or MatSeqAIJCUSPARSEFormExplicitTransposeForMult() */
                               matstruct->cuSpMV[opA].vecXDescr,
                               beta,
                               matstruct->cuSpMV[opA].vecYDescr,
                               cusparse_scalartype,
                               cusparsestruct->spmvAlg,
                               matstruct->cuSpMV[opA].spmvBuffer);CHKERRCUSPARSE(stat);
     #else
      CsrMatrix *mat = (CsrMatrix*)matstruct->mat;
      stat = cusparse_csr_spmv(cusparsestruct->handle, opA,
                               mat->num_rows, mat->num_cols,
                               mat->num_entries, matstruct->alpha_one, matstruct->descr,
                               mat->values->data().get(), mat->row_offsets->data().get(),
                               mat->column_indices->data().get(), xptr, beta,
                               dptr);CHKERRCUSPARSE(stat);
     #endif
    } else {
      if (cusparsestruct->nrows) {
       #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
        SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"MAT_CUSPARSE_ELL and MAT_CUSPARSE_HYB are not supported since CUDA-11.0");
       #else
        hipsparseHybMat_t hybMat = (hipsparseHybMat_t)matstruct->mat;
        stat = cusparse_hyb_spmv(cusparsestruct->handle, opA,
                                 matstruct->alpha_one, matstruct->descr, hybMat,
                                 xptr, beta,
                                 dptr);CHKERRCUSPARSE(stat);
       #endif
      }
    }
    cerr = WaitForCUDA();CHKERRCUDA(cerr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);

    if (opA == HIPSPARSE_OPERATION_NON_TRANSPOSE) {
      if (yy) { /* MatMultAdd: zz = A*xx + yy */
        if (compressed) { /* A is compressed. We first copy yy to zz, then ScatterAdd the work vector to zz */
          ierr = VecCopy_SeqCUDA(yy,zz);CHKERRQ(ierr); /* zz = yy */
        } else if (zz != yy) { /* A is not compressed. zz already contains A*xx, and we just need to add yy */
          ierr = VecAXPY_SeqCUDA(zz,1.0,yy);CHKERRQ(ierr); /* zz += yy */
        }
      } else if (compressed) { /* MatMult: zz = A*xx. A is compressed, so we zero zz first, then ScatterAdd the work vector to zz */
        ierr = VecSet_SeqCUDA(zz,0);CHKERRQ(ierr);
      }

      /* ScatterAdd the result from work vector into the full vector when A is compressed */
      if (compressed) {
        ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
        /* I wanted to make this for_each asynchronous but failed. thrust::async::for_each() returns an event (internally registerred)
           and in the destructor of the scope, it will call hipStreamSynchronize() on this stream. One has to store all events to
           prevent that. So I just add a ScatterAdd kernel.
         */
       #if 0
        thrust::device_ptr<PetscScalar> zptr = thrust::device_pointer_cast(zarray);
        thrust::async::for_each(thrust::cuda::par.on(cusparsestruct->stream),
                         thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(zptr, matstruct->cprowIndices->begin()))),
                         thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(zptr, matstruct->cprowIndices->begin()))) + matstruct->cprowIndices->size(),
                         VecCUDAPlusEquals());
       #else
        PetscInt n = matstruct->cprowIndices->size();
        ScatterAdd<<<(n+255)/256,256,0,PetscDefaultCudaStream>>>(n,matstruct->cprowIndices->data().get(),cusparsestruct->workVector->data().get(),zarray);
       #endif
        cerr = WaitForCUDA();CHKERRCUDA(cerr);
        ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
      }
    } else {
      if (yy && yy != zz) {
        ierr = VecAXPY_SeqCUDA(zz,1.0,yy);CHKERRQ(ierr); /* zz += yy */
      }
    }
    ierr = VecCUDARestoreArrayRead(xx,(const PetscScalar**)&xarray);CHKERRQ(ierr);
    if (yy == zz) {ierr = VecCUDARestoreArray(zz,&zarray);CHKERRQ(ierr);}
    else {ierr = VecCUDARestoreArrayWrite(zz,&zarray);CHKERRQ(ierr);}
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  if (yy) {
    ierr = PetscLogGpuFlops(2.0*a->nz);CHKERRQ(ierr);
  } else {
    ierr = PetscLogGpuFlops(2.0*a->nz-a->nonzerorowcnt);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMultTransposeAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAddKernel_SeqAIJCUSPARSE(A,xx,yy,zz,PETSC_TRUE,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatAssemblyEnd_SeqAIJCUSPARSE(Mat A,MatAssemblyType mode)
{
  PetscErrorCode              ierr;
  PetscSplitCSRDataStructure  *d_mat = NULL;
  PetscFunctionBegin;
  if (A->factortype == MAT_FACTOR_NONE) {
    d_mat = ((Mat_SeqAIJCUSPARSE*)A->spptr)->deviceMat;
  }
  ierr = MatAssemblyEnd_SeqAIJ(A,mode);CHKERRQ(ierr); // this does very little if assembled on GPU - call it?
  if (mode == MAT_FLUSH_ASSEMBLY || A->boundtocpu) PetscFunctionReturn(0);
  if (d_mat) {
    A->offloadmask = PETSC_OFFLOAD_GPU;
  }

  PetscFunctionReturn(0);
}

/* --------------------------------------------------------------------------------*/
/*@
   MatCreateSeqAIJCUSPARSE - Creates a sparse matrix in AIJ (compressed row) format
   (the default parallel PETSc format). This matrix will ultimately pushed down
   to NVidia GPUs and use the CUSPARSE library for calculations. For good matrix
   assembly performance the user should preallocate the matrix storage by setting
   the parameter nz (or the array nnz).  By setting these parameters accurately,
   performance during matrix assembly can be increased by more than a factor of 50.

   Collective

   Input Parameters:
+  comm - MPI communicator, set to PETSC_COMM_SELF
.  m - number of rows
.  n - number of columns
.  nz - number of nonzeros per row (same for all rows)
-  nnz - array containing the number of nonzeros in the various rows
         (possibly different for each row) or NULL

   Output Parameter:
.  A - the matrix

   It is recommended that one use the MatCreate(), MatSetType() and/or MatSetFromOptions(),
   MatXXXXSetPreallocation() paradgm instead of this routine directly.
   [MatXXXXSetPreallocation() is, for example, MatSeqAIJSetPreallocation]

   Notes:
   If nnz is given then nz is ignored

   The AIJ format (also called the Yale sparse matrix format or
   compressed row storage), is fully compatible with standard Fortran 77
   storage.  That is, the stored row and column indices can begin at
   either one (as in Fortran) or zero.  See the users' manual for details.

   Specify the preallocated storage with either nz or nnz (not both).
   Set nz=PETSC_DEFAULT and nnz=NULL for PETSc to control dynamic memory
   allocation.  For large problems you MUST preallocate memory or you
   will get TERRIBLE performance, see the users' manual chapter on matrices.

   By default, this format uses inodes (identical nodes) when possible, to
   improve numerical efficiency of matrix-vector products and solves. We
   search for consecutive rows with the same nonzero structure, thereby
   reusing matrix information to achieve increased efficiency.

   Level: intermediate

.seealso: MatCreate(), MatCreateAIJ(), MatSetValues(), MatSeqAIJSetColumnIndices(), MatCreateSeqAIJWithArrays(), MatCreateAIJ(), MATSEQAIJCUSPARSE, MATAIJCUSPARSE
@*/
PetscErrorCode  MatCreateSeqAIJCUSPARSE(MPI_Comm comm,PetscInt m,PetscInt n,PetscInt nz,const PetscInt nnz[],Mat *A)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate(comm,A);CHKERRQ(ierr);
  ierr = MatSetSizes(*A,m,n,m,n);CHKERRQ(ierr);
  ierr = MatSetType(*A,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
  ierr = MatSeqAIJSetPreallocation_SeqAIJ(*A,nz,(PetscInt*)nnz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDestroy_SeqAIJCUSPARSE(Mat A)
{
  PetscErrorCode              ierr;
  PetscSplitCSRDataStructure  *d_mat = NULL;

  PetscFunctionBegin;
  if (A->factortype == MAT_FACTOR_NONE) {
    d_mat = ((Mat_SeqAIJCUSPARSE*)A->spptr)->deviceMat;
    ((Mat_SeqAIJCUSPARSE*)A->spptr)->deviceMat = NULL;
    ierr = MatSeqAIJCUSPARSE_Destroy((Mat_SeqAIJCUSPARSE**)&A->spptr);CHKERRQ(ierr);
  } else {
    ierr = MatSeqAIJCUSPARSETriFactors_Destroy((Mat_SeqAIJCUSPARSETriFactors**)&A->spptr);CHKERRQ(ierr);
  }
  if (d_mat) {
    Mat_SeqAIJ                 *a = (Mat_SeqAIJ*)A->data;
    hipError_t                err;
    PetscSplitCSRDataStructure h_mat;
    ierr = PetscInfo(A,"Have device matrix\n");CHKERRQ(ierr);
    err = hipMemcpy( &h_mat, d_mat, sizeof(PetscSplitCSRDataStructure), hipMemcpyDeviceToHost);CHKERRCUDA(err);
    if (a->compressedrow.use) {
      err = hipFree(h_mat.diag.i);CHKERRCUDA(err);
    }
    err = hipFree(d_mat);CHKERRCUDA(err);
  }
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatSeqAIJCopySubArray_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatCUSPARSESetFormat_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqdensecuda_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqdense_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqaijcusparse_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatFactorGetSolverType_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatSetPreallocationCOO_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatSetValuesCOO_C",NULL);CHKERRQ(ierr);
  ierr = MatDestroy_SeqAIJ(A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatConvert_SeqAIJ_SeqAIJCUSPARSE(Mat,MatType,MatReuse,Mat*);
static PetscErrorCode MatBindToCPU_SeqAIJCUSPARSE(Mat,PetscBool);
static PetscErrorCode MatDuplicate_SeqAIJCUSPARSE(Mat A,MatDuplicateOption cpvalues,Mat *B)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatDuplicate_SeqAIJ(A,cpvalues,B);CHKERRQ(ierr);
  ierr = MatConvert_SeqAIJ_SeqAIJCUSPARSE(*B,MATSEQAIJCUSPARSE,MAT_INPLACE_MATRIX,B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatAXPY_SeqAIJCUSPARSE(Mat Y,PetscScalar a,Mat X,MatStructure str)
{
  PetscErrorCode     ierr;
  Mat_SeqAIJ         *x = (Mat_SeqAIJ*)X->data,*y = (Mat_SeqAIJ*)Y->data;
  Mat_SeqAIJCUSPARSE *cy;
  Mat_SeqAIJCUSPARSE *cx;
  PetscScalar        *ay;
  const PetscScalar  *ax;
  CsrMatrix          *csry,*csrx;
  hipError_t        cerr;

  PetscFunctionBegin;
  cy = (Mat_SeqAIJCUSPARSE*)Y->spptr;
  cx = (Mat_SeqAIJCUSPARSE*)X->spptr;
  if (X->ops->axpy != Y->ops->axpy) {
    ierr = MatSeqAIJCUSPARSEInvalidateTranspose(Y,PETSC_FALSE);CHKERRQ(ierr);
    ierr = MatAXPY_SeqAIJ(Y,a,X,str);CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }
  /* if we are here, it means both matrices are bound to GPU */
  ierr = MatSeqAIJCUSPARSECopyToGPU(Y);CHKERRQ(ierr);
  ierr = MatSeqAIJCUSPARSECopyToGPU(X);CHKERRQ(ierr);
  if (cy->format != MAT_CUSPARSE_CSR) SETERRQ(PetscObjectComm((PetscObject)Y),PETSC_ERR_GPU,"only MAT_CUSPARSE_CSR supported");
  if (cx->format != MAT_CUSPARSE_CSR) SETERRQ(PetscObjectComm((PetscObject)X),PETSC_ERR_GPU,"only MAT_CUSPARSE_CSR supported");
  csry = (CsrMatrix*)cy->mat->mat;
  csrx = (CsrMatrix*)cx->mat->mat;
  /* see if we can turn this into a cublas axpy */
  if (str != SAME_NONZERO_PATTERN && x->nz == y->nz && !x->compressedrow.use && !y->compressedrow.use) {
    bool eq = thrust::equal(thrust::device,csry->row_offsets->begin(),csry->row_offsets->end(),csrx->row_offsets->begin());
    if (eq) {
      eq = thrust::equal(thrust::device,csry->column_indices->begin(),csry->column_indices->end(),csrx->column_indices->begin());
    }
    if (eq) str = SAME_NONZERO_PATTERN;
  }
  /* spgeam is buggy with one column */
  if (Y->cmap->n == 1 && str != SAME_NONZERO_PATTERN) str = DIFFERENT_NONZERO_PATTERN;

  if (str == SUBSET_NONZERO_PATTERN) {
    hipsparseStatus_t stat;
    PetscScalar      b = 1.0;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    size_t           bufferSize;
    void             *buffer;
#endif

    ierr = MatSeqAIJCUSPARSEGetArrayRead(X,&ax);CHKERRQ(ierr);
    ierr = MatSeqAIJCUSPARSEGetArray(Y,&ay);CHKERRQ(ierr);
    stat = hipsparseSetPointerMode(cy->handle, HIPSPARSE_POINTER_MODE_HOST);CHKERRCUSPARSE(stat);
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    stat = cusparse_csr_spgeam_bufferSize(cy->handle,Y->rmap->n,Y->cmap->n,
                                          &a,cx->mat->descr,x->nz,ax,csrx->row_offsets->data().get(),csrx->column_indices->data().get(),
                                          &b,cy->mat->descr,y->nz,ay,csry->row_offsets->data().get(),csry->column_indices->data().get(),
                                             cy->mat->descr,      ay,csry->row_offsets->data().get(),csry->column_indices->data().get(),&bufferSize);CHKERRCUSPARSE(stat);
    cerr = hipMalloc(&buffer,bufferSize);CHKERRCUDA(cerr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    stat = cusparse_csr_spgeam(cy->handle,Y->rmap->n,Y->cmap->n,
                               &a,cx->mat->descr,x->nz,ax,csrx->row_offsets->data().get(),csrx->column_indices->data().get(),
                               &b,cy->mat->descr,y->nz,ay,csry->row_offsets->data().get(),csry->column_indices->data().get(),
                                  cy->mat->descr,      ay,csry->row_offsets->data().get(),csry->column_indices->data().get(),buffer);CHKERRCUSPARSE(stat);
    cerr = WaitForCUDA();CHKERRCUDA(cerr);
    ierr = PetscLogGpuFlops(x->nz + y->nz);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    cerr = hipFree(buffer);CHKERRCUDA(cerr);
#else
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    stat = cusparse_csr_spgeam(cy->handle,Y->rmap->n,Y->cmap->n,
                               &a,cx->mat->descr,x->nz,ax,csrx->row_offsets->data().get(),csrx->column_indices->data().get(),
                               &b,cy->mat->descr,y->nz,ay,csry->row_offsets->data().get(),csry->column_indices->data().get(),
                                  cy->mat->descr,      ay,csry->row_offsets->data().get(),csry->column_indices->data().get());CHKERRCUSPARSE(stat);
    cerr = WaitForCUDA();CHKERRCUDA(cerr);
    ierr = PetscLogGpuFlops(x->nz + y->nz);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
#endif
    stat = hipsparseSetPointerMode(cy->handle, HIPSPARSE_POINTER_MODE_DEVICE);CHKERRCUSPARSE(stat);
    ierr = MatSeqAIJCUSPARSERestoreArrayRead(X,&ax);CHKERRQ(ierr);
    ierr = MatSeqAIJCUSPARSERestoreArray(Y,&ay);CHKERRQ(ierr);
    ierr = MatSeqAIJInvalidateDiagonal(Y);CHKERRQ(ierr);
  } else if (str == SAME_NONZERO_PATTERN) {
    hipblasHandle_t cublasv2handle;
    hipblasStatus_t berr;
    PetscBLASInt   one = 1, bnz = 1;

    ierr = MatSeqAIJCUSPARSEGetArrayRead(X,&ax);CHKERRQ(ierr);
    ierr = MatSeqAIJCUSPARSEGetArray(Y,&ay);CHKERRQ(ierr);
    ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
    ierr = PetscBLASIntCast(x->nz,&bnz);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    berr = cublasXaxpy(cublasv2handle,bnz,&a,ax,one,ay,one);CHKERRCUBLAS(berr);
    cerr = WaitForCUDA();CHKERRCUDA(cerr);
    ierr = PetscLogGpuFlops(2.0*bnz);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = MatSeqAIJCUSPARSERestoreArrayRead(X,&ax);CHKERRQ(ierr);
    ierr = MatSeqAIJCUSPARSERestoreArray(Y,&ay);CHKERRQ(ierr);
    ierr = MatSeqAIJInvalidateDiagonal(Y);CHKERRQ(ierr);
  } else {
    ierr = MatSeqAIJCUSPARSEInvalidateTranspose(Y,PETSC_FALSE);CHKERRQ(ierr);
    ierr = MatAXPY_SeqAIJ(Y,a,X,str);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatScale_SeqAIJCUSPARSE(Mat Y,PetscScalar a)
{
  PetscErrorCode ierr;
  Mat_SeqAIJ     *y = (Mat_SeqAIJ*)Y->data;
  PetscScalar    *ay;
  hipError_t    cerr;
  hipblasHandle_t cublasv2handle;
  hipblasStatus_t berr;
  PetscBLASInt   one = 1, bnz = 1;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSEGetArray(Y,&ay);CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(y->nz,&bnz);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  berr = cublasXscal(cublasv2handle,bnz,&a,ay,one);CHKERRCUBLAS(berr);
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuFlops(bnz);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = MatSeqAIJCUSPARSERestoreArray(Y,&ay);CHKERRQ(ierr);
  ierr = MatSeqAIJInvalidateDiagonal(Y);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatZeroEntries_SeqAIJCUSPARSE(Mat A)
{
  PetscErrorCode             ierr;
  PetscBool                  both = PETSC_FALSE;
  Mat_SeqAIJ                 *a = (Mat_SeqAIJ*)A->data;

  PetscFunctionBegin;
  if (A->factortype == MAT_FACTOR_NONE) {
    Mat_SeqAIJCUSPARSE *spptr = (Mat_SeqAIJCUSPARSE*)A->spptr;
    if (spptr->mat) {
      CsrMatrix* matrix = (CsrMatrix*)spptr->mat->mat;
      if (matrix->values) {
        both = PETSC_TRUE;
        thrust::fill(thrust::device,matrix->values->begin(),matrix->values->end(),0.);
      }
    }
    if (spptr->matTranspose) {
      CsrMatrix* matrix = (CsrMatrix*)spptr->matTranspose->mat;
      if (matrix->values) {
        thrust::fill(thrust::device,matrix->values->begin(),matrix->values->end(),0.);
      }
    }
  }
  //ierr = MatZeroEntries_SeqAIJ(A);CHKERRQ(ierr);
  ierr = PetscArrayzero(a->a,a->i[A->rmap->n]);CHKERRQ(ierr);
  ierr = MatSeqAIJInvalidateDiagonal(A);CHKERRQ(ierr);
  if (both) A->offloadmask = PETSC_OFFLOAD_BOTH;
  else A->offloadmask = PETSC_OFFLOAD_CPU;

  PetscFunctionReturn(0);
}

static PetscErrorCode MatBindToCPU_SeqAIJCUSPARSE(Mat A,PetscBool flg)
{
  Mat_SeqAIJ     *a = (Mat_SeqAIJ*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (A->factortype != MAT_FACTOR_NONE) PetscFunctionReturn(0);
  if (flg) {
    ierr = MatSeqAIJCUSPARSECopyFromGPU(A);CHKERRQ(ierr);

    A->ops->scale                     = MatScale_SeqAIJ;
    A->ops->axpy                      = MatAXPY_SeqAIJ;
    A->ops->zeroentries               = MatZeroEntries_SeqAIJ;
    A->ops->mult                      = MatMult_SeqAIJ;
    A->ops->multadd                   = MatMultAdd_SeqAIJ;
    A->ops->multtranspose             = MatMultTranspose_SeqAIJ;
    A->ops->multtransposeadd          = MatMultTransposeAdd_SeqAIJ;
    A->ops->multhermitiantranspose    = NULL;
    A->ops->multhermitiantransposeadd = NULL;
    A->ops->productsetfromoptions     = MatProductSetFromOptions_SeqAIJ;
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatSeqAIJCopySubArray_C",NULL);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqdensecuda_C",NULL);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqdense_C",NULL);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatSetPreallocationCOO_C",NULL);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatSetValuesCOO_C",NULL);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatSeqAIJGetArray_C",MatSeqAIJGetArray_SeqAIJ);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqaijcusparse_C",NULL);CHKERRQ(ierr);
  } else {
    A->ops->scale                     = MatScale_SeqAIJCUSPARSE;
    A->ops->axpy                      = MatAXPY_SeqAIJCUSPARSE;
    A->ops->zeroentries               = MatZeroEntries_SeqAIJCUSPARSE;
    A->ops->mult                      = MatMult_SeqAIJCUSPARSE;
    A->ops->multadd                   = MatMultAdd_SeqAIJCUSPARSE;
    A->ops->multtranspose             = MatMultTranspose_SeqAIJCUSPARSE;
    A->ops->multtransposeadd          = MatMultTransposeAdd_SeqAIJCUSPARSE;
    A->ops->multhermitiantranspose    = MatMultHermitianTranspose_SeqAIJCUSPARSE;
    A->ops->multhermitiantransposeadd = MatMultHermitianTransposeAdd_SeqAIJCUSPARSE;
    A->ops->productsetfromoptions     = MatProductSetFromOptions_SeqAIJCUSPARSE;
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatSeqAIJCopySubArray_C",MatSeqAIJCopySubArray_SeqAIJCUSPARSE);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqdensecuda_C",MatProductSetFromOptions_SeqAIJCUSPARSE);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqdense_C",MatProductSetFromOptions_SeqAIJCUSPARSE);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatSetPreallocationCOO_C",MatSetPreallocationCOO_SeqAIJCUSPARSE);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatSetValuesCOO_C",MatSetValuesCOO_SeqAIJCUSPARSE);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatSeqAIJGetArray_C",MatSeqAIJGetArray_SeqAIJCUSPARSE);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqaijcusparse_C",MatProductSetFromOptions_SeqAIJCUSPARSE);CHKERRQ(ierr);
  }
  A->boundtocpu = flg;
  a->inode.use = flg;
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatConvert_SeqAIJ_SeqAIJCUSPARSE(Mat A, MatType mtype, MatReuse reuse, Mat* newmat)
{
  PetscErrorCode   ierr;
  hipsparseStatus_t stat;
  Mat              B;

  PetscFunctionBegin;
  ierr = PetscCUDAInitializeCheck();CHKERRQ(ierr); /* first use of CUSPARSE may be via MatConvert */
  if (reuse == MAT_INITIAL_MATRIX) {
    ierr = MatDuplicate(A,MAT_COPY_VALUES,newmat);CHKERRQ(ierr);
  } else if (reuse == MAT_REUSE_MATRIX) {
    ierr = MatCopy(A,*newmat,SAME_NONZERO_PATTERN);CHKERRQ(ierr);
  }
  B = *newmat;

  ierr = PetscFree(B->defaultvectype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(VECCUDA,&B->defaultvectype);CHKERRQ(ierr);

  if (reuse != MAT_REUSE_MATRIX && !B->spptr) {
    if (B->factortype == MAT_FACTOR_NONE) {
      Mat_SeqAIJCUSPARSE *spptr;
      ierr = PetscNew(&spptr);CHKERRQ(ierr);
      stat = hipsparseCreate(&spptr->handle);CHKERRCUSPARSE(stat);
      stat = hipsparseSetStream(spptr->handle,PetscDefaultCudaStream);CHKERRCUSPARSE(stat);
      spptr->format     = MAT_CUSPARSE_CSR;
     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      spptr->spmvAlg    = HIPSPARSE_CSRMV_ALG1;    /* default, since we only support csr */
      spptr->spmmAlg    = HIPSPARSE_SPMM_CSR_ALG1; /* default, only support column-major dense matrix B */
      spptr->csr2cscAlg = HIPSPARSE_CSR2CSC_ALG1;
     #endif
      B->spptr = spptr;
    } else {
      Mat_SeqAIJCUSPARSETriFactors *spptr;

      ierr = PetscNew(&spptr);CHKERRQ(ierr);
      stat = hipsparseCreate(&spptr->handle);CHKERRCUSPARSE(stat);
      stat = hipsparseSetStream(spptr->handle,PetscDefaultCudaStream);CHKERRCUSPARSE(stat);
      B->spptr = spptr;
    }
    B->offloadmask = PETSC_OFFLOAD_UNALLOCATED;
  }
  B->ops->assemblyend    = MatAssemblyEnd_SeqAIJCUSPARSE;
  B->ops->destroy        = MatDestroy_SeqAIJCUSPARSE;
  B->ops->setoption      = MatSetOption_SeqAIJCUSPARSE;
  B->ops->setfromoptions = MatSetFromOptions_SeqAIJCUSPARSE;
  B->ops->bindtocpu      = MatBindToCPU_SeqAIJCUSPARSE;
  B->ops->duplicate      = MatDuplicate_SeqAIJCUSPARSE;

  ierr = MatBindToCPU_SeqAIJCUSPARSE(B,PETSC_FALSE);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatCUSPARSESetFormat_C",MatCUSPARSESetFormat_SeqAIJCUSPARSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PETSC_EXTERN PetscErrorCode MatCreate_SeqAIJCUSPARSE(Mat B)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate_SeqAIJ(B);CHKERRQ(ierr);
  ierr = MatConvert_SeqAIJ_SeqAIJCUSPARSE(B,MATSEQAIJCUSPARSE,MAT_INPLACE_MATRIX,&B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*MC
   MATSEQAIJCUSPARSE - MATAIJCUSPARSE = "(seq)aijcusparse" - A matrix type to be used for sparse matrices.

   A matrix type type whose data resides on Nvidia GPUs. These matrices can be in either
   CSR, ELL, or Hybrid format. The ELL and HYB formats require CUDA 4.2 or later.
   All matrix calculations are performed on Nvidia GPUs using the CUSPARSE library.

   Options Database Keys:
+  -mat_type aijcusparse - sets the matrix type to "seqaijcusparse" during a call to MatSetFromOptions()
.  -mat_cusparse_storage_format csr - sets the storage format of matrices (for MatMult and factors in MatSolve) during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).
-  -mat_cusparse_mult_storage_format csr - sets the storage format of matrices (for MatMult) during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).

  Level: beginner

.seealso: MatCreateSeqAIJCUSPARSE(), MATAIJCUSPARSE, MatCreateAIJCUSPARSE(), MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M*/

PETSC_EXTERN PetscErrorCode MatGetFactor_seqaijcusparse_cusparse_band(Mat,MatFactorType,Mat*);

PETSC_EXTERN PetscErrorCode MatSolverTypeRegister_CUSPARSE(void)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSolverTypeRegister(MATSOLVERCUSPARSEBAND, MATSEQAIJ, MAT_FACTOR_LU,MatGetFactor_seqaijcusparse_cusparse_band);CHKERRQ(ierr);
  ierr = MatSolverTypeRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_LU,MatGetFactor_seqaijcusparse_cusparse);CHKERRQ(ierr);
  ierr = MatSolverTypeRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_CHOLESKY,MatGetFactor_seqaijcusparse_cusparse);CHKERRQ(ierr);
  ierr = MatSolverTypeRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_ILU,MatGetFactor_seqaijcusparse_cusparse);CHKERRQ(ierr);
  ierr = MatSolverTypeRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_ICC,MatGetFactor_seqaijcusparse_cusparse);CHKERRQ(ierr);

  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSE_Destroy(Mat_SeqAIJCUSPARSE **cusparsestruct)
{
  PetscErrorCode   ierr;
  hipsparseStatus_t stat;

  PetscFunctionBegin;
  if (*cusparsestruct) {
    ierr = MatSeqAIJCUSPARSEMultStruct_Destroy(&(*cusparsestruct)->mat,(*cusparsestruct)->format);CHKERRQ(ierr);
    ierr = MatSeqAIJCUSPARSEMultStruct_Destroy(&(*cusparsestruct)->matTranspose,(*cusparsestruct)->format);CHKERRQ(ierr);
    delete (*cusparsestruct)->workVector;
    delete (*cusparsestruct)->rowoffsets_gpu;
    delete (*cusparsestruct)->cooPerm;
    delete (*cusparsestruct)->cooPerm_a;
    delete (*cusparsestruct)->csr2csc_i;
    if ((*cusparsestruct)->handle) {stat = hipsparseDestroy((*cusparsestruct)->handle);CHKERRCUSPARSE(stat);}
    ierr = PetscFree(*cusparsestruct);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode CsrMatrix_Destroy(CsrMatrix **mat)
{
  PetscFunctionBegin;
  if (*mat) {
    delete (*mat)->values;
    delete (*mat)->column_indices;
    delete (*mat)->row_offsets;
    delete *mat;
    *mat = 0;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSETriFactorStruct **trifactor)
{
  hipsparseStatus_t stat;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (*trifactor) {
    if ((*trifactor)->descr) { stat = hipsparseDestroyMatDescr((*trifactor)->descr);CHKERRCUSPARSE(stat); }
    if ((*trifactor)->solveInfo) { stat = cusparse_destroy_analysis_info((*trifactor)->solveInfo);CHKERRCUSPARSE(stat); }
    ierr = CsrMatrix_Destroy(&(*trifactor)->csrMat);CHKERRQ(ierr);
    if ((*trifactor)->solveBuffer)   {hipError_t cerr = hipFree((*trifactor)->solveBuffer);CHKERRCUDA(cerr);}
    if ((*trifactor)->AA_h)   {hipError_t cerr = hipHostFree((*trifactor)->AA_h);CHKERRCUDA(cerr);}
   #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    if ((*trifactor)->csr2cscBuffer) {hipError_t cerr = hipFree((*trifactor)->csr2cscBuffer);CHKERRCUDA(cerr);}
   #endif
    ierr = PetscFree(*trifactor);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSEMultStruct **matstruct,MatCUSPARSEStorageFormat format)
{
  CsrMatrix        *mat;
  hipsparseStatus_t stat;
  hipError_t      err;

  PetscFunctionBegin;
  if (*matstruct) {
    if ((*matstruct)->mat) {
      if (format==MAT_CUSPARSE_ELL || format==MAT_CUSPARSE_HYB) {
       #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
        SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"MAT_CUSPARSE_ELL and MAT_CUSPARSE_HYB are not supported since CUDA-11.0");
       #else
        hipsparseHybMat_t hybMat = (hipsparseHybMat_t)(*matstruct)->mat;
        stat = hipsparseDestroyHybMat(hybMat);CHKERRCUSPARSE(stat);
       #endif
      } else {
        mat = (CsrMatrix*)(*matstruct)->mat;
        CsrMatrix_Destroy(&mat);
      }
    }
    if ((*matstruct)->descr) { stat = hipsparseDestroyMatDescr((*matstruct)->descr);CHKERRCUSPARSE(stat); }
    delete (*matstruct)->cprowIndices;
    if ((*matstruct)->alpha_one) { err=hipFree((*matstruct)->alpha_one);CHKERRCUDA(err); }
    if ((*matstruct)->beta_zero) { err=hipFree((*matstruct)->beta_zero);CHKERRCUDA(err); }
    if ((*matstruct)->beta_one)  { err=hipFree((*matstruct)->beta_one);CHKERRCUDA(err); }

   #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    Mat_SeqAIJCUSPARSEMultStruct *mdata = *matstruct;
    if (mdata->matDescr) {stat = hipsparseDestroySpMat(mdata->matDescr);CHKERRCUSPARSE(stat);}
    for (int i=0; i<3; i++) {
      if (mdata->cuSpMV[i].initialized) {
        err  = hipFree(mdata->cuSpMV[i].spmvBuffer);CHKERRCUDA(err);
        stat = hipsparseDestroyDnVec(mdata->cuSpMV[i].vecXDescr);CHKERRCUSPARSE(stat);
        stat = hipsparseDestroyDnVec(mdata->cuSpMV[i].vecYDescr);CHKERRCUSPARSE(stat);
      }
    }
   #endif
    delete *matstruct;
    *matstruct = NULL;
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqAIJCUSPARSETriFactors_Reset(Mat_SeqAIJCUSPARSETriFactors_p* trifactors)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (*trifactors) {
    ierr = MatSeqAIJCUSPARSEMultStruct_Destroy(&(*trifactors)->loTriFactorPtr);CHKERRQ(ierr);
    ierr = MatSeqAIJCUSPARSEMultStruct_Destroy(&(*trifactors)->upTriFactorPtr);CHKERRQ(ierr);
    ierr = MatSeqAIJCUSPARSEMultStruct_Destroy(&(*trifactors)->loTriFactorPtrTranspose);CHKERRQ(ierr);
    ierr = MatSeqAIJCUSPARSEMultStruct_Destroy(&(*trifactors)->upTriFactorPtrTranspose);CHKERRQ(ierr);
    delete (*trifactors)->rpermIndices;
    delete (*trifactors)->cpermIndices;
    delete (*trifactors)->workVector;
    (*trifactors)->rpermIndices = NULL;
    (*trifactors)->cpermIndices = NULL;
    (*trifactors)->workVector = NULL;
    if ((*trifactors)->a_band_d)   {hipError_t cerr = hipFree((*trifactors)->a_band_d);CHKERRCUDA(cerr);}
    if ((*trifactors)->i_band_d)   {hipError_t cerr = hipFree((*trifactors)->i_band_d);CHKERRCUDA(cerr);}
    (*trifactors)->init_dev_prop = PETSC_FALSE;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSETriFactors_Destroy(Mat_SeqAIJCUSPARSETriFactors** trifactors)
{
  PetscErrorCode   ierr;
  hipsparseHandle_t handle;
  hipsparseStatus_t stat;

  PetscFunctionBegin;
  if (*trifactors) {
    ierr = MatSeqAIJCUSPARSETriFactors_Reset(trifactors);CHKERRQ(ierr);
    if (handle = (*trifactors)->handle) {
      stat = hipsparseDestroy(handle);CHKERRCUSPARSE(stat);
    }
    ierr = PetscFree(*trifactors);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

struct IJCompare
{
  __host__ __device__
  inline bool operator() (const thrust::tuple<PetscInt, PetscInt> &t1, const thrust::tuple<PetscInt, PetscInt> &t2)
  {
    if (t1.get<0>() < t2.get<0>()) return true;
    if (t1.get<0>() == t2.get<0>()) return t1.get<1>() < t2.get<1>();
    return false;
  }
};

struct IJEqual
{
  __host__ __device__
  inline bool operator() (const thrust::tuple<PetscInt, PetscInt> &t1, const thrust::tuple<PetscInt, PetscInt> &t2)
  {
    if (t1.get<0>() != t2.get<0>() || t1.get<1>() != t2.get<1>()) return false;
    return true;
  }
};

struct IJDiff
{
  __host__ __device__
  inline PetscInt operator() (const PetscInt &t1, const PetscInt &t2)
  {
    return t1 == t2 ? 0 : 1;
  }
};

struct IJSum
{
  __host__ __device__
  inline PetscInt operator() (const PetscInt &t1, const PetscInt &t2)
  {
    return t1||t2;
  }
};

#include <thrust/iterator/discard_iterator.h>
PetscErrorCode MatSetValuesCOO_SeqAIJCUSPARSE(Mat A, const PetscScalar v[], InsertMode imode)
{
  Mat_SeqAIJCUSPARSE                    *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJ                            *a = (Mat_SeqAIJ*)A->data;
  THRUSTARRAY                           *cooPerm_v = NULL;
  thrust::device_ptr<const PetscScalar> d_v;
  CsrMatrix                             *matrix;
  PetscErrorCode                        ierr;
  hipError_t                           cerr;
  PetscInt                              n;

  PetscFunctionBegin;
  if (!cusp) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing CUSPARSE struct");
  if (!cusp->mat) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing CUSPARSE CsrMatrix");
  if (!cusp->cooPerm) {
    ierr = MatAssemblyBegin(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
    ierr = MatAssemblyEnd(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }
  matrix = (CsrMatrix*)cusp->mat->mat;
  if (!matrix->values) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing CUDA memory");
  if (!v) {
    if (imode == INSERT_VALUES) thrust::fill(thrust::device,matrix->values->begin(),matrix->values->end(),0.);
    goto finalize;
  }
  n = cusp->cooPerm->size();
  if (isCudaMem(v)) {
    d_v = thrust::device_pointer_cast(v);
  } else {
    cooPerm_v = new THRUSTARRAY(n);
    cooPerm_v->assign(v,v+n);
    d_v = cooPerm_v->data();
    ierr = PetscLogCpuToGpu(n*sizeof(PetscScalar));CHKERRQ(ierr);
  }
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (imode == ADD_VALUES) { /* ADD VALUES means add to existing ones */
    if (cusp->cooPerm_a) {
      THRUSTARRAY *cooPerm_w = new THRUSTARRAY(matrix->values->size());
      auto vbit = thrust::make_permutation_iterator(d_v,cusp->cooPerm->begin());
      thrust::reduce_by_key(cusp->cooPerm_a->begin(),cusp->cooPerm_a->end(),vbit,thrust::make_discard_iterator(),cooPerm_w->begin(),thrust::equal_to<PetscInt>(),thrust::plus<PetscScalar>());
      thrust::transform(cooPerm_w->begin(),cooPerm_w->end(),matrix->values->begin(),matrix->values->begin(),thrust::plus<PetscScalar>());
      delete cooPerm_w;
    } else {
      auto zibit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(d_v,cusp->cooPerm->begin()),
                                                                matrix->values->begin()));
      auto zieit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(d_v,cusp->cooPerm->end()),
                                                                matrix->values->end()));
      thrust::for_each(zibit,zieit,VecCUDAPlusEquals());
    }
  } else {
    if (cusp->cooPerm_a) { /* repeated entries in COO, with INSERT_VALUES -> reduce */
      auto vbit = thrust::make_permutation_iterator(d_v,cusp->cooPerm->begin());
      thrust::reduce_by_key(cusp->cooPerm_a->begin(),cusp->cooPerm_a->end(),vbit,thrust::make_discard_iterator(),matrix->values->begin(),thrust::equal_to<PetscInt>(),thrust::plus<PetscScalar>());
    } else {
      auto zibit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(d_v,cusp->cooPerm->begin()),
                                                                matrix->values->begin()));
      auto zieit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(d_v,cusp->cooPerm->end()),
                                                                matrix->values->end()));
      thrust::for_each(zibit,zieit,VecCUDAEquals());
    }
  }
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
finalize:
  delete cooPerm_v;
  A->offloadmask = PETSC_OFFLOAD_GPU;
  ierr = PetscObjectStateIncrease((PetscObject)A);CHKERRQ(ierr);
  /* shorter version of MatAssemblyEnd_SeqAIJ */
  ierr = PetscInfo3(A,"Matrix size: %D X %D; storage space: 0 unneeded,%D used\n",A->rmap->n,A->cmap->n,a->nz);CHKERRQ(ierr);
  ierr = PetscInfo(A,"Number of mallocs during MatSetValues() is 0\n");CHKERRQ(ierr);
  ierr = PetscInfo1(A,"Maximum nonzeros in any row is %D\n",a->rmax);CHKERRQ(ierr);
  a->reallocs         = 0;
  A->info.mallocs    += 0;
  A->info.nz_unneeded = 0;
  A->assembled = A->was_assembled = PETSC_TRUE;
  A->num_ass++;
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqAIJCUSPARSEInvalidateTranspose(Mat A, PetscBool destroy)
{
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  if (!cusp) PetscFunctionReturn(0);
  if (destroy) {
    ierr = MatSeqAIJCUSPARSEMultStruct_Destroy(&cusp->matTranspose,cusp->format);CHKERRQ(ierr);
    delete cusp->csr2csc_i;
    cusp->csr2csc_i = NULL;
  }
  A->transupdated = PETSC_FALSE;
  PetscFunctionReturn(0);
}

#include <thrust/binary_search.h>
PetscErrorCode MatSetPreallocationCOO_SeqAIJCUSPARSE(Mat A, PetscInt n, const PetscInt coo_i[], const PetscInt coo_j[])
{
  PetscErrorCode     ierr;
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJ         *a = (Mat_SeqAIJ*)A->data;
  PetscInt           cooPerm_n, nzr = 0;
  hipError_t        cerr;

  PetscFunctionBegin;
  ierr = PetscLayoutSetUp(A->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->cmap);CHKERRQ(ierr);
  cooPerm_n = cusp->cooPerm ? cusp->cooPerm->size() : 0;
  if (n != cooPerm_n) {
    delete cusp->cooPerm;
    delete cusp->cooPerm_a;
    cusp->cooPerm = NULL;
    cusp->cooPerm_a = NULL;
  }
  if (n) {
    THRUSTINTARRAY d_i(n);
    THRUSTINTARRAY d_j(n);
    THRUSTINTARRAY ii(A->rmap->n);

    if (!cusp->cooPerm)   { cusp->cooPerm   = new THRUSTINTARRAY(n); }
    if (!cusp->cooPerm_a) { cusp->cooPerm_a = new THRUSTINTARRAY(n); }

    ierr = PetscLogCpuToGpu(2.*n*sizeof(PetscInt));CHKERRQ(ierr);
    d_i.assign(coo_i,coo_i+n);
    d_j.assign(coo_j,coo_j+n);
    auto fkey = thrust::make_zip_iterator(thrust::make_tuple(d_i.begin(),d_j.begin()));
    auto ekey = thrust::make_zip_iterator(thrust::make_tuple(d_i.end(),d_j.end()));

    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    thrust::sequence(thrust::device, cusp->cooPerm->begin(), cusp->cooPerm->end(), 0);
    thrust::sort_by_key(fkey, ekey, cusp->cooPerm->begin(), IJCompare());
    *cusp->cooPerm_a = d_i;
    THRUSTINTARRAY w = d_j;

    auto nekey = thrust::unique(fkey, ekey, IJEqual());
    if (nekey == ekey) { /* all entries are unique */
      delete cusp->cooPerm_a;
      cusp->cooPerm_a = NULL;
    } else { /* I couldn't come up with a more elegant algorithm */
      adjacent_difference(cusp->cooPerm_a->begin(),cusp->cooPerm_a->end(),cusp->cooPerm_a->begin(),IJDiff());
      adjacent_difference(w.begin(),w.end(),w.begin(),IJDiff());
      (*cusp->cooPerm_a)[0] = 0;
      w[0] = 0;
      thrust::transform(cusp->cooPerm_a->begin(),cusp->cooPerm_a->end(),w.begin(),cusp->cooPerm_a->begin(),IJSum());
      thrust::inclusive_scan(cusp->cooPerm_a->begin(),cusp->cooPerm_a->end(),cusp->cooPerm_a->begin(),thrust::plus<PetscInt>());
    }
    thrust::counting_iterator<PetscInt> search_begin(0);
    thrust::upper_bound(d_i.begin(), nekey.get_iterator_tuple().get<0>(),
                        search_begin, search_begin + A->rmap->n,
                        ii.begin());
    cerr = WaitForCUDA();CHKERRCUDA(cerr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);

    ierr = MatSeqXAIJFreeAIJ(A,&a->a,&a->j,&a->i);CHKERRQ(ierr);
    a->singlemalloc = PETSC_FALSE;
    a->free_a       = PETSC_TRUE;
    a->free_ij      = PETSC_TRUE;
    ierr = PetscMalloc1(A->rmap->n+1,&a->i);CHKERRQ(ierr);
    a->i[0] = 0;
    cerr = hipMemcpy(a->i+1,ii.data().get(),A->rmap->n*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    a->nz = a->maxnz = a->i[A->rmap->n];
    a->rmax = 0;
    ierr = PetscMalloc1(a->nz,&a->a);CHKERRQ(ierr);
    ierr = PetscMalloc1(a->nz,&a->j);CHKERRQ(ierr);
    cerr = hipMemcpy(a->j,d_j.data().get(),a->nz*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    if (!a->ilen) { ierr = PetscMalloc1(A->rmap->n,&a->ilen);CHKERRQ(ierr); }
    if (!a->imax) { ierr = PetscMalloc1(A->rmap->n,&a->imax);CHKERRQ(ierr); }
    for (PetscInt i = 0; i < A->rmap->n; i++) {
      const PetscInt nnzr = a->i[i+1] - a->i[i];
      nzr += (PetscInt)!!(nnzr);
      a->ilen[i] = a->imax[i] = nnzr;
      a->rmax = PetscMax(a->rmax,nnzr);
    }
    a->nonzerorowcnt = nzr;
    A->preallocated = PETSC_TRUE;
    ierr = PetscLogGpuToCpu((A->rmap->n+a->nz)*sizeof(PetscInt));CHKERRQ(ierr);
    ierr = MatMarkDiagonal_SeqAIJ(A);CHKERRQ(ierr);
  } else {
    ierr = MatSeqAIJSetPreallocation(A,0,NULL);CHKERRQ(ierr);
  }
  ierr = MatSetOption(A,MAT_NEW_NONZERO_ALLOCATION_ERR,PETSC_TRUE);CHKERRQ(ierr);

  /* We want to allocate the CUSPARSE struct for matvec now.
     The code is so convoluted now that I prefer to copy zeros */
  ierr = PetscArrayzero(a->a,a->nz);CHKERRQ(ierr);
  ierr = MatCheckCompressedRow(A,nzr,&a->compressedrow,a->i,A->rmap->n,0.6);CHKERRQ(ierr);
  A->offloadmask = PETSC_OFFLOAD_CPU;
  A->nonzerostate++;
  ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  ierr = MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_TRUE);CHKERRQ(ierr);

  A->assembled = PETSC_FALSE;
  A->was_assembled = PETSC_FALSE;
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqAIJCUSPARSEGetArrayRead(Mat A, const PetscScalar** a)
{
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  CsrMatrix          *csr;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidPointer(a,2);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  if (cusp->format == MAT_CUSPARSE_ELL || cusp->format == MAT_CUSPARSE_HYB) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
  ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  if (!cusp->mat) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
  csr = (CsrMatrix*)cusp->mat->mat;
  if (!csr->values) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing CUDA memory");
  *a = csr->values->data().get();
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqAIJCUSPARSERestoreArrayRead(Mat A, const PetscScalar** a)
{
  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidPointer(a,2);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  *a = NULL;
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqAIJCUSPARSEGetArray(Mat A, PetscScalar** a)
{
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  CsrMatrix          *csr;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidPointer(a,2);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  if (cusp->format == MAT_CUSPARSE_ELL || cusp->format == MAT_CUSPARSE_HYB) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
  ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  if (!cusp->mat) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
  csr = (CsrMatrix*)cusp->mat->mat;
  if (!csr->values) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing CUDA memory");
  *a = csr->values->data().get();
  A->offloadmask = PETSC_OFFLOAD_GPU;
  ierr = MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqAIJCUSPARSERestoreArray(Mat A, PetscScalar** a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidPointer(a,2);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  ierr = PetscObjectStateIncrease((PetscObject)A);CHKERRQ(ierr);
  *a = NULL;
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqAIJCUSPARSEGetArrayWrite(Mat A, PetscScalar** a)
{
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  CsrMatrix          *csr;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidPointer(a,2);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  if (cusp->format == MAT_CUSPARSE_ELL || cusp->format == MAT_CUSPARSE_HYB) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
  if (!cusp->mat) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
  csr = (CsrMatrix*)cusp->mat->mat;
  if (!csr->values) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing CUDA memory");
  *a = csr->values->data().get();
  A->offloadmask = PETSC_OFFLOAD_GPU;
  ierr = MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqAIJCUSPARSERestoreArrayWrite(Mat A, PetscScalar** a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidPointer(a,2);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  ierr = PetscObjectStateIncrease((PetscObject)A);CHKERRQ(ierr);
  *a = NULL;
  PetscFunctionReturn(0);
}

struct IJCompare4
{
  __host__ __device__
  inline bool operator() (const thrust::tuple<int, int, PetscScalar, int> &t1, const thrust::tuple<int, int, PetscScalar, int> &t2)
  {
    if (t1.get<0>() < t2.get<0>()) return true;
    if (t1.get<0>() == t2.get<0>()) return t1.get<1>() < t2.get<1>();
    return false;
  }
};

struct Shift
{
  int _shift;

  Shift(int shift) : _shift(shift) {}
  __host__ __device__
  inline int operator() (const int &c)
  {
    return c + _shift;
  }
};

/* merges to SeqAIJCUSPARSE matrices, [A';B']' operation in matlab notation */
PetscErrorCode MatSeqAIJCUSPARSEMergeMats(Mat A,Mat B,MatReuse reuse,Mat* C)
{
  PetscErrorCode               ierr;
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data, *b = (Mat_SeqAIJ*)B->data, *c;
  Mat_SeqAIJCUSPARSE           *Acusp = (Mat_SeqAIJCUSPARSE*)A->spptr, *Bcusp = (Mat_SeqAIJCUSPARSE*)B->spptr, *Ccusp;
  Mat_SeqAIJCUSPARSEMultStruct *Cmat;
  CsrMatrix                    *Acsr,*Bcsr,*Ccsr;
  PetscInt                     Annz,Bnnz;
  hipsparseStatus_t             stat;
  PetscInt                     i,m,n,zero = 0;
  hipError_t                  cerr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidHeaderSpecific(B,MAT_CLASSID,2);
  PetscValidPointer(C,4);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  PetscCheckTypeName(B,MATSEQAIJCUSPARSE);
  if (A->rmap->n != B->rmap->n) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"Invalid number or rows %D != %D",A->rmap->n,B->rmap->n);
  if (reuse == MAT_INPLACE_MATRIX) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"MAT_INPLACE_MATRIX not supported");
  if (Acusp->format == MAT_CUSPARSE_ELL || Acusp->format == MAT_CUSPARSE_HYB) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
  if (Bcusp->format == MAT_CUSPARSE_ELL || Bcusp->format == MAT_CUSPARSE_HYB) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
  if (reuse == MAT_INITIAL_MATRIX) {
    m     = A->rmap->n;
    n     = A->cmap->n + B->cmap->n;
    ierr  = MatCreate(PETSC_COMM_SELF,C);CHKERRQ(ierr);
    ierr  = MatSetSizes(*C,m,n,m,n);CHKERRQ(ierr);
    ierr  = MatSetType(*C,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
    c     = (Mat_SeqAIJ*)(*C)->data;
    Ccusp = (Mat_SeqAIJCUSPARSE*)(*C)->spptr;
    Cmat  = new Mat_SeqAIJCUSPARSEMultStruct;
    Ccsr  = new CsrMatrix;
    Cmat->cprowIndices      = NULL;
    c->compressedrow.use    = PETSC_FALSE;
    c->compressedrow.nrows  = 0;
    c->compressedrow.i      = NULL;
    c->compressedrow.rindex = NULL;
    Ccusp->workVector       = NULL;
    Ccusp->nrows    = m;
    Ccusp->mat      = Cmat;
    Ccusp->mat->mat = Ccsr;
    Ccsr->num_rows  = m;
    Ccsr->num_cols  = n;
    stat = hipsparseCreateMatDescr(&Cmat->descr);CHKERRCUSPARSE(stat);
    stat = hipsparseSetMatIndexBase(Cmat->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
    stat = hipsparseSetMatType(Cmat->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUSPARSE(stat);
    cerr = hipMalloc((void **)&(Cmat->alpha_one),sizeof(PetscScalar));CHKERRCUDA(cerr);
    cerr = hipMalloc((void **)&(Cmat->beta_zero),sizeof(PetscScalar));CHKERRCUDA(cerr);
    cerr = hipMalloc((void **)&(Cmat->beta_one), sizeof(PetscScalar));CHKERRCUDA(cerr);
    cerr = hipMemcpy(Cmat->alpha_one,&PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    cerr = hipMemcpy(Cmat->beta_zero,&PETSC_CUSPARSE_ZERO,sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    cerr = hipMemcpy(Cmat->beta_one, &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
    ierr = MatSeqAIJCUSPARSECopyToGPU(B);CHKERRQ(ierr);
    ierr = MatSeqAIJCUSPARSEFormExplicitTransposeForMult(A);CHKERRQ(ierr);
    ierr = MatSeqAIJCUSPARSEFormExplicitTransposeForMult(B);CHKERRQ(ierr);
    if (!Acusp->mat) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
    if (!Bcusp->mat) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");

    Acsr = (CsrMatrix*)Acusp->mat->mat;
    Bcsr = (CsrMatrix*)Bcusp->mat->mat;
    Annz = (PetscInt)Acsr->column_indices->size();
    Bnnz = (PetscInt)Bcsr->column_indices->size();
    c->nz = Annz + Bnnz;
    Ccsr->row_offsets = new THRUSTINTARRAY32(m+1);
    Ccsr->column_indices = new THRUSTINTARRAY32(c->nz);
    Ccsr->values = new THRUSTARRAY(c->nz);
    Ccsr->num_entries = c->nz;
    Ccusp->cooPerm = new THRUSTINTARRAY(c->nz);
    if (c->nz) {
      auto Acoo = new THRUSTINTARRAY32(Annz);
      auto Bcoo = new THRUSTINTARRAY32(Bnnz);
      auto Ccoo = new THRUSTINTARRAY32(c->nz);
      THRUSTINTARRAY32 *Aroff,*Broff;

      if (a->compressedrow.use) { /* need full row offset */
        if (!Acusp->rowoffsets_gpu) {
          Acusp->rowoffsets_gpu  = new THRUSTINTARRAY32(A->rmap->n + 1);
          Acusp->rowoffsets_gpu->assign(a->i,a->i + A->rmap->n + 1);
          ierr = PetscLogCpuToGpu((A->rmap->n + 1)*sizeof(PetscInt));CHKERRQ(ierr);
        }
        Aroff = Acusp->rowoffsets_gpu;
      } else Aroff = Acsr->row_offsets;
      if (b->compressedrow.use) { /* need full row offset */
        if (!Bcusp->rowoffsets_gpu) {
          Bcusp->rowoffsets_gpu  = new THRUSTINTARRAY32(B->rmap->n + 1);
          Bcusp->rowoffsets_gpu->assign(b->i,b->i + B->rmap->n + 1);
          ierr = PetscLogCpuToGpu((B->rmap->n + 1)*sizeof(PetscInt));CHKERRQ(ierr);
        }
        Broff = Bcusp->rowoffsets_gpu;
      } else Broff = Bcsr->row_offsets;
      ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
      stat = hipsparseXcsr2coo(Acusp->handle,
                              Aroff->data().get(),
                              Annz,
                              m,
                              Acoo->data().get(),
                              HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
      stat = hipsparseXcsr2coo(Bcusp->handle,
                              Broff->data().get(),
                              Bnnz,
                              m,
                              Bcoo->data().get(),
                              HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
      /* Issues when using bool with large matrices on SUMMIT 10.2.89 */
      auto Aperm = thrust::make_constant_iterator(1);
      auto Bperm = thrust::make_constant_iterator(0);
#if PETSC_PKG_CUDA_VERSION_GE(10,0,0)
      auto Bcib = thrust::make_transform_iterator(Bcsr->column_indices->begin(),Shift(A->cmap->n));
      auto Bcie = thrust::make_transform_iterator(Bcsr->column_indices->end(),Shift(A->cmap->n));
#else
      /* there are issues instantiating the merge operation using a transform iterator for the columns of B */
      auto Bcib = Bcsr->column_indices->begin();
      auto Bcie = Bcsr->column_indices->end();
      thrust::transform(Bcib,Bcie,Bcib,Shift(A->cmap->n));
#endif
      auto wPerm = new THRUSTINTARRAY32(Annz+Bnnz);
      auto Azb = thrust::make_zip_iterator(thrust::make_tuple(Acoo->begin(),Acsr->column_indices->begin(),Acsr->values->begin(),Aperm));
      auto Aze = thrust::make_zip_iterator(thrust::make_tuple(Acoo->end(),Acsr->column_indices->end(),Acsr->values->end(),Aperm));
      auto Bzb = thrust::make_zip_iterator(thrust::make_tuple(Bcoo->begin(),Bcib,Bcsr->values->begin(),Bperm));
      auto Bze = thrust::make_zip_iterator(thrust::make_tuple(Bcoo->end(),Bcie,Bcsr->values->end(),Bperm));
      auto Czb = thrust::make_zip_iterator(thrust::make_tuple(Ccoo->begin(),Ccsr->column_indices->begin(),Ccsr->values->begin(),wPerm->begin()));
      auto p1 = Ccusp->cooPerm->begin();
      auto p2 = Ccusp->cooPerm->begin();
      thrust::advance(p2,Annz);
      PetscStackCallThrust(thrust::merge(thrust::device,Azb,Aze,Bzb,Bze,Czb,IJCompare4()));
#if PETSC_PKG_CUDA_VERSION_LT(10,0,0)
      thrust::transform(Bcib,Bcie,Bcib,Shift(-A->cmap->n));
#endif
      auto cci = thrust::make_counting_iterator(zero);
      auto cce = thrust::make_counting_iterator(c->nz);
#if 0 //Errors on SUMMIT cuda 11.1.0
      PetscStackCallThrust(thrust::partition_copy(thrust::device,cci,cce,wPerm->begin(),p1,p2,thrust::identity<int>()));
#else
      auto pred = thrust::identity<int>();
      PetscStackCallThrust(thrust::copy_if(thrust::device,cci,cce,wPerm->begin(),p1,pred));
      PetscStackCallThrust(thrust::remove_copy_if(thrust::device,cci,cce,wPerm->begin(),p2,pred));
#endif
      stat = hipsparseXcoo2csr(Ccusp->handle,
                              Ccoo->data().get(),
                              c->nz,
                              m,
                              Ccsr->row_offsets->data().get(),
                              HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
      cerr = WaitForCUDA();CHKERRCUDA(cerr);
      ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
      delete wPerm;
      delete Acoo;
      delete Bcoo;
      delete Ccoo;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      stat = hipsparseCreateCsr(&Cmat->matDescr, Ccsr->num_rows, Ccsr->num_cols, Ccsr->num_entries,
                               Ccsr->row_offsets->data().get(), Ccsr->column_indices->data().get(), Ccsr->values->data().get(),
                               HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                               HIPSPARSE_INDEX_BASE_ZERO, cusparse_scalartype);CHKERRCUSPARSE(stat);
#endif
      if (A->form_explicit_transpose && B->form_explicit_transpose) { /* if A and B have the transpose, generate C transpose too */
        PetscBool AT = Acusp->matTranspose ? PETSC_TRUE : PETSC_FALSE, BT = Bcusp->matTranspose ? PETSC_TRUE : PETSC_FALSE;
        Mat_SeqAIJCUSPARSEMultStruct *CmatT = new Mat_SeqAIJCUSPARSEMultStruct;
        CsrMatrix *CcsrT = new CsrMatrix;
        CsrMatrix *AcsrT = AT ? (CsrMatrix*)Acusp->matTranspose->mat : NULL;
        CsrMatrix *BcsrT = BT ? (CsrMatrix*)Bcusp->matTranspose->mat : NULL;

        (*C)->form_explicit_transpose = PETSC_TRUE;
        (*C)->transupdated = PETSC_TRUE;
        Ccusp->rowoffsets_gpu = NULL;
        CmatT->cprowIndices = NULL;
        CmatT->mat = CcsrT;
        CcsrT->num_rows = n;
        CcsrT->num_cols = m;
        CcsrT->num_entries = c->nz;

        CcsrT->row_offsets = new THRUSTINTARRAY32(n+1);
        CcsrT->column_indices = new THRUSTINTARRAY32(c->nz);
        CcsrT->values = new THRUSTARRAY(c->nz);

        ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
        auto rT = CcsrT->row_offsets->begin();
        if (AT) {
          rT = thrust::copy(AcsrT->row_offsets->begin(),AcsrT->row_offsets->end(),rT);
          thrust::advance(rT,-1);
        }
        if (BT) {
          auto titb = thrust::make_transform_iterator(BcsrT->row_offsets->begin(),Shift(a->nz));
          auto tite = thrust::make_transform_iterator(BcsrT->row_offsets->end(),Shift(a->nz));
          thrust::copy(titb,tite,rT);
        }
        auto cT = CcsrT->column_indices->begin();
        if (AT) cT = thrust::copy(AcsrT->column_indices->begin(),AcsrT->column_indices->end(),cT);
        if (BT) thrust::copy(BcsrT->column_indices->begin(),BcsrT->column_indices->end(),cT);
        auto vT = CcsrT->values->begin();
        if (AT) vT = thrust::copy(AcsrT->values->begin(),AcsrT->values->end(),vT);
        if (BT) thrust::copy(BcsrT->values->begin(),BcsrT->values->end(),vT);
        cerr = WaitForCUDA();CHKERRCUDA(cerr);
        ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);

        stat = hipsparseCreateMatDescr(&CmatT->descr);CHKERRCUSPARSE(stat);
        stat = hipsparseSetMatIndexBase(CmatT->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
        stat = hipsparseSetMatType(CmatT->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUSPARSE(stat);
        cerr = hipMalloc((void **)&(CmatT->alpha_one),sizeof(PetscScalar));CHKERRCUDA(cerr);
        cerr = hipMalloc((void **)&(CmatT->beta_zero),sizeof(PetscScalar));CHKERRCUDA(cerr);
        cerr = hipMalloc((void **)&(CmatT->beta_one), sizeof(PetscScalar));CHKERRCUDA(cerr);
        cerr = hipMemcpy(CmatT->alpha_one,&PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
        cerr = hipMemcpy(CmatT->beta_zero,&PETSC_CUSPARSE_ZERO,sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
        cerr = hipMemcpy(CmatT->beta_one, &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
        stat = hipsparseCreateCsr(&CmatT->matDescr, CcsrT->num_rows, CcsrT->num_cols, CcsrT->num_entries,
                                 CcsrT->row_offsets->data().get(), CcsrT->column_indices->data().get(), CcsrT->values->data().get(),
                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                 HIPSPARSE_INDEX_BASE_ZERO, cusparse_scalartype);CHKERRCUSPARSE(stat);
#endif
        Ccusp->matTranspose = CmatT;
      }
    }

    c->singlemalloc = PETSC_FALSE;
    c->free_a       = PETSC_TRUE;
    c->free_ij      = PETSC_TRUE;
    ierr = PetscMalloc1(m+1,&c->i);CHKERRQ(ierr);
    ierr = PetscMalloc1(c->nz,&c->j);CHKERRQ(ierr);
    if (PetscDefined(USE_64BIT_INDICES)) { /* 32 to 64 bit conversion on the GPU and then copy to host (lazy) */
      THRUSTINTARRAY ii(Ccsr->row_offsets->size());
      THRUSTINTARRAY jj(Ccsr->column_indices->size());
      ii   = *Ccsr->row_offsets;
      jj   = *Ccsr->column_indices;
      cerr = hipMemcpy(c->i,ii.data().get(),Ccsr->row_offsets->size()*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
      cerr = hipMemcpy(c->j,jj.data().get(),Ccsr->column_indices->size()*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    } else {
      cerr = hipMemcpy(c->i,Ccsr->row_offsets->data().get(),Ccsr->row_offsets->size()*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
      cerr = hipMemcpy(c->j,Ccsr->column_indices->data().get(),Ccsr->column_indices->size()*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    }
    ierr = PetscLogGpuToCpu((Ccsr->column_indices->size() + Ccsr->row_offsets->size())*sizeof(PetscInt));CHKERRQ(ierr);
    ierr = PetscMalloc1(m,&c->ilen);CHKERRQ(ierr);
    ierr = PetscMalloc1(m,&c->imax);CHKERRQ(ierr);
    c->maxnz = c->nz;
    c->nonzerorowcnt = 0;
    c->rmax = 0;
    for (i = 0; i < m; i++) {
      const PetscInt nn = c->i[i+1] - c->i[i];
      c->ilen[i] = c->imax[i] = nn;
      c->nonzerorowcnt += (PetscInt)!!nn;
      c->rmax = PetscMax(c->rmax,nn);
    }
    ierr = MatMarkDiagonal_SeqAIJ(*C);CHKERRQ(ierr);
    ierr = PetscMalloc1(c->nz,&c->a);CHKERRQ(ierr);
    (*C)->nonzerostate++;
    ierr = PetscLayoutSetUp((*C)->rmap);CHKERRQ(ierr);
    ierr = PetscLayoutSetUp((*C)->cmap);CHKERRQ(ierr);
    Ccusp->nonzerostate = (*C)->nonzerostate;
    (*C)->preallocated  = PETSC_TRUE;
  } else {
    if ((*C)->rmap->n != B->rmap->n) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"Invalid number or rows %D != %D",(*C)->rmap->n,B->rmap->n);
    c = (Mat_SeqAIJ*)(*C)->data;
    if (c->nz) {
      Ccusp = (Mat_SeqAIJCUSPARSE*)(*C)->spptr;
      if (!Ccusp->cooPerm) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing cooPerm");
      if (Ccusp->format == MAT_CUSPARSE_ELL || Ccusp->format == MAT_CUSPARSE_HYB) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
      if (Ccusp->nonzerostate != (*C)->nonzerostate) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Wrong nonzerostate");
      ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
      ierr = MatSeqAIJCUSPARSECopyToGPU(B);CHKERRQ(ierr);
      if (!Acusp->mat) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
      if (!Bcusp->mat) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
      Acsr = (CsrMatrix*)Acusp->mat->mat;
      Bcsr = (CsrMatrix*)Bcusp->mat->mat;
      Ccsr = (CsrMatrix*)Ccusp->mat->mat;
      if (Acsr->num_entries != (PetscInt)Acsr->values->size()) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_COR,"A nnz %D != %D",Acsr->num_entries,(PetscInt)Acsr->values->size());
      if (Bcsr->num_entries != (PetscInt)Bcsr->values->size()) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_COR,"B nnz %D != %D",Bcsr->num_entries,(PetscInt)Bcsr->values->size());
      if (Ccsr->num_entries != (PetscInt)Ccsr->values->size()) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_COR,"C nnz %D != %D",Ccsr->num_entries,(PetscInt)Ccsr->values->size());
      if (Ccsr->num_entries != Acsr->num_entries + Bcsr->num_entries) SETERRQ3(PETSC_COMM_SELF,PETSC_ERR_COR,"C nnz %D != %D + %D",Ccsr->num_entries,Acsr->num_entries,Bcsr->num_entries);
      if (Ccusp->cooPerm->size() != Ccsr->values->size()) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_COR,"permSize %D != %D",(PetscInt)Ccusp->cooPerm->size(),(PetscInt)Ccsr->values->size());
      auto pmid = Ccusp->cooPerm->begin();
      thrust::advance(pmid,Acsr->num_entries);
      ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
      auto zibait = thrust::make_zip_iterator(thrust::make_tuple(Acsr->values->begin(),
                                                                 thrust::make_permutation_iterator(Ccsr->values->begin(),Ccusp->cooPerm->begin())));
      auto zieait = thrust::make_zip_iterator(thrust::make_tuple(Acsr->values->end(),
                                                                 thrust::make_permutation_iterator(Ccsr->values->begin(),pmid)));
      thrust::for_each(zibait,zieait,VecCUDAEquals());
      auto zibbit = thrust::make_zip_iterator(thrust::make_tuple(Bcsr->values->begin(),
                                                                 thrust::make_permutation_iterator(Ccsr->values->begin(),pmid)));
      auto ziebit = thrust::make_zip_iterator(thrust::make_tuple(Bcsr->values->end(),
                                                                 thrust::make_permutation_iterator(Ccsr->values->begin(),Ccusp->cooPerm->end())));
      thrust::for_each(zibbit,ziebit,VecCUDAEquals());
      ierr = MatSeqAIJCUSPARSEInvalidateTranspose(*C,PETSC_FALSE);CHKERRQ(ierr);
      if (A->form_explicit_transpose && B->form_explicit_transpose && (*C)->form_explicit_transpose) {
        if (!Ccusp->matTranspose) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing transpose Mat_SeqAIJCUSPARSEMultStruct");
        PetscBool AT = Acusp->matTranspose ? PETSC_TRUE : PETSC_FALSE, BT = Bcusp->matTranspose ? PETSC_TRUE : PETSC_FALSE;
        CsrMatrix *AcsrT = AT ? (CsrMatrix*)Acusp->matTranspose->mat : NULL;
        CsrMatrix *BcsrT = BT ? (CsrMatrix*)Bcusp->matTranspose->mat : NULL;
        CsrMatrix *CcsrT = (CsrMatrix*)Ccusp->matTranspose->mat;
        auto vT = CcsrT->values->begin();
        if (AT) vT = thrust::copy(AcsrT->values->begin(),AcsrT->values->end(),vT);
        if (BT) thrust::copy(BcsrT->values->begin(),BcsrT->values->end(),vT);
        (*C)->transupdated = PETSC_TRUE;
      }
      cerr = WaitForCUDA();CHKERRCUDA(cerr);
      ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    }
  }
  ierr = PetscObjectStateIncrease((PetscObject)*C);CHKERRQ(ierr);
  (*C)->assembled     = PETSC_TRUE;
  (*C)->was_assembled = PETSC_FALSE;
  (*C)->offloadmask   = PETSC_OFFLOAD_GPU;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCopySubArray_SeqAIJCUSPARSE(Mat A, PetscInt n, const PetscInt idx[], PetscScalar v[])
{
  PetscErrorCode    ierr;
  bool              dmem;
  const PetscScalar *av;
  hipError_t       cerr;

  PetscFunctionBegin;
  dmem = isCudaMem(v);
  ierr = MatSeqAIJCUSPARSEGetArrayRead(A,&av);CHKERRQ(ierr);
  if (n && idx) {
    THRUSTINTARRAY widx(n);
    widx.assign(idx,idx+n);
    ierr = PetscLogCpuToGpu(n*sizeof(PetscInt));CHKERRQ(ierr);

    THRUSTARRAY *w = NULL;
    thrust::device_ptr<PetscScalar> dv;
    if (dmem) {
      dv = thrust::device_pointer_cast(v);
    } else {
      w = new THRUSTARRAY(n);
      dv = w->data();
    }
    thrust::device_ptr<const PetscScalar> dav = thrust::device_pointer_cast(av);

    auto zibit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(dav,widx.begin()),dv));
    auto zieit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(dav,widx.end()),dv+n));
    thrust::for_each(zibit,zieit,VecCUDAEquals());
    if (w) {
      cerr = hipMemcpy(v,w->data().get(),n*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    }
    delete w;
  } else {
    cerr = hipMemcpy(v,av,n*sizeof(PetscScalar),dmem ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
  }
  if (!dmem) { ierr = PetscLogCpuToGpu(n*sizeof(PetscScalar));CHKERRQ(ierr); }
  ierr = MatSeqAIJCUSPARSERestoreArrayRead(A,&av);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
