/*
     Defines the matrix operations for sequential dense with CUDA
*/
#include <petscpkg_version.h>
#define PETSC_SKIP_IMMINTRIN_H_CUDAWORKAROUND 1
#include <../src/mat/impls/dense/seq/dense.h> /*I "petscmat.h" I*/
#include <petsccublas.h>

/* cublas definitions are here */
#include <petsc/private/cudavecimpl.h>

#if defined(PETSC_USE_COMPLEX)
#if defined(PETSC_USE_REAL_SINGLE)
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnCpotrf((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnCpotrf_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnCpotrs((a),(b),(c),(d),(hipComplex*)(e),(f),(hipComplex*)(g),(h),(i))
#define cusolverDnXpotri(a,b,c,d,e,f,g,h)        hipsolverDnCpotri((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(g),(h))
#define cusolverDnXpotri_bufferSize(a,b,c,d,e,f) hipsolverDnCpotri_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnCsytrf((a),(b),(c),(hipComplex*)(d),(e),(f),(hipComplex*)(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnCsytrf_bufferSize((a),(b),(hipComplex*)(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnCgetrf((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnCgetrf_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnCgetrs((a),(b),(c),(d),(hipComplex*)(e),(f),(g),(hipComplex*)(h),(i),(j))
#else /* complex double */
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnZpotrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnZpotrf_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnZpotrs((a),(b),(c),(d),(hipDoubleComplex*)(e),(f),(hipDoubleComplex*)(g),(h),(i))
#define cusolverDnXpotri(a,b,c,d,e,f,g,h)        hipsolverDnZpotri((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(g),(h))
#define cusolverDnXpotri_bufferSize(a,b,c,d,e,f) hipsolverDnZpotri_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnZsytrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(f),(hipDoubleComplex*)(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnZsytrf_bufferSize((a),(b),(hipDoubleComplex*)(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnZgetrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnZgetrf_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnZgetrs((a),(b),(c),(d),(hipDoubleComplex*)(e),(f),(g),(hipDoubleComplex*)(h),(i),(j))
#endif
#else /* real single */
#if defined(PETSC_USE_REAL_SINGLE)
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnSpotrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnSpotrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnSpotrs((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXpotri(a,b,c,d,e,f,g,h)        hipsolverDnSpotri((a),(b),(c),(d),(e),(f),(g),(h))
#define cusolverDnXpotri_bufferSize(a,b,c,d,e,f) hipsolverDnSpotri_bufferSize((a),(b),(c),(d),(e),(f))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnSsytrf((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnSsytrf_bufferSize((a),(b),(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnSgetrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnSgetrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnSgetrs((a),(b),(c),(d),(e),(f),(g),(h),(i),(j))
#else /* real double */
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnDpotrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnDpotrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnDpotrs((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXpotri(a,b,c,d,e,f,g,h)        hipsolverDnDpotri((a),(b),(c),(d),(e),(f),(g),(h))
#define cusolverDnXpotri_bufferSize(a,b,c,d,e,f) hipsolverDnDpotri_bufferSize((a),(b),(c),(d),(e),(f))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnDsytrf((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnDsytrf_bufferSize((a),(b),(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnDgetrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnDgetrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnDgetrs((a),(b),(c),(d),(e),(f),(g),(h),(i),(j))
#endif
#endif

typedef struct {
  PetscScalar *d_v; /* pointer to the matrix on the GPU */
  PetscBool   user_alloc;
  PetscScalar *unplacedarray; /* if one called MatCUDADensePlaceArray(), this is where it stashed the original */
  PetscBool   unplaced_user_alloc;
  /* factorization support */
  int         *d_fact_ipiv; /* device pivots */
  PetscScalar *d_fact_work; /* device workspace */
  int         fact_lwork;
  int         *d_fact_info; /* device info */
  /* workspace */
  Vec         workvec;
} Mat_SeqDenseCUDA;

PetscErrorCode MatSeqDenseCUDASetPreallocation(Mat A, PetscScalar *d_data)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;
  PetscBool        iscuda;
  hipError_t      cerr;

  PetscFunctionBegin;
  ierr = PetscObjectTypeCompare((PetscObject)A,MATSEQDENSECUDA,&iscuda);CHKERRQ(ierr);
  if (!iscuda) PetscFunctionReturn(0);
  /* it may happen CPU preallocation has not been performed */
  ierr = PetscLayoutSetUp(A->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->cmap);CHKERRQ(ierr);
  if (cA->lda <= 0) cA->lda = A->rmap->n;
  if (!dA->user_alloc) { cerr = hipFree(dA->d_v);CHKERRCUDA(cerr); }
  if (!d_data) { /* petsc-allocated storage */
    size_t sz;
    ierr = PetscIntMultError(cA->lda,A->cmap->n,NULL);CHKERRQ(ierr);
    sz   = cA->lda*A->cmap->n*sizeof(PetscScalar);
    cerr = hipMalloc((void**)&dA->d_v,sz);CHKERRCUDA(cerr);
    cerr = hipMemset(dA->d_v,0,sz);CHKERRCUDA(cerr);
    dA->user_alloc = PETSC_FALSE;
  } else { /* user-allocated storage */
    dA->d_v        = d_data;
    dA->user_alloc = PETSC_TRUE;
  }
  A->offloadmask  = PETSC_OFFLOAD_GPU;
  A->preallocated = PETSC_TRUE;
  A->assembled    = PETSC_TRUE;
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqDenseCUDACopyFromGPU(Mat A)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;
  hipError_t      cerr;

  PetscFunctionBegin;
  PetscCheckTypeName(A,MATSEQDENSECUDA);
  ierr = PetscInfo3(A,"%s matrix %d x %d\n",A->offloadmask == PETSC_OFFLOAD_GPU ? "Copy" : "Reusing",A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  if (A->offloadmask == PETSC_OFFLOAD_GPU) {
    if (!cA->v) { /* MatCreateSeqDenseCUDA may not allocate CPU memory. Allocate if needed */
      ierr = MatSeqDenseSetPreallocation(A,NULL);CHKERRQ(ierr);
    }
    ierr = PetscLogEventBegin(MAT_DenseCopyFromGPU,A,0,0,0);CHKERRQ(ierr);
    if (cA->lda > A->rmap->n) {
      PetscInt j,m = A->rmap->n;

      for (j=0; j<A->cmap->n; j++) { /* TODO: it can be done better */
        cerr = hipMemcpy(cA->v + j*cA->lda,dA->d_v + j*cA->lda,m*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
      }
    } else {
      cerr = hipMemcpy(cA->v,dA->d_v,cA->lda*sizeof(PetscScalar)*A->cmap->n,hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    }
    ierr = PetscLogGpuToCpu(cA->lda*sizeof(PetscScalar)*A->cmap->n);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(MAT_DenseCopyFromGPU,A,0,0,0);CHKERRQ(ierr);

    A->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqDenseCUDACopyToGPU(Mat A)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscBool        copy;
  PetscErrorCode   ierr;
  hipError_t      cerr;

  PetscFunctionBegin;
  PetscCheckTypeName(A,MATSEQDENSECUDA);
  if (A->boundtocpu) PetscFunctionReturn(0);
  copy = (PetscBool)(A->offloadmask == PETSC_OFFLOAD_CPU || A->offloadmask == PETSC_OFFLOAD_UNALLOCATED);
  ierr = PetscInfo3(A,"%s matrix %d x %d\n",copy ? "Copy" : "Reusing",A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  if (copy) {
    if (!dA->d_v) { /* Allocate GPU memory if not present */
      ierr = MatSeqDenseCUDASetPreallocation(A,NULL);CHKERRQ(ierr);
    }
    ierr = PetscLogEventBegin(MAT_DenseCopyToGPU,A,0,0,0);CHKERRQ(ierr);
    if (cA->lda > A->rmap->n) {
      PetscInt j,m = A->rmap->n;

      for (j=0; j<A->cmap->n; j++) { /* TODO: it can be done better */
        cerr = hipMemcpy(dA->d_v + j*cA->lda,cA->v + j*cA->lda,m*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
      }
    } else {
      cerr = hipMemcpy(dA->d_v,cA->v,cA->lda*sizeof(PetscScalar)*A->cmap->n,hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    }
    ierr = PetscLogCpuToGpu(cA->lda*sizeof(PetscScalar)*A->cmap->n);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(MAT_DenseCopyToGPU,A,0,0,0);CHKERRQ(ierr);

    A->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCopy_SeqDenseCUDA(Mat A,Mat B,MatStructure str)
{
  Mat_SeqDense      *a = (Mat_SeqDense*)A->data,*b = (Mat_SeqDense*)B->data;
  PetscErrorCode    ierr;
  const PetscScalar *va;
  PetscScalar       *vb;
  PetscInt          lda1=a->lda,lda2=b->lda, m=A->rmap->n,n=A->cmap->n, j;
  hipError_t       cerr;

  PetscFunctionBegin;
  /* If the two matrices don't have the same copy implementation, they aren't compatible for fast copy. */
  if (A->ops->copy != B->ops->copy) {
    ierr = MatCopy_Basic(A,B,str);CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }
  if (m != B->rmap->n || n != B->cmap->n) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"size(B) != size(A)");
  ierr = MatDenseCUDAGetArrayRead(A,&va);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayWrite(B,&vb);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (lda1>m || lda2>m) {
    for (j=0; j<n; j++) {
      cerr = hipMemcpy(vb+j*lda2,va+j*lda1,m*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
    }
  } else {
    cerr = hipMemcpy(vb,va,m*(n*sizeof(PetscScalar)),hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
  }
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArray(B,&vb);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&va);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAPlaceArray_SeqDenseCUDA(Mat A, const PetscScalar *a)
{
  Mat_SeqDense     *aa = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (aa->vecinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  if (aa->matinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  if (dA->unplacedarray) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"MatDenseCUDAResetArray() must be called first");
  if (aa->v) { ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr); }
  dA->unplacedarray = dA->d_v;
  dA->unplaced_user_alloc = dA->user_alloc;
  dA->d_v = (PetscScalar*)a;
  dA->user_alloc = PETSC_TRUE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAResetArray_SeqDenseCUDA(Mat A)
{
  Mat_SeqDense     *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (a->vecinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  if (a->matinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  if (a->v) { ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr); }
  dA->d_v = dA->unplacedarray;
  dA->user_alloc = dA->unplaced_user_alloc;
  dA->unplacedarray = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAReplaceArray_SeqDenseCUDA(Mat A, const PetscScalar *a)
{
  Mat_SeqDense     *aa = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  hipError_t      cerr;

  PetscFunctionBegin;
  if (aa->vecinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  if (aa->matinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  if (dA->unplacedarray) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"MatDenseCUDAResetArray() must be called first");
  if (!dA->user_alloc) { cerr = hipFree(dA->d_v);CHKERRCUDA(cerr); }
  dA->d_v = (PetscScalar*)a;
  dA->user_alloc = PETSC_FALSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAGetArrayWrite_SeqDenseCUDA(Mat A, PetscScalar **a)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (!dA->d_v) {
    ierr = MatSeqDenseCUDASetPreallocation(A,NULL);CHKERRQ(ierr);
  }
  *a = dA->d_v;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDARestoreArrayWrite_SeqDenseCUDA(Mat A, PetscScalar **a)
{
  PetscFunctionBegin;
  *a = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAGetArrayRead_SeqDenseCUDA(Mat A, const PetscScalar **a)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr);
  *a   = dA->d_v;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDARestoreArrayRead_SeqDenseCUDA(Mat A, const PetscScalar **a)
{
  PetscFunctionBegin;
  *a = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAGetArray_SeqDenseCUDA(Mat A, PetscScalar **a)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr);
  *a   = dA->d_v;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDARestoreArray_SeqDenseCUDA(Mat A, PetscScalar **a)
{
  PetscFunctionBegin;
  *a = NULL;
  PetscFunctionReturn(0);
}

PETSC_EXTERN PetscErrorCode MatSeqDenseCUDAInvertFactors_Private(Mat A)
{
#if PETSC_PKG_CUDA_VERSION_GE(10,1,0)
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  PetscErrorCode     ierr;
  hipError_t        ccer;
  hipsolverStatus_t   cerr;
  hipsolverHandle_t handle;
  int                n,lda;
#if defined(PETSC_USE_DEBUG)
  int                info;
#endif

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->cmap->n,&n);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(a->lda,&lda);CHKERRQ(ierr);
  if (A->factortype == MAT_FACTOR_LU) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"cusolverDngetri not implemented");
  else if (A->factortype == MAT_FACTOR_CHOLESKY) {
    if (!dA->d_fact_ipiv) { /* spd */
      int il;

      ierr = MatDenseCUDAGetArray(A,&da);CHKERRQ(ierr);
      cerr = cusolverDnXpotri_bufferSize(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,&il);CHKERRCUSOLVER(cerr);
      if (il > dA->fact_lwork) {
        dA->fact_lwork = il;

        ccer = hipFree(dA->d_fact_work);CHKERRCUDA(ccer);
        ccer = hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work));CHKERRCUDA(ccer);
      }
      ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
      cerr = cusolverDnXpotri(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info);CHKERRCUSOLVER(cerr);
      ccer = WaitForCUDA();CHKERRCUDA(ccer);
      ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
      ierr = MatDenseCUDARestoreArray(A,&da);CHKERRQ(ierr);
      /* TODO (write cuda kernel) */
      ierr = MatSeqDenseSymmetrize_Private(A,PETSC_TRUE);CHKERRQ(ierr);
    } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"cusolverDnsytri not implemented");
  }
#if defined(PETSC_USE_DEBUG)
  ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(int), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
  if (info > 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: leading minor of order %d is zero",info);
  else if (info < 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
  ierr = PetscLogGpuFlops(1.0*n*n*n/3.0);CHKERRQ(ierr);
  A->ops->solve          = NULL;
  A->ops->solvetranspose = NULL;
  A->ops->matsolve       = NULL;
  A->factortype          = MAT_FACTOR_NONE;

  ierr = PetscFree(A->solvertype);CHKERRQ(ierr);
  PetscFunctionReturn(0);
#else
  SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Upgrade to CUDA version 10.1.0 or higher");
#endif
}

static PetscErrorCode MatMatSolve_SeqDenseCUDA(Mat A,Mat B,Mat X)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDense       *x = (Mat_SeqDense*)X->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  const PetscScalar  *da;
  PetscScalar        *dx;
  hipsolverHandle_t handle;
  PetscBool          iscuda;
  int                nrhs,n,lda,ldx;
#if defined(PETSC_USE_DEBUG)
  int                info;
#endif
  hipError_t        ccer;
  hipsolverStatus_t   cerr;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (A->factortype == MAT_FACTOR_NONE) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
  if (!dA->d_fact_work) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
  ierr = PetscObjectTypeCompareAny((PetscObject)X,&iscuda,VECSEQCUDA,VECMPICUDA,"");CHKERRQ(ierr);
  if (X != B) {
    ierr = MatCopy(B,X,SAME_NONZERO_PATTERN);CHKERRQ(ierr);
  }
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  /* MatMatSolve does not have a dispatching mechanism, we may end up with a MATSEQDENSE here */
  ierr = PetscObjectTypeCompare((PetscObject)X,MATSEQDENSECUDA,&iscuda);CHKERRQ(ierr);
  if (!iscuda) {
    ierr = MatConvert(X,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&X);CHKERRQ(ierr);
  }
  ierr = MatDenseCUDAGetArray(X,&dx);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->rmap->n,&n);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(X->cmap->n,&nrhs);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(a->lda,&lda);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(x->lda,&ldx);CHKERRQ(ierr);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (A->factortype == MAT_FACTOR_LU) {
    ierr = PetscInfo2(A,"LU solve %d x %d on backend\n",n,n);CHKERRQ(ierr);
    cerr = hipsolverDnXgetrs(handle,HIPBLAS_OP_N,n,nrhs,da,lda,dA->d_fact_ipiv,dx,ldx,dA->d_fact_info);CHKERRCUSOLVER(cerr);
  } else if (A->factortype == MAT_FACTOR_CHOLESKY) {
    ierr = PetscInfo2(A,"Cholesky solve %d x %d on backend\n",n,n);CHKERRQ(ierr);
    if (!dA->d_fact_ipiv) { /* spd */
      /* ========= Program hit hipErrorNotReady (error 34) due to "device not ready" on CUDA API call to hipEventQuery. */
      cerr = hipsolverDnXpotrs(handle,HIPBLAS_FILL_MODE_LOWER,n,nrhs,da,lda,dx,ldx,dA->d_fact_info);CHKERRCUSOLVER(cerr);
    } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"cusolverDnsytrs not implemented");
  } else SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"Unknown factor type %d",A->factortype);
  ccer = WaitForCUDA();CHKERRCUDA(ccer);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArray(X,&dx);CHKERRQ(ierr);
  if (!iscuda) {
    ierr = MatConvert(X,MATSEQDENSE,MAT_INPLACE_MATRIX,&X);CHKERRQ(ierr);
  }
#if defined(PETSC_USE_DEBUG)
  ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(int), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
  if (info > 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
  else if (info < 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
  ierr = PetscLogGpuFlops(nrhs*(2.0*n*n - n));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_Private(Mat A,Vec xx,Vec yy,PetscBool trans)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  const PetscScalar  *da;
  PetscScalar        *y;
  hipsolverHandle_t handle;
  int                one = 1,n,lda;
#if defined(PETSC_USE_DEBUG)
  int                info;
#endif
  hipError_t        ccer;
  hipsolverStatus_t   cerr;
  PetscBool          iscuda;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (A->factortype == MAT_FACTOR_NONE) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
  if (!dA->d_fact_work) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
  ierr = PetscMPIIntCast(A->rmap->n,&n);CHKERRQ(ierr);
  /* MatSolve does not have a dispatching mechanism, we may end up with a VECSTANDARD here */
  ierr = PetscObjectTypeCompareAny((PetscObject)yy,&iscuda,VECSEQCUDA,VECMPICUDA,"");CHKERRQ(ierr);
  if (iscuda) {
    ierr = VecCopy(xx,yy);CHKERRQ(ierr);
    ierr = VecCUDAGetArray(yy,&y);CHKERRQ(ierr);
  } else {
    if (!dA->workvec) {
      ierr = MatCreateVecs(A,&dA->workvec,NULL);CHKERRQ(ierr);
    }
    ierr = VecCopy(xx,dA->workvec);CHKERRQ(ierr);
    ierr = VecCUDAGetArray(dA->workvec,&y);CHKERRQ(ierr);
  }
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(a->lda,&lda);CHKERRQ(ierr);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (A->factortype == MAT_FACTOR_LU) {
    ierr = PetscInfo2(A,"LU solve %d x %d on backend\n",n,n);CHKERRQ(ierr);
    cerr = hipsolverDnXgetrs(handle,trans ? HIPBLAS_OP_T : HIPBLAS_OP_N,n,one,da,lda,dA->d_fact_ipiv,y,n,dA->d_fact_info);CHKERRCUSOLVER(cerr);
  } else if (A->factortype == MAT_FACTOR_CHOLESKY) {
    ierr = PetscInfo2(A,"Cholesky solve %d x %d on backend\n",n,n);CHKERRQ(ierr);
    if (!dA->d_fact_ipiv) { /* spd */
      /* ========= Program hit hipErrorNotReady (error 34) due to "device not ready" on CUDA API call to hipEventQuery. */
      cerr = hipsolverDnXpotrs(handle,HIPBLAS_FILL_MODE_LOWER,n,one,da,lda,y,n,dA->d_fact_info);CHKERRCUSOLVER(cerr);
    } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"cusolverDnsytrs not implemented");
  } else SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"Unknown factor type %d",A->factortype);
  ccer = WaitForCUDA();CHKERRCUDA(ccer);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  if (iscuda) {
    ierr = VecCUDARestoreArray(yy,&y);CHKERRQ(ierr);
  } else {
    ierr = VecCUDARestoreArray(dA->workvec,&y);CHKERRQ(ierr);
    ierr = VecCopy(dA->workvec,yy);CHKERRQ(ierr);
  }
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
#if defined(PETSC_USE_DEBUG)
  ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(int), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
  if (info > 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
  else if (info < 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
  ierr = PetscLogGpuFlops(2.0*n*n - n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr = MatSolve_SeqDenseCUDA_Private(A,xx,yy,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolveTranspose_SeqDenseCUDA(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr = MatSolve_SeqDenseCUDA_Private(A,xx,yy,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatLUFactor_SeqDenseCUDA(Mat A,IS rperm,IS cperm,const MatFactorInfo *factinfo)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  int                m,n,lda;
#if defined(PETSC_USE_DEBUG)
  int                info;
#endif
  hipsolverStatus_t   cerr;
  hipsolverHandle_t handle;
  hipError_t        ccer;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArray(A,&da);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->cmap->n,&n);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(a->lda,&lda);CHKERRQ(ierr);
  ierr = PetscInfo2(A,"LU factor %d x %d on backend\n",m,n);CHKERRQ(ierr);
  if (!dA->d_fact_ipiv) {
    ccer = hipMalloc((void**)&dA->d_fact_ipiv,n*sizeof(*dA->d_fact_ipiv));CHKERRCUDA(ccer);
  }
  if (!dA->fact_lwork) {
    cerr = hipsolverDnXgetrf_bufferSize(handle,m,n,da,lda,&dA->fact_lwork);CHKERRCUSOLVER(cerr);
    ccer = hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work));CHKERRCUDA(ccer);
  }
  if (!dA->d_fact_info) {
    ccer = hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info));CHKERRCUDA(ccer);
  }
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  cerr = hipsolverDnXgetrf(handle,m,n,da,lda,dA->d_fact_work,dA->d_fact_ipiv,dA->d_fact_info);CHKERRCUSOLVER(cerr);
  ccer = WaitForCUDA();CHKERRCUDA(ccer);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArray(A,&da);CHKERRQ(ierr);
#if defined(PETSC_USE_DEBUG)
  ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(int), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
  if (info > 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_MAT_LU_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
  else if (info < 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
  A->factortype = MAT_FACTOR_LU;
  ierr = PetscLogGpuFlops(2.0*n*n*m/3.0);CHKERRQ(ierr);

  A->ops->solve          = MatSolve_SeqDenseCUDA;
  A->ops->solvetranspose = MatSolveTranspose_SeqDenseCUDA;
  A->ops->matsolve       = MatMatSolve_SeqDenseCUDA;

  ierr = PetscFree(A->solvertype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATSOLVERCUDA,&A->solvertype);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCholeskyFactor_SeqDenseCUDA(Mat A,IS perm,const MatFactorInfo *factinfo)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  int                n,lda;
#if defined(PETSC_USE_DEBUG)
  int                info;
#endif
  hipsolverStatus_t   cerr;
  hipsolverHandle_t handle;
  hipError_t        ccer;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->rmap->n,&n);CHKERRQ(ierr);
  ierr = PetscInfo2(A,"Cholesky factor %d x %d on backend\n",n,n);CHKERRQ(ierr);
  if (A->spd) {
    ierr = MatDenseCUDAGetArray(A,&da);CHKERRQ(ierr);
    ierr = PetscMPIIntCast(a->lda,&lda);CHKERRQ(ierr);
    if (!dA->fact_lwork) {
      cerr = hipsolverDnXpotrf_bufferSize(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,&dA->fact_lwork);CHKERRCUSOLVER(cerr);
      ccer = hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work));CHKERRCUDA(ccer);
    }
    if (!dA->d_fact_info) {
      ccer = hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info));CHKERRCUDA(ccer);
    }
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cerr = hipsolverDnXpotrf(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info);CHKERRCUSOLVER(cerr);
    ccer = WaitForCUDA();CHKERRCUDA(ccer);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);

    ierr = MatDenseCUDARestoreArray(A,&da);CHKERRQ(ierr);
#if defined(PETSC_USE_DEBUG)
    ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(int), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
    if (info > 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
    else if (info < 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
    A->factortype = MAT_FACTOR_CHOLESKY;
    ierr = PetscLogGpuFlops(1.0*n*n*n/3.0);CHKERRQ(ierr);
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"cusolverDnsytrs unavailable. Use MAT_FACTOR_LU");
#if 0
    /* at the time of writing this interface (cuda 10.0), cusolverDn does not implement *sytrs and *hetr* routines
       The code below should work, and it can be activated when *sytrs routines will be available */
    if (!dA->d_fact_ipiv) {
      ccer = hipMalloc((void**)&dA->d_fact_ipiv,n*sizeof(*dA->d_fact_ipiv));CHKERRCUDA(ccer);
    }
    if (!dA->fact_lwork) {
      cerr = cusolverDnXsytrf_bufferSize(handle,n,da,lda,&dA->fact_lwork);CHKERRCUSOLVER(cerr);
      ccer = hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work));CHKERRCUDA(ccer);
    }
    if (!dA->d_fact_info) {
      ccer = hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info));CHKERRCUDA(ccer);
    }
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cerr = cusolverDnXsytrf(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,dA->d_fact_ipiv,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info);CHKERRCUSOLVER(cerr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
#endif

  A->ops->solve          = MatSolve_SeqDenseCUDA;
  A->ops->solvetranspose = MatSolveTranspose_SeqDenseCUDA;
  A->ops->matsolve       = MatMatSolve_SeqDenseCUDA;

  ierr = PetscFree(A->solvertype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATSOLVERCUDA,&A->solvertype);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/* GEMM kernel: C = op(A)*op(B), tA, tB flag transposition */
PETSC_INTERN PetscErrorCode MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(Mat A,Mat B,Mat C,PetscBool tA,PetscBool tB)
{
  const PetscScalar *da,*db;
  PetscScalar       *dc;
  PetscScalar       one=1.0,zero=0.0;
  int               m,n,k;
  PetscInt          alda,blda,clda;
  PetscErrorCode    ierr;
  hipblasHandle_t    cublasv2handle;
  PetscBool         Aiscuda,Biscuda;
  hipblasStatus_t    berr;
  hipError_t       cerr;

  PetscFunctionBegin;
  /* we may end up with SEQDENSE as one of the arguments */
  ierr = PetscObjectTypeCompare((PetscObject)A,MATSEQDENSECUDA,&Aiscuda);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)B,MATSEQDENSECUDA,&Biscuda);CHKERRQ(ierr);
  if (!Aiscuda) {
    ierr = MatConvert(A,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&A);CHKERRQ(ierr);
  }
  if (!Biscuda) {
    ierr = MatConvert(B,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&B);CHKERRQ(ierr);
  }
  ierr = PetscMPIIntCast(C->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(C->cmap->n,&n);CHKERRQ(ierr);
  if (tA) {
    ierr = PetscMPIIntCast(A->rmap->n,&k);CHKERRQ(ierr);
  } else {
    ierr = PetscMPIIntCast(A->cmap->n,&k);CHKERRQ(ierr);
  }
  if (!m || !n || !k) PetscFunctionReturn(0);
  ierr = PetscInfo3(C,"Matrix-Matrix product %d x %d x %d on backend\n",m,k,n);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(B,&db);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayWrite(C,&dc);CHKERRQ(ierr);
  ierr = MatDenseGetLDA(A,&alda);CHKERRQ(ierr);
  ierr = MatDenseGetLDA(B,&blda);CHKERRQ(ierr);
  ierr = MatDenseGetLDA(C,&clda);CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  berr = cublasXgemm(cublasv2handle,tA ? HIPBLAS_OP_T : HIPBLAS_OP_N,tB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                     m,n,k,&one,da,alda,db,blda,&zero,dc,clda);CHKERRCUBLAS(berr);
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(1.0*m*n*k + 1.0*m*n*(k-1));CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(B,&db);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayWrite(C,&dc);CHKERRQ(ierr);
  if (!Aiscuda) {
    ierr = MatConvert(A,MATSEQDENSE,MAT_INPLACE_MATRIX,&A);CHKERRQ(ierr);
  }
  if (!Biscuda) {
    ierr = MatConvert(B,MATSEQDENSE,MAT_INPLACE_MATRIX,&B);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatTransposeMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA(Mat A,Mat B,Mat C)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(A,B,C,PETSC_TRUE,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA(Mat A,Mat B,Mat C)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(A,B,C,PETSC_FALSE,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMatTransposeMultNumeric_SeqDenseCUDA_SeqDenseCUDA(Mat A,Mat B,Mat C)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(A,B,C,PETSC_FALSE,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatProductSetFromOptions_SeqDenseCUDA(Mat C)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatProductSetFromOptions_SeqDense(C);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/* zz = op(A)*xx + yy
   if yy == NULL, only MatMult */
static PetscErrorCode MatMultAdd_SeqDenseCUDA_Private(Mat A,Vec xx,Vec yy,Vec zz,PetscBool trans)
{
  Mat_SeqDense      *mat = (Mat_SeqDense*)A->data;
  const PetscScalar *xarray,*da;
  PetscScalar       *zarray;
  PetscScalar       one=1.0,zero=0.0;
  int               m, n, lda; /* Use PetscMPIInt as it is typedef'ed to int */
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    berr;
  PetscErrorCode    ierr;

  PetscFunctionBegin;
  if (yy && yy != zz) { /* mult add */
    ierr = VecCopy_SeqCUDA(yy,zz);CHKERRQ(ierr);
  }
  if (!A->rmap->n || !A->cmap->n) {
    if (!yy) { /* mult only */
      ierr = VecSet_SeqCUDA(zz,0.0);CHKERRQ(ierr);
    }
    PetscFunctionReturn(0);
  }
  ierr = PetscInfo2(A,"Matrix-vector product %d x %d on backend\n",A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->cmap->n,&n);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(mat->lda,&lda);CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArray(zz,&zarray);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  berr = cublasXgemv(cublasv2handle,trans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                     m,n,&one,da,lda,xarray,1,(yy ? &one : &zero),zarray,1);CHKERRCUBLAS(berr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*A->rmap->n*A->cmap->n - (yy ? 0 : A->rmap->n));CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArray(zz,&zarray);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultAdd_SeqDenseCUDA(Mat A,Vec xx,Vec yy,Vec zz)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqDenseCUDA_Private(A,xx,yy,zz,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultTransposeAdd_SeqDenseCUDA(Mat A,Vec xx,Vec yy,Vec zz)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqDenseCUDA_Private(A,xx,yy,zz,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMult_SeqDenseCUDA(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqDenseCUDA_Private(A,xx,NULL,yy,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultTranspose_SeqDenseCUDA(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqDenseCUDA_Private(A,xx,NULL,yy,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetArrayRead_SeqDenseCUDA(Mat A,const PetscScalar **array)
{
  Mat_SeqDense   *mat = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSeqDenseCUDACopyFromGPU(A);CHKERRQ(ierr);
  *array = mat->v;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetArrayWrite_SeqDenseCUDA(Mat A,PetscScalar **array)
{
  Mat_SeqDense   *mat = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (!mat->v) { /* MatCreateSeqDenseCUDA may not allocate CPU memory. Allocate if needed */
    ierr = MatSeqDenseSetPreallocation(A,NULL);CHKERRQ(ierr);
  }
  *array = mat->v;
  A->offloadmask = PETSC_OFFLOAD_CPU;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetArray_SeqDenseCUDA(Mat A,PetscScalar **array)
{
  Mat_SeqDense   *mat = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSeqDenseCUDACopyFromGPU(A);CHKERRQ(ierr);
  *array = mat->v;
  A->offloadmask = PETSC_OFFLOAD_CPU;
  PetscFunctionReturn(0);
}

PetscErrorCode MatScale_SeqDenseCUDA(Mat Y,PetscScalar alpha)
{
  Mat_SeqDense   *y = (Mat_SeqDense*)Y->data;
  PetscScalar    *dy;
  int            j,N,m,lday,one = 1;
  hipblasHandle_t cublasv2handle;
  hipblasStatus_t berr;
  PetscErrorCode ierr;
  hipError_t    cerr;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArray(Y,&dy);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(Y->rmap->n*Y->cmap->n,&N);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(Y->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(y->lda,&lday);CHKERRQ(ierr);
  ierr = PetscInfo2(Y,"Performing Scale %d x %d on backend\n",Y->rmap->n,Y->cmap->n);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (lday>m) {
    for (j=0; j<Y->cmap->n; j++) {
      berr = cublasXscal(cublasv2handle,m,&alpha,dy+lday*j,one);CHKERRCUBLAS(berr);
    }
  } else {
    berr = cublasXscal(cublasv2handle,N,&alpha,dy,one);CHKERRCUBLAS(berr);
  }
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(N);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArray(Y,&dy);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatAXPY_SeqDenseCUDA(Mat Y,PetscScalar alpha,Mat X,MatStructure str)
{
  Mat_SeqDense      *x = (Mat_SeqDense*)X->data;
  Mat_SeqDense      *y = (Mat_SeqDense*)Y->data;
  const PetscScalar *dx;
  PetscScalar       *dy;
  int               j,N,m,ldax,lday,one = 1;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    berr;
  PetscErrorCode    ierr;
  hipError_t       cerr;

  PetscFunctionBegin;
  if (!X->rmap->n || !X->cmap->n) PetscFunctionReturn(0);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(X,&dx);CHKERRQ(ierr);
  if (alpha != 0.0) {
    ierr = MatDenseCUDAGetArray(Y,&dy);CHKERRQ(ierr);
  } else {
    ierr = MatDenseCUDAGetArrayWrite(Y,&dy);CHKERRQ(ierr);
  }
  ierr = PetscMPIIntCast(X->rmap->n*X->cmap->n,&N);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(X->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(x->lda,&ldax);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(y->lda,&lday);CHKERRQ(ierr);
  ierr = PetscInfo2(Y,"Performing AXPY %d x %d on backend\n",Y->rmap->n,Y->cmap->n);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (ldax>m || lday>m) {
    for (j=0; j<X->cmap->n; j++) {
      berr = cublasXaxpy(cublasv2handle,m,&alpha,dx+j*ldax,one,dy+j*lday,one);CHKERRCUBLAS(berr);
    }
  } else {
    berr = cublasXaxpy(cublasv2handle,N,&alpha,dx,one,dy,one);CHKERRCUBLAS(berr);
  }
  cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(PetscMax(2.*N-1,0));CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(X,&dx);CHKERRQ(ierr);
  if (alpha != 0.0) {
    ierr = MatDenseCUDARestoreArray(Y,&dy);CHKERRQ(ierr);
  } else {
    ierr = MatDenseCUDARestoreArrayWrite(Y,&dy);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatReset_SeqDenseCUDA(Mat A)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  hipError_t      cerr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (dA) {
    if (dA->unplacedarray) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"MatDenseCUDAResetArray() must be called first");
    if (!dA->user_alloc) { cerr = hipFree(dA->d_v);CHKERRCUDA(cerr); }
    cerr = hipFree(dA->d_fact_ipiv);CHKERRCUDA(cerr);
    cerr = hipFree(dA->d_fact_info);CHKERRCUDA(cerr);
    cerr = hipFree(dA->d_fact_work);CHKERRCUDA(cerr);
    ierr = VecDestroy(&dA->workvec);CHKERRQ(ierr);
  }
  ierr = PetscFree(A->spptr);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatDestroy_SeqDenseCUDA(Mat A)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  /* prevent to copy back data if we own the data pointer */
  if (!a->user_alloc) { A->offloadmask = PETSC_OFFLOAD_CPU; }
  ierr = MatConvert_SeqDenseCUDA_SeqDense(A,MATSEQDENSE,MAT_INPLACE_MATRIX,&A);CHKERRQ(ierr);
  ierr = MatDestroy_SeqDense(A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatDuplicate_SeqDenseCUDA(Mat A,MatDuplicateOption cpvalues,Mat *B)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate(PetscObjectComm((PetscObject)A),B);CHKERRQ(ierr);
  ierr = MatSetSizes(*B,A->rmap->n,A->cmap->n,A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  ierr = MatSetType(*B,((PetscObject)A)->type_name);CHKERRQ(ierr);
  ierr = MatDuplicateNoCreate_SeqDense(*B,A,cpvalues);CHKERRQ(ierr);
  if (cpvalues == MAT_COPY_VALUES && A->offloadmask != PETSC_OFFLOAD_CPU) {
    Mat_SeqDense      *a = (Mat_SeqDense*)A->data;
    const PetscScalar *da;
    PetscScalar       *db;
    hipError_t       cerr;
    PetscInt          ldb;

    ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
    ierr = MatDenseCUDAGetArrayWrite(*B,&db);CHKERRQ(ierr);
    ierr = MatDenseGetLDA(*B,&ldb);CHKERRQ(ierr);
    if (a->lda > A->rmap->n || ldb > A->rmap->n) {
      PetscInt j,m = A->rmap->n;

      for (j=0; j<A->cmap->n; j++) { /* it can be done better */
        cerr = hipMemcpy(db+j*ldb,da+j*a->lda,m*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
      }
    } else {
      cerr = hipMemcpy(db,da,(sizeof(PetscScalar)*A->cmap->n)*A->rmap->n,hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
    }
    ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
    ierr = MatDenseCUDARestoreArrayWrite(*B,&db);CHKERRQ(ierr);
    (*B)->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

#include <petsc/private/vecimpl.h>

static PetscErrorCode MatGetColumnVector_SeqDenseCUDA(Mat A,Vec v,PetscInt col)
{
  Mat_SeqDense     *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;
  PetscScalar      *x;
  PetscBool        viscuda;
  hipError_t      cerr;

  PetscFunctionBegin;
  ierr = PetscObjectTypeCompareAny((PetscObject)v,&viscuda,VECSEQCUDA,VECMPICUDA,VECCUDA,"");CHKERRQ(ierr);
  if (viscuda && !v->boundtocpu) { /* update device data */
    ierr = VecCUDAGetArrayWrite(v,&x);CHKERRQ(ierr);
    if (A->offloadmask & PETSC_OFFLOAD_GPU) {
      cerr = hipMemcpy(x,dA->d_v + col*a->lda,A->rmap->n*sizeof(PetscScalar),hipMemcpyHostToHost);CHKERRCUDA(cerr);
    } else {
      cerr = hipMemcpy(x,a->v + col*a->lda,A->rmap->n*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    }
    ierr = VecCUDARestoreArrayWrite(v,&x);CHKERRQ(ierr);
  } else { /* update host data */
    ierr = VecGetArrayWrite(v,&x);CHKERRQ(ierr);
    if (A->offloadmask & PETSC_OFFLOAD_CPU) {
      ierr = PetscArraycpy(x,a->v+col*a->lda,A->rmap->n);CHKERRQ(ierr);
    } else {
      cerr = hipMemcpy(x,dA->d_v + col*a->lda,A->rmap->n*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    }
    ierr = VecRestoreArrayWrite(v,&x);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatGetFactor_seqdense_cuda(Mat A,MatFactorType ftype,Mat *fact)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate(PetscObjectComm((PetscObject)A),fact);CHKERRQ(ierr);
  ierr = MatSetSizes(*fact,A->rmap->n,A->cmap->n,A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  ierr = MatSetType(*fact,MATSEQDENSECUDA);CHKERRQ(ierr);
  if (ftype == MAT_FACTOR_LU || ftype == MAT_FACTOR_ILU) {
    (*fact)->ops->lufactorsymbolic = MatLUFactorSymbolic_SeqDense;
  } else {
    (*fact)->ops->choleskyfactorsymbolic = MatCholeskyFactorSymbolic_SeqDense;
  }
  (*fact)->factortype = ftype;
  ierr = PetscFree((*fact)->solvertype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATSOLVERCUDA,&(*fact)->solvertype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATORDERINGEXTERNAL,(char**)&(*fact)->preferredordering[MAT_FACTOR_LU]);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATORDERINGEXTERNAL,(char**)&(*fact)->preferredordering[MAT_FACTOR_ILU]);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATORDERINGEXTERNAL,(char**)&(*fact)->preferredordering[MAT_FACTOR_CHOLESKY]);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATORDERINGEXTERNAL,(char**)&(*fact)->preferredordering[MAT_FACTOR_ICC]);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetColumnVec_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (a->vecinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  if (a->matinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  ierr = MatDenseCUDAGetArray(A,(PetscScalar**)&a->ptrinuse);CHKERRQ(ierr);
  if (!a->cvec) { /* we pass the data of A, to prevent allocating needless GPU memory the first time VecCUDAPlaceArray is called */
    ierr = VecCreateSeqCUDAWithArray(PetscObjectComm((PetscObject)A),A->rmap->bs,A->rmap->n,a->ptrinuse,&a->cvec);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)A,(PetscObject)a->cvec);CHKERRQ(ierr);
  }
  a->vecinuse = col + 1;
  ierr = VecCUDAPlaceArray(a->cvec,a->ptrinuse + (size_t)col * (size_t)a->lda);CHKERRQ(ierr);
  *v   = a->cvec;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreColumnVec_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (!a->vecinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseGetColumnVec() first");
  if (!a->cvec) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing internal column vector");
  a->vecinuse = 0;
  ierr = VecCUDAResetArray(a->cvec);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArray(A,(PetscScalar**)&a->ptrinuse);CHKERRQ(ierr);
  *v   = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetColumnVecRead_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (a->vecinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  if (a->matinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  ierr = MatDenseCUDAGetArrayRead(A,&a->ptrinuse);CHKERRQ(ierr);
  if (!a->cvec) { /* we pass the data of A, to prevent allocating needless GPU memory the first time VecCUDAPlaceArray is called */
    ierr = VecCreateSeqCUDAWithArray(PetscObjectComm((PetscObject)A),A->rmap->bs,A->rmap->n,a->ptrinuse,&a->cvec);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)A,(PetscObject)a->cvec);CHKERRQ(ierr);
  }
  a->vecinuse = col + 1;
  ierr = VecCUDAPlaceArray(a->cvec,a->ptrinuse + (size_t)col * (size_t)a->lda);CHKERRQ(ierr);
  ierr = VecLockReadPush(a->cvec);CHKERRQ(ierr);
  *v   = a->cvec;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreColumnVecRead_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (!a->vecinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseGetColumnVec() first");
  if (!a->cvec) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing internal column vector");
  a->vecinuse = 0;
  ierr = VecLockReadPop(a->cvec);CHKERRQ(ierr);
  ierr = VecCUDAResetArray(a->cvec);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&a->ptrinuse);CHKERRQ(ierr);
  *v   = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetColumnVecWrite_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (a->vecinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  if (a->matinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  ierr = MatDenseCUDAGetArrayWrite(A,(PetscScalar**)&a->ptrinuse);CHKERRQ(ierr);
  if (!a->cvec) { /* we pass the data of A, to prevent allocating needless GPU memory the first time VecCUDAPlaceArray is called */
    ierr = VecCreateSeqCUDAWithArray(PetscObjectComm((PetscObject)A),A->rmap->bs,A->rmap->n,a->ptrinuse,&a->cvec);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)A,(PetscObject)a->cvec);CHKERRQ(ierr);
  }
  a->vecinuse = col + 1;
  ierr = VecCUDAPlaceArray(a->cvec,a->ptrinuse + (size_t)col * (size_t)a->lda);CHKERRQ(ierr);
  *v   = a->cvec;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreColumnVecWrite_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (!a->vecinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseGetColumnVec() first");
  if (!a->cvec) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing internal column vector");
  a->vecinuse = 0;
  ierr = VecCUDAResetArray(a->cvec);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayWrite(A,(PetscScalar**)&a->ptrinuse);CHKERRQ(ierr);
  *v   = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetSubMatrix_SeqDenseCUDA(Mat A,PetscInt cbegin,PetscInt cend,Mat *v)
{
  Mat_SeqDense     *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (a->vecinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  if (a->matinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  if (a->cmat && cend-cbegin != a->cmat->cmap->N) {
    ierr = MatDestroy(&a->cmat);CHKERRQ(ierr);
  }
  ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr);
  if (!a->cmat) {
    ierr = MatCreateDenseCUDA(PetscObjectComm((PetscObject)A),A->rmap->n,PETSC_DECIDE,A->rmap->N,cend-cbegin,dA->d_v + (size_t)cbegin * (size_t)a->lda,&a->cmat);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)A,(PetscObject)a->cmat);CHKERRQ(ierr);
  } else {
    ierr = MatDenseCUDAPlaceArray(a->cmat,dA->d_v + (size_t)cbegin * (size_t)a->lda);CHKERRQ(ierr);
  }
  ierr = MatDenseSetLDA(a->cmat,a->lda);CHKERRQ(ierr);
  if (a->v) { ierr = MatDensePlaceArray(a->cmat,a->v + (size_t)cbegin * (size_t)a->lda);CHKERRQ(ierr); }
  a->cmat->offloadmask = A->offloadmask;
  a->matinuse = cbegin + 1;
  *v = a->cmat;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreSubMatrix_SeqDenseCUDA(Mat A,Mat *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (!a->matinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseGetSubMatrix() first");
  if (!a->cmat) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing internal column matrix");
  if (*v != a->cmat) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Not the matrix obtained from MatDenseGetSubMatrix()");
  a->matinuse = 0;
  A->offloadmask = PETSC_OFFLOAD_GPU;
  ierr = MatDenseCUDAResetArray(a->cmat);CHKERRQ(ierr);
  ierr = MatDenseResetArray(a->cmat);CHKERRQ(ierr);
  *v = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode  MatDenseSetLDA_SeqDenseCUDA(Mat A,PetscInt lda)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscBool        data;

  PetscFunctionBegin;
  data = (PetscBool)((A->rmap->n > 0 && A->cmap->n > 0) ? (dA->d_v ? PETSC_TRUE : PETSC_FALSE) : PETSC_FALSE);
  if (!dA->user_alloc && data && cA->lda!=lda) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"LDA cannot be changed after allocation of internal storage");
  if (lda < A->rmap->n) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"LDA %D must be at least matrix dimension %D",lda,A->rmap->n);
  cA->lda = lda;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatBindToCPU_SeqDenseCUDA(Mat A,PetscBool flg)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (a->vecinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  if (a->matinuse) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  A->boundtocpu = flg;
  if (!flg) {
    PetscBool iscuda;

    ierr = PetscObjectTypeCompare((PetscObject)a->cvec,VECSEQCUDA,&iscuda);CHKERRQ(ierr);
    if (!iscuda) {
      ierr = VecDestroy(&a->cvec);CHKERRQ(ierr);
    }
    ierr = PetscObjectTypeCompare((PetscObject)a->cmat,MATSEQDENSECUDA,&iscuda);CHKERRQ(ierr);
    if (!iscuda) {
      ierr = MatDestroy(&a->cmat);CHKERRQ(ierr);
    }
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArray_C",MatDenseGetArray_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayRead_C",MatDenseGetArrayRead_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayWrite_C",MatDenseGetArrayWrite_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVec_C",MatDenseGetColumnVec_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVec_C",MatDenseRestoreColumnVec_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVecRead_C",MatDenseGetColumnVecRead_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVecRead_C",MatDenseRestoreColumnVecRead_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVecWrite_C",MatDenseGetColumnVecWrite_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVecWrite_C",MatDenseRestoreColumnVecWrite_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetSubMatrix_C",MatDenseGetSubMatrix_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreSubMatrix_C",MatDenseRestoreSubMatrix_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseSetLDA_C",MatDenseSetLDA_SeqDenseCUDA);CHKERRQ(ierr);

    A->ops->duplicate               = MatDuplicate_SeqDenseCUDA;
    A->ops->mult                    = MatMult_SeqDenseCUDA;
    A->ops->multadd                 = MatMultAdd_SeqDenseCUDA;
    A->ops->multtranspose           = MatMultTranspose_SeqDenseCUDA;
    A->ops->multtransposeadd        = MatMultTransposeAdd_SeqDenseCUDA;
    A->ops->matmultnumeric          = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA;
    A->ops->mattransposemultnumeric = MatMatTransposeMultNumeric_SeqDenseCUDA_SeqDenseCUDA;
    A->ops->transposematmultnumeric = MatTransposeMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA;
    A->ops->axpy                    = MatAXPY_SeqDenseCUDA;
    A->ops->choleskyfactor          = MatCholeskyFactor_SeqDenseCUDA;
    A->ops->lufactor                = MatLUFactor_SeqDenseCUDA;
    A->ops->productsetfromoptions   = MatProductSetFromOptions_SeqDenseCUDA;
    A->ops->getcolumnvector         = MatGetColumnVector_SeqDenseCUDA;
    A->ops->scale                   = MatScale_SeqDenseCUDA;
    A->ops->copy                    = MatCopy_SeqDenseCUDA;
  } else {
    /* make sure we have an up-to-date copy on the CPU */
    ierr = MatSeqDenseCUDACopyFromGPU(A);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArray_C",MatDenseGetArray_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayRead_C",MatDenseGetArray_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayWrite_C",MatDenseGetArray_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVec_C",MatDenseGetColumnVec_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVec_C",MatDenseRestoreColumnVec_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVecRead_C",MatDenseGetColumnVecRead_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVecRead_C",MatDenseRestoreColumnVecRead_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVecWrite_C",MatDenseGetColumnVecWrite_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVecWrite_C",MatDenseRestoreColumnVecWrite_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetSubMatrix_C",MatDenseGetSubMatrix_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreSubMatrix_C",MatDenseRestoreSubMatrix_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseSetLDA_C",MatDenseSetLDA_SeqDense);CHKERRQ(ierr);

    A->ops->duplicate               = MatDuplicate_SeqDense;
    A->ops->mult                    = MatMult_SeqDense;
    A->ops->multadd                 = MatMultAdd_SeqDense;
    A->ops->multtranspose           = MatMultTranspose_SeqDense;
    A->ops->multtransposeadd        = MatMultTransposeAdd_SeqDense;
    A->ops->productsetfromoptions   = MatProductSetFromOptions_SeqDense;
    A->ops->matmultnumeric          = MatMatMultNumeric_SeqDense_SeqDense;
    A->ops->mattransposemultnumeric = MatMatTransposeMultNumeric_SeqDense_SeqDense;
    A->ops->transposematmultnumeric = MatTransposeMatMultNumeric_SeqDense_SeqDense;
    A->ops->axpy                    = MatAXPY_SeqDense;
    A->ops->choleskyfactor          = MatCholeskyFactor_SeqDense;
    A->ops->lufactor                = MatLUFactor_SeqDense;
    A->ops->productsetfromoptions   = MatProductSetFromOptions_SeqDense;
    A->ops->getcolumnvector         = MatGetColumnVector_SeqDense;
    A->ops->scale                   = MatScale_SeqDense;
    A->ops->copy                    = MatCopy_SeqDense;
  }
  if (a->cmat) {
    ierr = MatBindToCPU(a->cmat,flg);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatConvert_SeqDenseCUDA_SeqDense(Mat M,MatType type,MatReuse reuse,Mat *newmat)
{
  Mat              B;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (reuse == MAT_REUSE_MATRIX || reuse == MAT_INITIAL_MATRIX) {
    /* TODO these cases should be optimized */
    ierr = MatConvert_Basic(M,type,reuse,newmat);CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }

  B    = *newmat;
  ierr = MatBindToCPU_SeqDenseCUDA(B,PETSC_TRUE);CHKERRQ(ierr);
  ierr = MatReset_SeqDenseCUDA(B);CHKERRQ(ierr);
  ierr = PetscFree(B->defaultvectype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(VECSTANDARD,&B->defaultvectype);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSEQDENSE);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatConvert_seqdensecuda_seqdense_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArray_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArrayRead_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArrayWrite_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArray_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArrayRead_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArrayWrite_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAPlaceArray_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAResetArray_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAReplaceArray_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatProductSetFromOptions_seqaij_seqdensecuda_C",NULL);CHKERRQ(ierr);

  B->ops->bindtocpu = NULL;
  B->ops->destroy = MatDestroy_SeqDense;
  B->offloadmask = PETSC_OFFLOAD_CPU;
  PetscFunctionReturn(0);
}

PetscErrorCode MatConvert_SeqDense_SeqDenseCUDA(Mat M,MatType type,MatReuse reuse,Mat *newmat)
{
  Mat_SeqDenseCUDA *dB;
  Mat              B;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  ierr = PetscCUDAInitializeCheck();CHKERRQ(ierr);
  if (reuse == MAT_REUSE_MATRIX || reuse == MAT_INITIAL_MATRIX) {
    /* TODO these cases should be optimized */
    ierr = MatConvert_Basic(M,type,reuse,newmat);CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }

  B    = *newmat;
  ierr = PetscFree(B->defaultvectype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(VECCUDA,&B->defaultvectype);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSEQDENSECUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatConvert_seqdensecuda_seqdense_C",            MatConvert_SeqDenseCUDA_SeqDense);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArray_C",                        MatDenseCUDAGetArray_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArrayRead_C",                    MatDenseCUDAGetArrayRead_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArrayWrite_C",                   MatDenseCUDAGetArrayWrite_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArray_C",                    MatDenseCUDARestoreArray_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArrayRead_C",                MatDenseCUDARestoreArrayRead_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArrayWrite_C",               MatDenseCUDARestoreArrayWrite_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAPlaceArray_C",                      MatDenseCUDAPlaceArray_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAResetArray_C",                      MatDenseCUDAResetArray_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAReplaceArray_C",                    MatDenseCUDAReplaceArray_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatProductSetFromOptions_seqaij_seqdensecuda_C",MatProductSetFromOptions_SeqAIJ_SeqDense);CHKERRQ(ierr);

  ierr     = PetscNewLog(B,&dB);CHKERRQ(ierr);
  B->spptr = dB;

  B->offloadmask = PETSC_OFFLOAD_UNALLOCATED;

  ierr = MatBindToCPU_SeqDenseCUDA(B,PETSC_FALSE);CHKERRQ(ierr);
  B->ops->bindtocpu = MatBindToCPU_SeqDenseCUDA;
  B->ops->destroy  = MatDestroy_SeqDenseCUDA;
  PetscFunctionReturn(0);
}

/*@C
   MatCreateSeqDenseCUDA - Creates a sequential matrix in dense format using CUDA.

   Collective

   Input Parameters:
+  comm - MPI communicator
.  m - number of rows
.  n - number of columns
-  data - optional location of GPU matrix data.  Set data=NULL for PETSc
   to control matrix memory allocation.

   Output Parameter:
.  A - the matrix

   Notes:

   Level: intermediate

.seealso: MatCreate(), MatCreateSeqDense()
@*/
PetscErrorCode  MatCreateSeqDenseCUDA(MPI_Comm comm,PetscInt m,PetscInt n,PetscScalar *data,Mat *A)
{
  PetscErrorCode ierr;
  PetscMPIInt    size;

  PetscFunctionBegin;
  ierr = MPI_Comm_size(comm,&size);CHKERRMPI(ierr);
  if (size > 1) SETERRQ1(comm,PETSC_ERR_ARG_WRONG,"Invalid communicator size %d",size);
  ierr = MatCreate(comm,A);CHKERRQ(ierr);
  ierr = MatSetSizes(*A,m,n,m,n);CHKERRQ(ierr);
  ierr = MatSetType(*A,MATSEQDENSECUDA);CHKERRQ(ierr);
  ierr = MatSeqDenseCUDASetPreallocation(*A,data);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*MC
   MATSEQDENSECUDA - MATSEQDENSECUDA = "seqdensecuda" - A matrix type to be used for sequential dense matrices on GPUs.

   Options Database Keys:
. -mat_type seqdensecuda - sets the matrix type to "seqdensecuda" during a call to MatSetFromOptions()

  Level: beginner
M*/
PETSC_EXTERN PetscErrorCode MatCreate_SeqDenseCUDA(Mat B)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = PetscCUDAInitializeCheck();CHKERRQ(ierr);
  ierr = MatCreate_SeqDense(B);CHKERRQ(ierr);
  ierr = MatConvert_SeqDense_SeqDenseCUDA(B,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
